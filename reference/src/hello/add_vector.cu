
#include <hip/hip_runtime.h>
#include "stdio.h"

/*  First Hello World, Achim Grolms, 2009-10-20  */
//  nvcc -deviceemu -o add_vector_emu add_vector.cu


__global__ void add_arrays_gpu( float *in1, float *in2, float *out, int Ntot)
{
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	if ( i )
		out[i] = in1[i] + in2[i];
}

int main()
{
	/* pointers to host memory */
	float *a, *b, *c;
	/* pointers to device memory */
	float *a_d, *b_d, *c_d;
	int N=1800000;
    long i;

	/* Allocate arrays a, b and c on host*/
	a = (float*) malloc(N*sizeof(float));
	b = (float*) malloc(N*sizeof(float));
	c = (float*) malloc(N*sizeof(float));

	/* Allocate arrays a_d, b_d and c_d on device*/
	hipMalloc ((void **) &a_d, sizeof(float)*N);
	hipMalloc ((void **) &b_d, sizeof(float)*N);
	hipMalloc ((void **) &c_d, sizeof(float)*N);

	/* Initialize arrays a and b */
	for (i=0; i<N; i++)
	{
		a[i] = (float) 1;
		b[i] = (float) -1;
	}


	/* Copy data from host memory to device memory */
	hipMemcpy(a_d, a, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, sizeof(float)*N, hipMemcpyHostToDevice);

	/* Compute the execution configuration */
	int block_size=64;
	dim3 dimBlock(block_size);
	dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

	/* Add arrays a and b, store result in c */
	add_arrays_gpu<<<dimGrid,dimBlock>>>(a_d, b_d, c_d, N);

	/* Copy data from deveice memory to host memory */
	hipMemcpy(c, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

	/* Print c */
	for (i=0; i<N; i++)
		printf(" c[%d]=%f, a[%d]=%f, b[%d]=%f\n",i,c[i],i,a[i],i,b[i]);

	/* Free the memory */

    hipFree(a_d); hipFree(b_d);hipFree(c_d);
	free(a); free(b); free(c);


}

