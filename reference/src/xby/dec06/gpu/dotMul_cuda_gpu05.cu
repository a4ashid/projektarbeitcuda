#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */

__global__ void device_dotMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
	__shared__ float Cs[VECTOR_BLOCK_SIZE];
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	
	Cs[threadIdx.x] = 0;

	if ( idx < N ) {
	    Cs[threadIdx.x] = in1[ idx ] * in2[ idx ];
	}

	t_ve blocksum = 0;
	
	if(threadIdx.x==0){
		out[blockIdx.x]=0;
	}
	__syncthreads();
	
	if(threadIdx.x==0){
		int kEnd = N-(blockIdx.x*VECTOR_BLOCK_SIZE);
				if(kEnd > VECTOR_BLOCK_SIZE)kEnd = VECTOR_BLOCK_SIZE;
	    //for ( int i = 1; i < blockDim.x; i++ ) {
		for ( int i = 1; i < kEnd; i++ ) {
		     Cs[0] += Cs[i];
		}
		out[blockIdx.x]=Cs[0];
	}
	__syncthreads();
	if ( idx == 0 ) {
	     for ( int i = 1; i < gridDim.x; i++ ) {
		     out[0] += out[i];
		 }
	}

}