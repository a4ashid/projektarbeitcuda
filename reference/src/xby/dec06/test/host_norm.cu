#include "hip/hip_runtime.h"
#include "test_comm.h"

void host_norm(double* pIn, double *pOut, int sizeIn, int sizeOut)
{

	int i, j;
	double *data1, *data2;
	float *data1f, *data2f;
	float *data1f_gpu, *data2f_gpu;
	int sizeBlock;

	// variable for time measure
	int it;
	float t_avg;
	t_avg = 0;
	//ITERATE defined in project_comm.h
	it = ITERATE;

	sizeBlock = VECTOR_BLOCK_SIZE;
	// get Input data pointer
	data1 = pIn;
	data2 = pOut;

	// Find the dimensions of the data 

	// Create an mxArray for the output data 

	// Create an input and output data array on the GPU
	hipMalloc( (void **) &data1f_gpu,sizeof(float)*sizeIn);
	hipMalloc( (void **) &data2f_gpu,sizeof(float)*sizeOut);
	// Retrieve the input data 

	// Check if the input array is single or double precision 

	// The input array is in double precision, it needs to be converted t floats before being sent to the card 
	data1f = (float *) malloc(sizeof(float)*sizeIn);
	for (j = 0; j < sizeIn; j++)
	{
		data1f[j] = (float) data1[j];
	}


	hipMemcpy( data1f_gpu, data1f, sizeof(float)*sizeIn, hipMemcpyHostToDevice);

	data2f = (float *) malloc(sizeof(float)*sizeOut);
	//hipMemcpy( data2f_gpu, data2f, sizeof(float)*sizeOut, hipMemcpyHostToDevice);

	// Compute execution configuration using 128 threads per block 
	dim3 dimBlock(sizeBlock);
	dim3 dimGrid((sizeIn)/dimBlock.x);
	if ( (sizeIn) % sizeBlock !=0 ) dimGrid.x+=1;
	for (i = 0; i < it ; i++){
		clock_t startTime;
		clock_t endTime;
		startTime=clock();	 
	// Call function on GPU 
	norm_elements<<<dimGrid,dimBlock>>>(data1f_gpu, data2f_gpu, sizeIn);
	hipError_t e;
	e = hipGetLastError();
	if ( e != hipSuccess)
	{
		fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
		exit(-1);
	}
	
	endTime=clock();
	t_avg += endTime-startTime;
	}//for it
	printf("laufTime  in CPU = %lf (ms)\n", ((double) t_avg)*1000 /(it* CLOCKS_PER_SEC));
	
	// Copy result back to host 
	hipMemcpy( data2f, data2f_gpu, sizeof(float)*sizeOut, hipMemcpyDeviceToHost);

	// Create a pointer to the output data 

	// Convert from single to double before returning 
	for (j = 0; j < sizeOut; j++)
	{
		data2[j] = (double) data2f[j];
	}
	// Clean-up memory on device and host 
	free(data1f);
	free(data2f);
	hipFree(data1f_gpu);
	hipFree(data2f_gpu);
}

int test_norm()
{

    double *pIn, *pOut;
    int sizeIn, sizeOut;
    int i;
    sizeIn = 1000;
    sizeOut = sizeIn/VECTOR_BLOCK_SIZE;
    pIn = (double*)malloc(sizeof(double)*sizeIn);
    pOut = (double*)malloc(sizeof(double)*sizeOut);
    /*
	pIn[0] = 3;
    pIn[1] = 4;
    //pIn[2] = 3;
	*/
	for (i = 0; i < sizeIn; i++){
		pIn[i] = 1;
	}
    host_norm(pIn, pOut, sizeIn, sizeOut);
	
    printf("output square result");
    for (i = 0; i < sizeOut; i++)
    {	
        printf(" pOut[%d] = %lf, ", i, pOut[i]);
    }
        printf("\n");
	printf("output norm result");
    for (i = 0; i < sizeOut; i++)
    {
		//pOut[i] = sqrt(pOut[i]);
        printf("squre of pOut[%d] = %lf, ", i, pOut[i]);
    }
        printf("\n");   
    free(pIn);
    free(pOut);
    return 0;
}
int mexTest_norm(double *pIn,double *pOut,int sizeIn)
{
    //double *pOut;
    int sizeOut;
    int i;

	//sizeOut =sizeIn/VECTOR_BLOCK_SIZE + 1;
	sizeOut=1;
    //pOut = (double*)malloc(sizeof(double)*sizeOut);

    host_norm(pIn, pOut, sizeIn, sizeOut);
	double expect=sizeIn;
	//printf("output square result");
	
	//if(pOut[0] != expect){
		
		//for (i = 0; i < sizeOut; i++)
		//{	
			//printf(" pOut[%d] = %lf, ", i, pOut[i]);
		//}

	//}

    //free(pOut);
	
    return 0;

}