/*
mexUnitTestMain.cu
*/
#include <stdio.h>
#include "mex.h"
#include "host_dotMul.cu"
#include "host_norm.cu"
#include "host_matrixMul.cu"



void callTestFunction(double** ppIn,int *pmIn,int *pnIn, int ArgNum){
//call gpu
//testdotMul
printf("pnIn[0]=%d,pmIn[0]=%d, \n",pnIn[0],pmIn[0]);
if(ArgNum ==2){
	if((pmIn[0]==pmIn[1])&&(pnIn[0]==pnIn[1])){
		if(1==pnIn[0])mexTest_dotMul(ppIn[0],ppIn[1],pmIn[0]);
	}
}

//call cpu

}


/* Gateway function */
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{
     int inputArgNum = nrhs;
     int outputArgNum = nlhs;
     int i,j,k,m, n;
     double *pMatrix;
     double * pIn;  
     double ** ppIn;
     int *pmIn;
     int *pnIn;
     pnIn = (int*)mxMalloc(sizeof(int)*nrhs);
     pmIn = (int*)mxMalloc(sizeof(int)*nrhs);
     ppIn = (double**)mxMalloc(sizeof(double*)*nrhs);
     //printf("nrhs = %d \n",nrhs);
     for (i = 0; i < nrhs; i++){
        /* Find the dimensions of the data */
        m = mxGetM(prhs[i]);
        n = mxGetN(prhs[i]);
        printf("m = %d , n= %d \n",m,n);
        pmIn[i] = (int)m;
        pnIn[i] = (int)n;
        pMatrix = mxGetPr(prhs[i]); 
        pIn = (double*)mxMalloc(sizeof(double)*m*n);       
        ppIn[i]=pIn;
        
        for( k = 0; k < m; k++)
            for( j = 0; j < n; j++){
                pIn[k*n+j] = (double)pMatrix[j*m+k];
			//	printf("%f \n",pIn[k*n+j]);
            }
       
     }// for i
	 
	 //
	 callTestFunction(ppIn,pmIn,pnIn, inputArgNum);
	 
     for (i = 0; i < nrhs; i++){
        pIn=ppIn[i];
        //printf("m = %d , n= %d \n",pmIn[i],pnIn[i]);
        //for(j = 0; j < pnIn[i]*pmIn[i]; j++)printf("%f ,",pIn[j]);
        //printf("\n");
        mxFree(pIn);
     }
     
     mxFree(pnIn);
     mxFree(pmIn);
     mxFree(ppIn);
}