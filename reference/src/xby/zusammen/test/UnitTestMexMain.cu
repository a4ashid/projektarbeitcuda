/*
UnitTestMexMain.cu
*/
void callTestFunction(float** ppIn,int *pmIn,int *pnIn, int ArgNum){
//call gpu

//call

}

/* Gateway function */
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{
     int inputArgNum = nrhs;
     int outputArgNum = nlhs;
     int i,j,k,m, n;
     double *pMatrix;
     float * pIn;  
     float ** ppIn;
     int *pmIn;
     int *pnIn;
     pnIn = (int*)mxMalloc(sizeof(int)*nrhs);
     pmIn = (int*)mxMalloc(sizeof(int)*nrhs);
     ppIn = (float**)mxMalloc(sizeof(float*)*nrhs);
     printf("nrhs = %d \n",nrhs);
     for (i = 0; i < nrhs; i++){
        /* Find the dimensions of the data */
        m = mxGetM(prhs[i]);
        n = mxGetN(prhs[i]);
        printf("m = %d , n= %d \n",m,n);
        pmIn[i] = (int)m;
        pnIn[i] = (int)n;
        pMatrix = mxGetPr(prhs[i]); 
        pIn = (float*)mxMalloc(sizeof(float)*m*n);       
        ppIn[i]=pIn;
        
        for( k = 0; k < m; k++)
            for( j = 0; j < n; j++){
                pIn[k*n+j] = (float)pMatrix[j*m+k];
            }
        for(k=0; k < m*n; k++){
            //pIn[k] = (float)pMatrix[k];
            printf("%f \n",pIn[k]);
        }
       
     }// for i
	 
	 //
	 callTestFunction(ppIn,pmIn,pnIn, inputArgNum);
	 
     for (i = 0; i < nrhs; i++){
        pIn=ppIn[i];
        printf("m = %d , n= %d \n",pmIn[i],pnIn[i]);
        for(j = 0; j < pnIn[i]*pmIn[i]; j++)printf("%f ,",pIn[j]);
        printf("\n");
        mxFree(pIn);
     }
     
     mxFree(pnIn);
     mxFree(pmIn);
     mxFree(ppIn);
}
