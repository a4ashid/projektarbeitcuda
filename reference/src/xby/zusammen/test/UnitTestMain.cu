/*
UnitTestMain
*/

#include <stdio.h>
#define GPU 1
#ifdef GPU
#include "host_dotMul.cu"
#include "host_norm.cu"
#include "host_matrixMul.cu"
#else

#endif //ifdef GPU
int main()
{
	 //test_matrixMul();
     test_dotMul();
	 //test_norm();
	 return 1;
}
