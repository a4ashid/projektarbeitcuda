#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

//#include "mex.h"
/* Kernel to square elements of the array on the GPU */

/*
typedef struct{
        int width;
        int height;
        int stride;
        float * elements;
} Matrix;
///
__device__ float GetVectorElement(const Matrix A, int row, int offset){
          return A.elements[row * VECTOR_BLOCK_SIZE + offset];           
}

///??????????????????
__device__ void setVectorElement(Matrix A, int row, int offset, float value){
           A.elements[row * VECTOR_BLOCK_SIZE + offset] = value;           
}

__device__ Matrix GetSubVector(Matrix A, int row){
           Matrix Asub;
           Asub.width = 1;     
           Asub.height = VECTOR_BLOCK_SIZE;
           Asub.stride = 1;
           Asub.elements = & A.elements[row * VECTOR_BLOCK_SIZE]      
}
*/
/*
N size of Vector  
*/

__global__ void device_dotMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if(idx > N) return;
	if(idx == 0)out[blockIdx.x] = 0;
	__syncthreads();
	
   //block index
   int blockRow = blockIdx.x;
   // thread index
   int row = threadIdx.x;
   int aBegin = blockRow*VECTOR_BLOCK_SIZE;
   int aEnd = aBegin + VECTOR_BLOCK_SIZE - 1;
   int aStep = VECTOR_BLOCK_SIZE;
   //
   
   
   // comupted by the thread
   t_ve outValue = 0;
   //for (int a = aBegin;(a <= aEnd)&&(a <= N);a += aStep){
   for (int a = aBegin;(a <= aEnd);a += aStep){
         // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[VECTOR_BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[VECTOR_BLOCK_SIZE];
		
		__shared__ float Cs[VECTOR_BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        AS(row) = in1[a + row];
        BS(row) = in2[a + row];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();    
		
		Cs[row] = AS(row) * BS(row);
        
		/*
        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
		for (int k = 0; (k < VECTOR_BLOCK_SIZE)&&(k < N); ++k)
        //for (int k = 0; (k < VECTOR_BLOCK_SIZE); ++k)
            outValue += AS(k) * BS(k);
		*/
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();  
		
		if (row == 0) {
			
			for (int k = 0; (k < VECTOR_BLOCK_SIZE)&&(idx < N); k++)
			out[blockIdx.x] += Cs[k];
			//out[0] += 1;
			//outValue += 1;
		}
		__syncthreads();
   }
   //__syncthreads();
   
   if(idx==0){
		for(int k = 1; k <= gridDim.x; k++)out[0] += out[k];
   }
   
   //out[0] = outValue;


//	__syncthreads();

}

