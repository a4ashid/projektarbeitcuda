#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

//#include "mex.h"
/* Kernel to square elements of the array on the GPU */

/*
  create in 28.11.2009
 It should be runging only in more than one block.
 separate vectors in parts in length of VECTOR_BLOCK_SIZE

*/

/*

N size of Vector  
*/

__global__ void device_dotMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if(idx > N) return;
   //block index
   int blockRow = blockIdx.x;
   // thread index
   int row = threadIdx.x;
   int aBegin = blockRow*VECTOR_BLOCK_SIZE;   
   //int aEnd = aBegin + VECTOR_BLOCK_SIZE - 1;
   //working only in one block
   //int aBegin = 0;
   int aEnd = N;
   
   int aStep = VECTOR_BLOCK_SIZE;

   // comupted by the thread
   t_ve outValue = 0;
  
         // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[VECTOR_BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[VECTOR_BLOCK_SIZE];
		
		__shared__ float Cs[VECTOR_BLOCK_SIZE];
		Cs[row] = 0;

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        AS(row) = in1[aBegin + row];
        BS(row) = in2[aBegin + row];
        // Synchronize to make sure the matrices are loaded
        __syncthreads();    	
		
		Cs[row] = AS(row) * BS(row);   
		//Cs[row] = in1[aBegin + row] *in2[aBegin + row];   
		
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        
		__syncthreads();  
		if(threadIdx.x == 0)out[blockIdx.x] = 0;
		__syncthreads();
		
		// computing summe in one thread for one Loop 
		if (threadIdx.x == 0) {			
			for (int k = 1; (k < VECTOR_BLOCK_SIZE)&&(idx < N); k++){
                 Cs[0]+= Cs[k];
				 //out[k]= Cs[k];
            }   
			out[blockIdx.x] = Cs[0];
		}
		__syncthreads();
   //__syncthreads();   
   
   /*
   if(idx==0){
		for(int k = 1; k <= gridDim.x; k++)out[0] += out[k];
   }
   */
}

