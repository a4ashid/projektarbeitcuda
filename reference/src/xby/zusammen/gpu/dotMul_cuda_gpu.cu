#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */

__global__ void device_dotMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
	
 

	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	//__shared__ float vOut[16];
	if(idx == 0)out[0] = 0;
	
	//if ( idx < N)vOut[idx] = in1[idx]*in2[idx];
	if ( idx < N)out[idx] = in1[idx]*in2[idx];
	__syncthreads();
	
	//if(idx < N)out[0] += vOut[idx];
	
	
	if(idx == 0) {
		
		int i;
		for ( i = 1; i < N; i++ ) {
			//out[0] += vOut[i];
			out[0] += out[i];
		}
	}
	
	__syncthreads();

}

