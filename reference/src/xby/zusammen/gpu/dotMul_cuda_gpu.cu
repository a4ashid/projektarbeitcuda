#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */

__global__ void device_dotMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
 
	__shared__ float vOut[16];
	int idx = blockIdx.x*blockDim.x+threadIdx.x;

	if ( idx < N)vOut[idx] = in1[idx]*in2[idx];

	__syncthreads();

	if(idx == 0) {
		out[0] = 0;
		int i;
		for ( i = 0; i < N; i++ ) {
			out[0] += vOut[i];
		}
	}

	__syncthreads();

}

