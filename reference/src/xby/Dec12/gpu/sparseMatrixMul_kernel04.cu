#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

/* Kernel to square elements of the array on the GPU */
/*
	input Matrix pSparseMatrix  ,input Vector pVector 
	Vector pResultVector output vector 
	C=A*B
description:
	each row of A occuppy one block. if gridDim is smaller than the row number of A  
	
	enchance of sparseMatrixMul_kernel03.cu for supprting Vector size biger than Maximu block size
	
*/
__global__ void sparseMatrixMul(t_FullMatrix pResultVector,t_SparseMatrix pSparseMatrix, t_FullMatrix pVector)
{
	//__shared__ float As[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	//__shared__ float Bs[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	__shared__ float Cs[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	//define gridIndex, if gridDim < mA, gridIndex > 0; 
	int gridIndex = 0;
	//int idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
    t_ve *pMatrixElements, *pVectorElements, *pResultElements;
    unsigned int m, n;//, i, j;
    unsigned int *pRow, *pCol;
    //unsigned int colbegin, colend;
    pMatrixElements = pSparseMatrix.pNZElement;
    pVectorElements = pVector.pElement;
    pResultElements = pResultVector.pElement;
    m = pSparseMatrix.m;
    n = pSparseMatrix.n;
	//aBegin,aEnd,aStep are defined for 
	int aBegin = 0;
	int aEnd = pSparseMatrix.m;
	int aStep = gridDim.x;
	
	int bBegin = 0;
	//int aStep = gridDim.x;
	int bStep = VECTOR_BLOCK_SIZE; // blockDim.x
	//int aEnd = mA;
	int bEnd;
    //==check size of Arguments========================================================
    if(m != pResultVector.m*(pResultVector.n)){
        //printf("Result Vector does not match the Matrix\n");
        return;
    }   
    if(n != pVector.m*(pVector.n)){
        //printf("input Vector does not match the Matrix\n");
        return;
    }
	//pRow and pCol may should in share memory or texture
    pRow = pSparseMatrix.pRow;
    pCol = pSparseMatrix.pCol;
    //cal
	for(int a = aBegin; (a < aEnd)&&((gridIndex*gridDim.x+blockIdx.x)<aEnd); a += aStep, gridIndex++){
		if(threadIdx.x==0){
			//pResultElements[blockIdx.x]=0;
		//C[gridIndex*gridDim.x+blockIdx.x]=0;
			pResultElements[gridIndex*gridDim.x+blockIdx.x]=0;
		}
		__syncthreads();
		
		
	
		//following is operations within one block 
		// initialize the dot product for each row in A and vector B
		t_ve blocksum = 0;
		//if nB> blockDim, split repeat the
		bBegin = pRow[blockIdx.x];
		bEnd = pRow[blockIdx.x + 1];
		for(int b = bBegin; (b < bEnd)&&((threadIdx.x+b) < bEnd); b += bStep ) {
			//initialise Cs 
			//As[threadIdx.x] = 0;
			//Bs[threadIdx.x] = 0;// consider text memory
			Cs[threadIdx.x] = 0;
			__syncthreads();
			// compute scalar product
			// for (i = 0; i < m; i++){
			// colbegin = pRow[i];
			// colend = pRow[i+1];
			// for(j=colbegin;j<colend;j++)pResultElements[i] += pMatrixElements[j]*pVectorElements[pCol[j]];
			// } 
			if (( (gridIndex*gridDim.x+blockIdx.x)<aEnd)&&((b+threadIdx.x) < bEnd)) {
				
				Cs[threadIdx.x] = pMatrixElements[b + threadIdx.x] * pVectorElements[pCol[b + threadIdx.x ]];
			}
			__syncthreads();
				
			if(threadIdx.x == 0){
				int kEnd = bEnd-b;
				if(kEnd > VECTOR_BLOCK_SIZE)kEnd = VECTOR_BLOCK_SIZE;
				//Because I add Cs[0...k], if blockSize and Matrix does not fit, Parts of Cs[k] are not initialized as 0.  		
				for (int k = 0; k < kEnd; k++) blocksum += Cs[k];
			
			}
			__syncthreads();
			
			//Cs[threadIdx.x] = 0;
			//__syncthreads();	
		}//for b
		__syncthreads();

		if(threadIdx.x == 0) pResultElements[blockIdx.x] = blocksum;//?????????????
		__syncthreads();	
    
	}//for {int a = aBegin;....
}