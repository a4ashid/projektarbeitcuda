#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"
//#include "mex.h"
/*
	release norm_cuda_gpu02
 Kernel to square elements of the array on the GPU
 */


__global__ void norm_elements(t_ve* in,t_ve* out, unsigned int N)
{
	__shared__ float Cs[VECTOR_BLOCK_SIZE];
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	
	Cs[threadIdx.x] = 0;

	if ( idx < N ) {
	    Cs[threadIdx.x] = in[ idx ] * in[ idx ];
	}

	t_ve blocksum = 0;
	
	if(threadIdx.x==0){
		out[blockIdx.x]=0;
	}
	__syncthreads();
	
	if(threadIdx.x==0){
	    for ( int i = 0; i < blockDim.x; i++ ) {
		     blocksum += Cs[i];
		}
		out[blockIdx.x]=blocksum;
	}
	__syncthreads();
	if ( idx == 0 ) {
	     for ( int i = 1; i < gridDim.x; i++ ) {
		     out[0] += out[i];
		 }
		out[0] = sqrt(out[0]); 
	}

}