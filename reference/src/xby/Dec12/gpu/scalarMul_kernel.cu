#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

/* Kernel to computing Vector x scalar on the GPU */

/*scalarMul_kernel.cu*/
//pIn1: input Vector;
// N: Vectorsize;
// pIn2: input scalar;
//pOut: output Vector;
__global__ void device_scalarMul(t_ve* pIn1, t_ve* pIn2,t_ve* pOut, unsigned int N)
{
	//__shared__ float Cs[VECTOR_BLOCK_SIZE];
	
	int tid = threadIdx.x;
	int idx = blockIdx.x*blockDim.x + tid;
	
	//Cs[threadIdx.x] = 0;

	if ( idx < N ) {
	    //Cs[threadIdx.x] = pin1[ idx ] * in2;
		pOut[idx] = pIn1[idx]*pIn2[0];
	}

}