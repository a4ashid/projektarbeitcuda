#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */

/*
typedef struct{
        int width;
        int height;
        int stride;
        float * elements;
} Matrix;

*/
/*
N size of Vector  
*/

__global__ void norm_elements(float* in, float* out, unsigned int N)
{
   //block index
   int blockRow = blockIdx.x;
   // thread index
   int row = threadIdx.x;
   int aBegin = blockRow*VECTOR_BLOCK_SIZE;
   int aEnd = aBegin + VECTOR_BLOCK_SIZE - 1;
   int aStep = VECTOR_BLOCK_SIZE;
   //
   // comupted by the thread
   t_ve outValue = 0;
   
   for (int a = aBegin;a <= aEnd;a += aStep){
         // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[VECTOR_BLOCK_SIZE];



        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        AS(row) = in1[a + row];


        // Synchronize to make sure the matrices are loaded
        __syncthreads();    
        
        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < VECTOR_BLOCK_SIZE; ++k)
            outValue += AS(k) * AS(k);

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();  
   }
   
   out[0] = outValue;
   out[0] = sqrt(out[0]);


//	__syncthreads();

}

