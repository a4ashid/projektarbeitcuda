#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

/* Kernel to square elements of the array on the GPU */
/*	sparseMatrixMul_kernel05.cu
	input Matrix pSparseMatrix  ,input Vector pVector 
	Vector pResultVector output vector 
	C=A*B
description:
	
	each row of A occuppy one block. if gridDim is smaller than the row number of A  
	
	enchance of sparseMatrixMul_kernel03.cu for supprting Vector size biger than Maximu block size
	
*/

__device__ unsigned int getSparseRowIdx(unsigned int gridIdx,unsigned int gridStep,unsigned int blockIdx,unsigned int blockStepY,unsigned int threadIdxY)
{
	return gridIdx*gridStep+blockIdx*blockStepY+threadIdxY;
}
__global__ void sparseMatrixMul(t_FullMatrix pResultVector,t_SparseMatrix pSparseMatrix, t_FullMatrix pVector)
{
	//__shared__ float As[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	__shared__ float Bs[VECTOR_BLOCK_X];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	__shared__ float Cs[VECTOR_BLOCK_Y][VECTOR_BLOCK_X];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	//define gridIndex, if gridDim < mA, gridIndex > 0; 
	int gridIndex = 0;
	int tx,bx;
	//int idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
    t_ve *pMatrixElements, *pVectorElements, *pResultElements;
    unsigned int m, n;//, i, j;
    unsigned int *pRow, *pCol;
    tx = threadIdx.x;
	bx = blockIdx.x;
	//unsigned int colbegin, colend;
    pMatrixElements = pSparseMatrix.pNZElement;
    pVectorElements = pVector.pElement;
    pResultElements = pResultVector.pElement;
    m = pSparseMatrix.m;
    n = pSparseMatrix.n;
	//aBegin,aEnd,aStep are defined for 
	int aBegin = 0;
	int aEnd = pSparseMatrix.m;
	//int aStep = gridDim.x;
	int aStep = gridDim.x*blockDim.y;//gridDim.x
	
	int bBegin = 0;

	int bStep = blockDim.x;
	//int aEnd = mA;
	int bEnd;
	int rowIdx;
    //==check size of Arguments========================================================
    if(m != pResultVector.m*(pResultVector.n)){
        //printf("Result Vector does not match the Matrix\n");
        return;
    }   
    if(n != pVector.m*(pVector.n)){
        //printf("input Vector does not match the Matrix\n");
        return;
    }
	//pRow and pCol may should in share memory or texture
    pRow = pSparseMatrix.pRow;
    pCol = pSparseMatrix.pCol;
    //cal
	
	//for(int a = aBegin; (a < aEnd)&&((gridIndex*gridDim.x+blockIdx.x)<aEnd); a += aStep, gridIndex++){
	for(int a = aBegin; (a < aEnd)&&((gridIndex*aStep+bx*blockDim.y+ty)<aEnd); a += aStep, gridIndex++){
	//for(int a = aBegin; a < aEnd; a += aStep, gridIndex++){

	rowIdx = getSparseRowIdx(gridIndex,aStep,bx,blockDim.y,ty);
		if((tx==0)&&(rowIdx<aEnd)){
			//pResultElements[gridIndex*gridDim.x+blockIdx.x]=0;
			pResultElements[rowIdx]=0;
		}
		__syncthreads();
		
		//following is operations within one block 
		// initialize the dot product for each row in A and vector B
		//t_ve blocksum = 0;
		bBegin = 0;
		bEnd = n;
		t_ve blocksum[VECTOR_BLOCK_Y]; 
		blocksum[????]= 0;
		for(b=bBegin; b<bEnd;b+=bStep){
			if((b+tx)<bEnd)
			Bs[tx] = pVectorElements[b+tx];
			for(i=0;i<VECTOR_BLOCK_Y;i++){
				Cs[tx][i] = 0;			
				for(k=pRow[i];k<pRow[i+1];i++){
					if(b+tx == pCol[k]){
						Cs[tx][i] = Bs[tx]*pMatrixElements[k];
						break;
					}
				}
				
				
			}
			
		}//for b block
		
		
		
		
		
		
		bBegin = pRow[rowIdx];
		bEnd = pRow[rowIdx + 1];
		
		//for(int b = bBegin; (b < bEnd)&&((tx+b) < bEnd); b += bStep ) {
		for(int b = bBegin; b < bEnd; b += bStep ) {

		//initialise Cs 
			//As[threadIdx.x] = 0;		
			//if(ty==0)
			
			Cs[ty][tx] = 0;
			
			// compute scalar product
	
			if ((rowIdx<aEnd)&&((b+tx) < bEnd)) {
				Bs[tx] = pVectorElements[pCol[b + tx ]];
				__syncthreads();
				//Cs[ty][tx] = pMatrixElements[b + tx] * pVectorElements[pCol[b + tx ]];//???
				Cs[ty][tx] = pMatrixElements[b + tx] * Bs[tx];//???
			}
			__syncthreads();
				
			if(tx == 0){
				int kEnd = bEnd-b;
				if(kEnd > blockDim.x)kEnd = blockDim.x;
				//Because I add Cs[0...k], if blockSize and Matrix does not fit, Parts of Cs[k] are not initialized as 0.  		
				//for (int k = 0; k < kEnd; k++) blocksum += Cs[k];
				for (int k = 0; k < kEnd; k++) blocksum[ty] += Cs[ty][k];
				
			
			}
			
			/*
			
						
			int offset; 
			offset = VECTOR_BLOCK_SIZE/2;
			while (offset > 0) {
				if(tx < offset) {
					Cs[tx] += Cs[tx + offset];
				}
				offset >>= 1;
				__syncthreads();
			}
			__syncthreads();
			if(threadIdx.x == 0)
			blocksum += Cs[0]; //??? blocksum = Cs[0];
			
			*/
			/*
			int offset; 
			offset = blockDim.x/2;
			while (offset > 0) {
				if(tx < offset) {
					Cs[ty][tx] += Cs[ty][tx + offset];
				}
				offset >>= 1;
				__syncthreads();
			}
			blocksum[ty] += Cs[ty][0];
			__syncthreads();
			*/
		}//for b
		__syncthreads();

		if(tx == 0) pResultElements[rowIdx] = blocksum[ty];//?????????????
		__syncthreads();	
    
	}//for {int a = aBegin;....
}