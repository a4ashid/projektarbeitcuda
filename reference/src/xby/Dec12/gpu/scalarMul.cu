#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */

/**/
__global__ void device_scalarMul(t_ve* pIn1, t_ve* pIn2,t_ve* pOut, unsigned int N)
{
	//__shared__ float Cs[VECTOR_BLOCK_SIZE];
	
	int tid = threadIdx.x;
	int idx = blockIdx.x*blockDim.x + tid;
	
	//Cs[threadIdx.x] = 0;

	if ( idx < N ) {
	    //Cs[threadIdx.x] = pin1[ idx ] * in2;
		pOut[idx] = pIn1[idx]*pIn2[0];
	}

}