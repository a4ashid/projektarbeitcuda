#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

/* Kernel to square elements of the array on the GPU */
/*
	Matrix A is mA x nB  , Vector B is nB
	Vector C output vector in size of mA
	C=A*B
description:
	each row of A occuppy one block. if gridDim is smaller than the row number of A  
*/
__global__ void sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector)
{
	//__shared__ float As[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 512
	//__shared__ float Bs[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 512
	__shared__ float Cs[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 512
	int idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
    t_ve *pMatrixElements, *pVectorElements, *pResultElements;
    unsigned int m, n, i, j;
    unsigned int *pRow, *pCol;
    unsigned int colbegin, colend;
    pMatrixElements = pSparseMatrix->pNZElement;
    pVectorElements = pVector->pElement;
    pResultElements = pResultVector->pElement;
    m = pSparseMatrix->m;
    n = pSparseMatrix->n;
	int aBegin = 0;
	int bBegin = 0;
	//int aStep = gridDim.x;
	int bStep = VECTOR_BLOCK_SIZE; // blockDim.x
	//int aEnd = mA;
	int bEnd;
    //==check size of Arguments========================================================
    if(m != pResultVector->m*(pResultVector->n)){
        //printf("Result Vector does not match the Matrix\n");
        return;
    }   
    if(n != pVector->m*(pVector->n)){
        //printf("input Vector does not match the Matrix\n");
        return;
    }
	//pRow and pCol may should in share memory or texture
    pRow = pSparseMatrix->pRow;
    pCol = pSparseMatrix->pCol;
    //cal
	

		if(threadIdx.x==0){
			pResultElements[blockIdx.x]=0;
		//C[gridIndex*gridDim.x+blockIdx.x]=0;
		}
		//following is operations within one block 
		// initialize the dot product for each row in A and vector B
		t_ve blocksum = 0;
		//if nB> blockDim, split repeat the
		bBegin = pRow[blockIdx.x];
		bEnd = pRow[blockIdx.x + 1];
		for(int b = bBegin; (b < bEnd)&&((threadIdx.x+b) < bEnd); b += bStep ) {
			//initialise Cs 
			//As[threadIdx.x] = 0;
			//Bs[threadIdx.x] = 0;// consider text memory
			Cs[threadIdx.x] = 0;
			__syncthreads();
			// compute scalar product
			// for (i = 0; i < m; i++){
			// colbegin = pRow[i];
			// colend = pRow[i+1];
			// for(j=colbegin;j<colend;j++)pResultElements[i] += pMatrixElements[j]*pVectorElements[pCol[j]];
			// } 
			if (( (gridIndex*gridDim.x+blockIdx.x)<aEnd)&&((b+threadIdx.x) < bEnd)) {
				
				Cs[threadIdx.x] = pMatrixElements[b + threadIdx.x] * pVectorElements[pCol[b + threadIdx.x ]];
			}
			__syncthreads();
				
			if(threadIdx.x == 0){
				//30.Nov.2009 fixeded for Cs summe
				int kEnd = bEnd-b;
				if(kEnd > VECTOR_BLOCK_SIZE)kEnd = VECTOR_BLOCK_SIZE;
				//Because I add Cs[0...k], if blockSize and Matrix does not fit, Parts of Cs[k] are not initialized as 0.  		
				for (int k = 0; k < kEnd; k++) blocksum += Cs[k];
			
			}
			__syncthreads();
			
			//Cs[threadIdx.x] = 0;
			//__syncthreads();	
		}//for b
		__syncthreads();

		if(threadIdx.x == 0) pResultElements[blockIdx.x] = blocksum;
		__syncthreads();	
    
 
}