#include "hip/hip_runtime.h"
/* matlab interface for 
__host__ void idrs(
                     t_ve* A_h,
                     t_ve* b_h,
                     unsigned int s,
                     t_ve  tol,
                     unsigned int maxit,
                     t_ve* x0_h,

                     unsigned int N,

                     t_ve* x_h,  //output vector 
                     t_ve* resvec_h,
                     unsigned int* piter
                  )
				  
				  */


#include <math.h> 
#include "mex.h"

/* Gateway function */
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{
	int inputIdx;
	A_h;????
	t_ve *b_h,tol, *x0_h, *x_h, *resvec_h;
	unsigned int s, maxit, N, *piter;
	//=======read input===============================================================
	//read spaser Matrix A_h
	inputIdx = 0;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
    ir = mxGetIr(prhs[inputIdx]);
    jc = mxGetJc(prhs[inputIdx]);
    nzmax = mxGetNzmax(prhs[inputIdx]);
	A_h[i] = (t_ve)pr[i];
	=ir[i];
	=jc[i];
	//read b_h
	inputIdx = 1;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	b_h = (t_ve*)mxMalloc(sizeof(t_ve)*m*n);
	for()
	
	//read s
	inputIdx = 2;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	s = (unsigned int)pr[0];
	
	//read tol
	inputIdx = 3;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	tol = (unsigned int)pr[0];
	 
	//read maxit
	inputIdx = 4;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	maxit = (unsigned int)pr[0];
	 
	//read N
	inputIdx = 5;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	N = (unsigned int)pr[0];
	
	//
	x_h = (t_ve*)mxMalloc(sizeof(t_ve)*m*n)?????
	resvec_h = (t_ve*)mxMalloc(sizeof(t_ve)*m*n) ???
	piter = (unsigned int *)mxMalloc(sizeof(unsigned int)*m*n)????
	//
	idrs(A_h,b_h,s,tol,maxit,N,x_h,resvec_h,piter);
	//=======================================================================
	//output x_h
    plhs[0] = mxCreateDoubleMatrix(......,1,mxREAL);
	pr = mxGetPr(plhs[0]);
	for(i = 0; i < max...; i++){
		pr[i] = (double)x_h;
	}
	//output resvec_h
    plhs[1] = mxCreateDoubleMatrix(......,1,mxREAL);
	pr = mxGetPr(plhs[1]);
	for(i = 0; i < max...; i++){
		pr[i] = (double)resvec_h;
	}
	//output iter
	plhs[2] = mxCreateNumericMatrix(......,1,mxUINT32_CLASS,mxREAL);
	pr = mxGetPr(plhs[1]);
	for(i = 0; i < max...; i++){
		pr[i] = piter[i];
	}
	
	
	//=======================================================================
	
	
	
}