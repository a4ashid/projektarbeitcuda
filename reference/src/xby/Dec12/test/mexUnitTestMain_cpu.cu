#include "hip/hip_runtime.h"
/*
mexUnitTestMain.cu
*/
#include <stdio.h>
#include "mex.h"

#include "..\cpu\dotMul_cpu.c"
#include "..\cpu\norm_cpu.c"
#include "..\cpu\matrixMul_cpu.c"


void callTestFunction(double** ppIn,int *pmIn,int *pnIn, int callFuncType,double** ppOut){
//call cpu
	//printf("in callTestFunction \n");

	switch(callFuncType){
		case 0://dotMul
			if((pmIn[0]==pmIn[1])&&(pnIn[0]==pnIn[1])){
				if(1==pnIn[0])test_dotMul_cpu(ppIn[0],ppIn[1],ppOut[0],pmIn[0]);
			}
			break;
		case 1://norm	
				if(1==pnIn[0])test_norm_cpu(ppIn[0],ppOut[0],pmIn[0]);
		break;
		case 2: //matrixMul
			//test matrixMul A*B = C
			//ppIn[0]:matrix A, ppIn[1]: vector B, ppOut[0]: result verctor C
			//pmIn[0]=mA pmIn[1]=nB, pnIn[0]=nB,pnIn[1]=1
			if((pmIn[1]==pnIn[0])&&(1==pnIn[1])){
				test_matrixMul_cpu(ppOut[0],ppIn[0],ppIn[1],pmIn[0], pmIn[1]);
			}
		default:  
			if((pmIn[0]==pmIn[1])&&(pnIn[0]==pnIn[1])){
				if(1==pnIn[0])test_dotMul_cpu(ppIn[0],ppIn[1],ppOut[0],pmIn[0]);
			}
	}

}

/* Gateway function */
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{
     int outputArgNum;
     int i,j,k,m, n, retNum, outNum;
     double *pMatrix;
     double * pIn, *pOut;  
     double ** ppIn, **ppOut;
     int *pmIn;
     int *pnIn;	 
	 outputArgNum = nlhs;
	 
	 if((nrhs > 0)){
		pMatrix = mxGetPr(prhs[nrhs-1]);
		retNum = (int)pMatrix[0];
		switch (retNum){
			case 0://doutMul
				outNum = 1;
			break;
			case 1://norm
				outNum = 1;
			break;
			case 2://matrixMul
				//get mA from matrix A
				outNum = mxGetM(prhs[0]);;
			break;
			default://0
				outNum = 1;
		}
		//error checking
		//if((outputArgNum!=outNum)&&(outputArgNum!=0)){
        //    printf("outputArgNum =%d,  outNum =%d \n",outputArgNum,outNum);
		//	mexErrMsgTxt("the number of output arguments does not metch the function! ");
		//}
		if(outNum < 1) {
			printf("please define function Type! \ n");
			printf("0:dotMul,1:norm, 2: matrixMul \ n");
			return;
		}
	
	///////////////
		pnIn = (int*)mxMalloc(sizeof(int)*nrhs);
		pmIn = (int*)mxMalloc(sizeof(int)*nrhs);
		ppIn = (double**)mxMalloc(sizeof(double*)*nrhs);
		ppOut = (double**)mxMalloc(sizeof(double*)*1);
		for (i = 0; i < nrhs; i++){
        /* Find the dimensions of the data */
			m = mxGetM(prhs[i]);
			n = mxGetN(prhs[i]);
			pmIn[i] = (int)m;
			pnIn[i] = (int)n;
			pMatrix = mxGetPr(prhs[i]); 
			pIn = (double*)mxMalloc(sizeof(double)*m*n);       
			ppIn[i]=pIn;
			for( k = 0; k < m; k++)
				for( j = 0; j < n; j++){
					pIn[k*n+j] = (double)pMatrix[j*m+k];
				}//for j k
       
		}// for i
	 ///////////////////
		plhs[0] = mxCreateDoubleMatrix(outNum,1,mxREAL);
		pOut = mxGetPr(plhs[0]);
		
		ppOut[0] = pOut;
	 }//if nrhs>0
	 
	 callTestFunction(ppIn,pmIn,pnIn, retNum, ppOut);
     for (i = 0; i < nrhs; i++){
        pIn=ppIn[i];
        mxFree(pIn);
     }
     
     mxFree(pnIn);
     mxFree(pmIn);
     mxFree(ppIn);
	 mxFree(ppOut);
}