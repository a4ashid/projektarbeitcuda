#include "hip/hip_runtime.h"
#include <math.h> /* Needed for the ceil() prototype */
#include "mex.h"
#include "host_sparseMatrixMul01.cu"
#if defined(NAN_EQUALS_ZERO)
#define IsNonZero(d) ((d)!=0.0 || mxIsNaN(d))
#else
#define IsNonZero(d) ((d)!=0.0)
#endif
//typedef NULL 0;
typedef float t_ve;
typedef int mwIndex;


void initElement(t_FullMatrix * pMatrix)
{
    int i;
    if (pMatrix != 0)
        for (i = 0; i < (pMatrix->m)*(pMatrix->n); i++){
            pMatrix->pElement[i] = 0;  
        }
}
void setElement(t_FullMatrix * pMatrix, unsigned int row, unsigned int col, float val)
{
    if((row < pMatrix->m)&&(col < pMatrix->n)){
        pMatrix->pElement[(row)*(pMatrix->n) + col ] = val;
    }
}
void calMV(t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector,t_FullMatrix * pResultVector)
{
    t_ve *pMatrixElements, *pVectorElements, *pResultElements;
    unsigned int m, n, i, j;
    unsigned int *pRow, *pCol;
    int colbegin, colend;
    pMatrixElements = pSparseMatrix->pNZElement;
    pVectorElements = pVector->pElement;
    pResultElements = pResultVector->pElement;
    m = pSparseMatrix->m;
    n = pSparseMatrix->n;
    //==check size of Arguments========================================================
    if(m != pResultVector->m*(pResultVector->n)){
        printf("Result Vector does not match the Matrix\n");
        return;
    }   
    if(n != pVector->m*(pVector->n)){
        printf("input Vector does not match the Matrix\n");
        return;
    }
    pRow = pSparseMatrix->pRow;
    pCol = pSparseMatrix->pCol;
    //cal
    printf("in calMV \n");
    for (i = 0; i < m; i++){
        colbegin = pRow[i];
        colend = pRow[i+1];
        for(j=colbegin;j<colend;j++)pResultElements[i] += pMatrixElements[j]*pVectorElements[pCol[j]];
    }  
}
/* Gateway function */
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{
    /* Declare variable */
    unsigned int m,n;
    unsigned int nzmax;
    int *ir,*jc,i;
    //unsigned int *irs,*jcs;
    //int cmplx,isfull;
    double *pr,*pi;//,*si,*sr;
    //double percent_sparse;
    //double *pMatrix;
    t_SparseMatrix sparseMatrix;
    t_SparseMatrix *pSparseMatrix ;     
    //declare Vector 
    t_FullMatrix fullVector, ResultVector;
    t_FullMatrix * pVector, *pResultVector;
    
	hipError_t e;	
	float t_avg;
	t_avg = 0;
	
    pSparseMatrix = &sparseMatrix; 
    pVector = &fullVector;
    pResultVector = &ResultVector;
    /* Check for proper number of input and output arguments */    
    if (nrhs < 2) {
	mexErrMsgTxt("Two input argument required. First Sparse Matrix, Second Vector");
    } 
    if(nlhs > 1){
	mexErrMsgTxt("Too many output arguments.");
    }
     
    if (mxGetNumberOfDimensions(prhs[0]) != 2){
	mexErrMsgTxt("Input argument must be two dimensional\n");
    } 
     
     
    /* Get the size and pointers to input data */
    //prepare MV caculation
    //====get SparseMatrix============================================
    m  = mxGetM(prhs[0]);
    n  = mxGetN(prhs[0]);
    pr = mxGetPr(prhs[0]);
    pi = mxGetPi(prhs[0]);
    ir = mxGetIr(prhs[0]);
    jc = mxGetJc(prhs[0]);
    nzmax = mxGetNzmax(prhs[0]);
    //in c exchange m and n
    pSparseMatrix->m = n;
    pSparseMatrix->n = m;
    pSparseMatrix->nzmax = nzmax;

	
	//=========================================================================================
	

    pSparseMatrix->pNZElement = (t_ve *)mxMalloc(sizeof(t_ve)*nzmax);
    pSparseMatrix->pCol = (unsigned int*) mxMalloc(sizeof(unsigned int)*nzmax); 
    pSparseMatrix->pRow = (unsigned int*) mxMalloc(sizeof(unsigned int)*(pSparseMatrix->m+1)); 
    
	for(i = 0; i < nzmax; i++){
        pSparseMatrix->pNZElement[i] =(t_ve) pr[i];
        pSparseMatrix->pCol[i] = ir[i];
    }
    for(i = 0; i < n+1; i++){  
        pSparseMatrix->pRow[i] = jc[i];
    }
    
    //=====get Vector==========================================================
 //   t_Matrix fullVector;
 //   t_Matrix * pVector;

    m  = mxGetM(prhs[1]);
    n  = mxGetN(prhs[1]);
    pr = mxGetPr(prhs[1]);
    pi = mxGetPi(prhs[1]);
    pVector->m=m;
    pVector->n=n;
    if (!((m == 1)||(n==1))){
        mexErrMsgTxt("Second Argument must be Vector! \n");
    } 
    pVector->pElement = (t_ve*)mxMalloc(sizeof(t_ve)*m*n);
    for(i = 0; i < m*n; i++) pVector->pElement[i] = pr[i];
    //====create Result Vector==================================================================
    pResultVector->m = pSparseMatrix->m; 
    pResultVector->n = 1;
    pResultVector->pElement = (t_ve*)mxMalloc(sizeof(t_ve)*m*n);
    initElement(pResultVector);
    //======================================================================================
    START_CUDA_TIMER;
	// call cpu
    calMV(pSparseMatrix, pVector, pResultVector);
    
	//call gpu
	//host_sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector)
	//printf("call host \n");
	//host_sparseMatrixMul(pResultVector, pSparseMatrix, pVector);
    //printf("after host \n");
	STOP_CUDA_TIMER( &t_avg);
	printf("CPU runing time =%lf (ms) \n",t_avg);
	
	
	//ir = mxGetIr(prhs[0]);
    //jc = mxGetJc(prhs[0]);
    //nzmax = mxGetNzmax(prhs[0]);
    plhs[0] = mxCreateDoubleMatrix(pResultVector->m*pResultVector->n,1,mxREAL);
	pr = mxGetPr(plhs[0]);
    //for (i = 0; i<(pResultVector->m*pResultVector->n); i++)pr[i] = pResultVector->pElement[i];
    
    /* Allocate space for sparse matrix 
     * NOTE:  Assume at most 20% of the data is sparse.  Use ceil
     * to cause it to round up. 
     *Nonzero elements
     *if(jc[i]!=jc[i+1]) for(int k = jc[i]; k<jc[i+1]; k++)A[ir[k][i]=pr[k]+pi[k]
     */

       // printf("printf Result Vector \n");
       for(i=0; i < pResultVector->n*pResultVector->m; i++){  
			//copy result back to matlab;
			pr[i] = (double)pResultVector->pElement[i];
        }

        mxFree(pResultVector->pElement);
        mxFree(pVector->pElement);
        mxFree( pSparseMatrix->pNZElement);
        mxFree( pSparseMatrix->pCol);
        mxFree( pSparseMatrix->pRow);
       
   
}
