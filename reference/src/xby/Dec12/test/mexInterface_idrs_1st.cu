/* matlab interface for 
extern "C" void idrs_1st(

                     t_SparseMatrix A_in,    // A Matrix in buyu-sparse-format //0
                     t_ve*          b_in,    // b as in A * b = x //1
                     t_ve*          xe_in, //2

                     t_mindex N,	//3

                     t_ve*          r_out,    // the r from idrs.m line 6 : r = b - A*x; 

                     t_idrshandle*  ih_out  // handle for haloding all the device pointers between matlab calls 

           );
				  
				  */


#include <math.h> 
#include "mex.h"

typedef unsigned int t_mindex;
typedef float t_ve;

typedef struct SparseMatrix{
    t_mindex m;
    t_mindex n;
    t_mindex nzmax;
	//size m+1
    t_mindex *pRow;
    //size nzmax
	t_mindex *pCol;
	//size : nzmax
    t_ve* pNZElement;
} t_SparseMatrix;

int smat_size( int count_nzmax, int cunt_rows ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * count_nzmax
           + sizeof(t_mindex)  * (cunt_rows + 1);
}

// ---------------------------------------------------------------------
 void set_sparse_data( t_SparseMatrix* m, void* mv ) {

   m->pCol = (t_mindex *) mv;
   m->pNZElement = (t_ve *) (&m->pCol[m->nzmax] ) ;
   m->pRow = (t_mindex *) (&m->pNZElement[m->nzmax]);

}
/* Gateway function */
//[x,resvec,iter]=mexInterface_idrs(A,b,s,tol,maxit,x0);
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{
	int inputIdx;
	t_SparseMatrix A_in;
	t_ve *b_in,tol, *xe_in, *r_out;
	t_idrshandle*  ih_out;
	unsigned int s, N;
    unsigned int m,n;
    unsigned int nzmax;
    int *ir,*jc,i;
    //unsigned int *irs,*jcs;
    //int cmplx,isfull;
    double *pr,*pi;//,*si,*sr;
    int size_resvec;
	int msize ;
	void *devicemem;
    //=======read input===============================================================
	//read spaser Matrix A_in
	printf("read spaser Matrix A!\n");
	inputIdx = 0;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
    ir = mxGetIr(prhs[inputIdx]);
    jc = mxGetJc(prhs[inputIdx]);
    nzmax = mxGetNzmax(prhs[inputIdx]);
	
	// inupt matrix rotate
	A_in.m = n;
	A_in.n = m;
	A_in.nzmax = nzmax;
	msize = smat_size( A_in.nzmax, A_in.n );
	
	devicemem = mxMalloc ( msize );
	set_sparse_data(&A_in, devicemem);
	printf("ir0 = %d, pr0 = %lf\n",ir[0],pr[0]);
	for(i = 0; i < nzmax; i++){
        A_in.pNZElement[i] =(t_ve)pr[i];
        A_in.pCol[i] = ir[i];
		printf("ir = %d, pr = %lf\n",ir[i],pr[i]);
    }
    for(i = 0; i < n+1; i++){  
        A_in.pRow[i] = jc[i];
		
    }

	//read b_in size of N = m*n
	printf("read Vector b!\n");
	inputIdx = 1;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	b_in = (t_ve*)mxMalloc(sizeof(t_ve)*m*n);
	for(i = 0; i < m*n; i++){
		b_in[i] = (t_ve)pr[i];
	}
	
	
	//read xe_in size of N = m*n
	inputIdx = 2;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
    xe_in = (t_ve*)mxMalloc(sizeof(t_ve)*m*n);
	for(i = 0; i < m*n; i++){
		xe_in[i] = (t_ve)pr[i];
	}
	//read N
	inputIdx = 3;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	N = (unsigned int)pr[0];
	
	 printf("create output!\n");

	
	//create output vector r_out of size N
	r_out = (t_ve*)mxMalloc(sizeof(t_ve)*N);
	//output ih_out ???????
	ih_out = (t_idrshandle *)mxMalloc(sizeof(t_idrshandle)*1);
	//=======================================================================
	//call idrs interface 
	//idrs_1st(A_in, b_in, xe_in, N, r_out, ih_out);
	//=======================================================================
	//output r_out, resvec,piter in matlab
    plhs[0] = mxCreateDoubleMatrix(N,1,mxREAL);
	pr = mxGetPr(plhs[0]);
	for(i = 0; i < N; i++){
		pr[i] = (double)r_out[i];
	}
	//output t_idrshandle
	
   //????????
	//output iter of scalar value

	//=======================================================================
	mxFree(b_in); 
	mxFree(xe_in); 
	mxFree(r_out);
	mxFree(t_idrshandle);??????
	mxFree(devicemem);???????
	
	
}