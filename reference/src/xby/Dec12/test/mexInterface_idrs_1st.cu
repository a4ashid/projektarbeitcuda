/* matlab interface for 
extern "C" void idrs_1st(

                     t_SparseMatrix A_in,    // A Matrix in buyu-sparse-format //0
                     t_ve*          b_in,    // b as in A * b = x //1
                     t_ve*          xe_in, //2

                     t_mindex N,	//3

                     t_ve*          r_out,    // the r from idrs.m line 6 : r = b - A*x; 

                     t_idrshandle*  ih_out  // handle for haloding all the device pointers between matlab calls 

           );
				  
				  */


#include <math.h> 
#include "mex.h"
#include "..\\gpu\\projektcuda.h"

int smat_size( int count_nzmax, int cunt_rows ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * count_nzmax
           + sizeof(t_mindex)  * (cunt_rows + 1);
}

// ---------------------------------------------------------------------
 void set_sparse_data( t_SparseMatrix* m, void* mv ) {

   m->pCol = (t_mindex *) mv;
   m->pNZElement = (t_ve *) (&m->pCol[m->nzmax] ) ;
   m->pRow = (t_mindex *) (&m->pNZElement[m->nzmax]);

}
/* Gateway function */
//[x,resvec,iter]=mexInterface_idrs(A,b,s,tol,maxit,x0);
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{
	int inputIdx,outputIdx;
	t_SparseMatrix A_in;
	t_ve *b_in,tol, *xe_in, *r_out;
	t_idrshandle*  ih_out;
	unsigned int s, N;
    unsigned int m,n;
    unsigned int nzmax;
    int *ir,*jc,i;
    //unsigned int *irs,*jcs;
    //int cmplx,isfull;
    double *pr,*pi;//,*si,*sr;
    int size_resvec;
	int msize ;
	void *devicemem;
	
	if( nrhs < 4 ) {
		printf("not enough input argument!\n");
		printf("[r_out,ih_out]=mexInterface_idrs_1st(A_in, b_in, xe_in, N);\n");
		return;
	}
	
    //=======read input===============================================================
	//read spaser Matrix A_in
	printf("read spaser Matrix A!\n");
	inputIdx = 0;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
    ir = mxGetIr(prhs[inputIdx]);
    jc = mxGetJc(prhs[inputIdx]);
    nzmax = mxGetNzmax(prhs[inputIdx]);
	
	// inupt matrix rotate
	A_in.m = n;
	A_in.n = m;
	A_in.nzmax = nzmax;
	msize = smat_size( A_in.nzmax, A_in.n );
	
	devicemem = mxMalloc ( msize );
	set_sparse_data(&A_in, devicemem);
	printf("ir0 = %d, pr0 = %lf\n",ir[0],pr[0]);
	for(i = 0; i < nzmax; i++){
        A_in.pNZElement[i] =(t_ve)pr[i];
        A_in.pCol[i] = ir[i];
		printf("ir = %d, pr = %lf\n",ir[i],pr[i]);
    }
    for(i = 0; i < n+1; i++){  
        A_in.pRow[i] = jc[i];
		
    }

	//read b_in size of N = m*n
	printf("read Vector b!\n");
	inputIdx = 1;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	b_in = (t_ve*)mxMalloc(sizeof(t_ve)*m*n);
	for(i = 0; i < m*n; i++){
		b_in[i] = (t_ve)pr[i];
	}
	
	
	//read xe_in size of N = m*n
	inputIdx = 2;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
    xe_in = (t_ve*)mxMalloc(sizeof(t_ve)*m*n);
	for(i = 0; i < m*n; i++){
		xe_in[i] = (t_ve)pr[i];
	}
	//read N
	inputIdx = 3;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	N = (unsigned int)pr[0];
	
	 printf("create output!\n");

	
	//create output vector r_out of size N
	r_out = (t_ve*)mxMalloc(sizeof(t_ve)*N);
	//output ih_out ???????
	ih_out = (t_idrshandle *)mxMalloc(sizeof(t_idrshandle)*1);
	//=======================================================================
	//call idrs interface 
	//idrs_1st(A_in, b_in, xe_in, N, r_out, ih_out);
	/*
	extern "C" void idrs_1st(
                     t_SparseMatrix A_in,   //A Matrix in buyu-sparse-format 
                     t_ve*          b_in,   // b as in A * b = x 
                     t_ve*          xe_in,

                     t_mindex N,

                     t_ve*          r_out,    // the r from idrs.m line 6 : r = b - A*x; 

                     t_idrshandle*  ih_out  // handle for haloding all the device pointers between matlab calls 
           );
	*/
	//=======================================================================
	//output r_out,t_idrshandle in matlab
	outputIdx = 0; //r_out
    plhs[outputIdx] = mxCreateDoubleMatrix(N,1,mxREAL);
	pr = mxGetPr(plhs[outputIdx]);
	for(i = 0; i < N; i++){
		//r_out[i] = 1;
		pr[i] = (double)r_out[i];
	}
	//output t_idrshandle
	outputIdx = 1; //t_idrshandle
	//plhs[outputIdx] = mxCreateNumericMatrix(1,1,mxINT32_CLASS,mxREAL);
	plhs[outputIdx] = mxCreateDoubleMatrix(1,1,mxREAL);;
	pr = mxGetPr(plhs[outputIdx]);
	
	for(i = 0; i < 1; i++){
		printf("output ihout");
		pr[i] = (int)ih_out[0];
	}
	//=======================================================================
	mxFree(devicemem);
	mxFree(b_in); 
	mxFree(xe_in); 
	mxFree(r_out);
	mxFree(ih_out);
	
	
	
}