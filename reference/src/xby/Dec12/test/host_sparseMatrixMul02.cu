//host_SparseMV
#include "test_comm.h"

// ---------------------------------------------------------------------
__host__ int smat_size( int count_nzmax, int cunt_rows ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * count_nzmax
           + sizeof(t_mindex)  * (cunt_rows + 1);
}

// ---------------------------------------------------------------------
__host__ void set_sparse_data( t_SparseMatrix* m, void* mv ) {

   m->pCol = (t_mindex *) mv;
   m->pNZElement = (t_ve *) (&m->pCol[m->nzmax] ) ;
   m->pRow = (t_mindex *) (&m->pNZElement[m->nzmax]);

}
void host_sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector){

	t_SparseMatrix host_SparseMatrix,dev_SparseMatrix;
	//t_SparseMatrix data_in1,*data_in1_host,*data_in1_gpu;//input sparse Matrix
	t_FullMatrix host_Vector,dev_Vector,host_ResultVector,dev_ResultVector;
	//t_FullMatrix data_in2,*data_in2_host,*data_in2_gpu;//input vector
	//t_FullMatrix data_out,*data_out_host,*data_out_gpu;//output vector
	size_t size_NZElement,size_Row,size_Col;
	int sizeBlock,i;
	//data_in1_host=&data_in1;
	//data_in2_host=&data_in2;
	//data_out_host=&data_out;
	
	sizeBlock = VECTOR_BLOCK_SIZE;
	//=====debug==================
	printf("=======in host========== \n");
	printf("pSparseMatrix->m=%d \n",pSparseMatrix->m);
	printf("pSparseMatrix->n=%d \n",pSparseMatrix->n);
	//============================
	/*
	host_SparseMatrix.m = pSparseMatrix->m;
	host_SparseMatrix.n = pSparseMatrix->n;
	host_SparseMatrix.nzmax = pSparseMatrix->nzmax;
	host_SparseMatrix.pCol = pSparseMatrix->pCol;
	host_SparseMatrix.pNZElement = pSparseMatrix->pNZElement;
	host_SparseMatrix.pRow = pSparseMatrix->pRow;

	host_Vector.m = pVector->m;
	host_Vector.n = pVector->n;
	host_Vector.pElement = pVector->pElement;	
	
	host_ResultVector.m = pResultVector->m;
	host_Vector.n = pResultVector->n;
	host_ResultVector.pElement = pResultVector->pElement;
	*/
	//size_NZElement = sizeof(t_ve)*pSparseMatrix->nzmax;
	//size_Row =sizeof(t_mindex)*(pSparseMatrix->m+1); //sizeof(int)*pSparseMatrix->m;
	//size_Col = sizeof(t_mindex)*pSparseMatrix->nzmax;//sizeof(int)*(pSparseMatrix->n+1);
	// Create an input and output data array on the GPU
	//malloc memory for Input Sparse-Matrix
	printf("malloc sparse Matrix \n");
	dev_SparseMatrix.m = pSparseMatrix->m;
	dev_SparseMatrix.n = pSparseMatrix->n;
	dev_SparseMatrix.nzmax = pSparseMatrix->nzmax;
	int msize = smat_size( dev_SparseMatrix.nzmax, dev_SparseMatrix.n );
    printf(" got result %u \n", msize);
	void *devicemem;
    hipError_t e;
	e = hipMalloc ( &devicemem, msize );
	CUDA_UTIL_ERRORCHECK("hipMalloc")
	//pSparseMatrix->pCol is the begin of memery block
	e = hipMemcpy(  devicemem, pSparseMatrix->pCol, msize , hipMemcpyHostToDevice);
   CUDA_UTIL_ERRORCHECK("hipMemcpy")
   set_sparse_data( &dev_SparseMatrix, devicemem);
	//malloc device memory for Input vector
	printf("malloc vector \n");
	size_t size_VElement, size_RElement;
	size_VElement = sizeof(t_ve)*pVector->m*pVector->n;
	size_RElement = sizeof(t_ve)*pSparseMatrix->m;
	hipMalloc( (void **) &(dev_Vector.pElement),size_VElement);
	dev_Vector.m = pVector->m;//host_Vector.m;
	dev_Vector.n = pVector->n;//host_Vector.n;
	hipMemcpy(dev_Vector.pElement,pVector->pElement,size_VElement,hipMemcpyHostToDevice);
	
	printf("malloc output \n");
	//malloc output Vector
	dev_ResultVector.m = pSparseMatrix->m;
	dev_ResultVector.n = 1;
	hipMalloc( (void **) &(dev_ResultVector.pElement),size_RElement);

	// Compute execution configuration using 128 threads per block 
	dim3 dimBlock(sizeBlock);
	//dim3 dimGrid((sizeIn)/dimBlock.x);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp,0);
	printf("number of multiProcessors: %d \n",deviceProp.multiProcessorCount);
	int sizeGrid = 65535;
	if (sizeGrid > pSparseMatrix->m)sizeGrid = pSparseMatrix->m;
	printf("grid size = %d\n",sizeGrid);
	dim3 dimGrid(sizeGrid);
	//if ( (sizeA) % sizeBlock !=0 ) dimGrid.x+=1;
	//sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector)
	//sparseMatrixMul<<<dimGrid,dimBlock>>>(data_out_gpu,data_in1_gpu,data_in2_gpu);
	printf("calling kernel \n");
	sparseMatrixMul<<<dimGrid,dimBlock>>>(dev_ResultVector,dev_SparseMatrix,dev_Vector);
	hipError_t e;	
	e = hipGetLastError();
	if ( e != hipSuccess)
	{
			fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
			exit(-1);
	}
	
	printf("get Result \n");
	//hipMemcpy( data_out_host->pElement,pResultVector->pElement,  size_RElement, hipMemcpyDeviceToHost);
	hipMemcpy( pResultVector->pElement,dev_ResultVector.pElement,  size_RElement, hipMemcpyDeviceToHost);
	
	pResultVector->m = pSparseMatrix->m;
	pResultVector->n = 1;
	//=========debug==============
		printf("==================Result in host============\n");
		for( i = 0; i < pResultVector->m; i++) printf("pResultVector->pElement[%d]=%f \n",i,pResultVector->pElement[i]);
	//=======================
	
	printf("free host \n");
	hipFree(devicemem);
	hipFree(dev_Vector.pElement);
	hipFree(dev_ResultVector.pElement);

}


