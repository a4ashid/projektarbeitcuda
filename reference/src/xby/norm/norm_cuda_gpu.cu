#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */
__global__ void square_elements(float* in, float* out, int N)
{
int idx = blockIdx.x*blockDim.x+threadIdx.x;
//if ( idx < N) out[idx]=in[idx]*in[idx];
if ( idx < N)out[0]=out[0]+in[idx]*in[idx];
//if ( idx < N) *out=*out+in[idx]*in[idx];
__syncthreads();
}
void square_host(double* pIn, double *pOut, int sizeIn, int sizeOut)
{

int i, j;
double *data1, *data2;
float *data1f, *data2f;
float *data1f_gpu, *data2f_gpu;
int sizeBlock;
sizeBlock = 16;
data1 = pIn;


/* Find the dimensions of the data */

/* Create an mxArray for the output data */

/* Create an input and output data array on the GPU*/
hipMalloc( (void **) &data1f_gpu,sizeof(float)*sizeIn);
hipMalloc( (void **) &data2f_gpu,sizeof(float)*sizeOut);
/* Retrieve the input data */

/* Check if the input array is single or double precision */

/* The input array is in double precision, it needs to be converted t
floats before being sent to the card */
data1f = (float *) malloc(sizeof(float)*sizeIn);
for (j = 0; j < sizeIn; j++)
{
data1f[j] = (float) data1[j];
}
    for (i = 0; i < sizeIn; i++)
    {
        printf("data1f[%d] = %f, ", i, data1f[i]);
    }
        printf("\n");

hipMemcpy( data1f_gpu, data1f, sizeof(float)*sizeIn, hipMemcpyHostToDevice);

data2f = (float *) malloc(sizeof(float)*sizeOut);
//cudaMemcpy( data2f_gpu, data2f, sizeof(float)*sizeOut, cudaMemcpyHostToDevice);

/* Compute execution configuration using 128 threads per block */
dim3 dimBlock(sizeBlock);
dim3 dimGrid((sizeIn)/dimBlock.x);
if ( (sizeIn) % sizeBlock !=0 ) dimGrid.x+=1;
    
/* Call function on GPU */
square_elements<<<dimGrid,dimBlock>>>(data1f_gpu, data2f_gpu, sizeIn);
/* Copy result back to host */
hipMemcpy( data2f, data2f_gpu, sizeof(float)*sizeOut, hipMemcpyDeviceToHost);
    for (i = 0; i < sizeOut; i++)
    {
        printf("data2f[%d] = %f, ", i, data2f[i]);
    }
        printf("\n");


/* Create a pointer to the output data */
data2 = pOut;
/* Convert from single to double before returning */
for (j = 0; j < sizeOut; j++)
{
data2[j] = (double) data2f[j];
}
/* Clean-up memory on device and host */
free(data1f);
free(data2f);
hipFree(data1f_gpu);
hipFree(data2f_gpu);
}

int main()
{

    double *pIn, *pOut;
    int sizeIn, sizeOut;
    int i;
    sizeIn = 3;
    sizeOut = 3;
    pIn = (double*)malloc(sizeof(double)*sizeIn);
    pOut = (double*)malloc(sizeof(double)*sizeOut);
    pIn[0] = 1;
    pIn[1] = 2;
    pIn[2] = 3;
    square_host(pIn, pOut, sizeIn, sizeOut);
    for (i = 0; i < sizeOut; i++)
    {
        printf("pOut[%d] = %lf, ", i, pOut[i]);
    }
        printf("\n");
    

    free(pIn);
    free(pOut);

    return 0;
}




/* Gateway function */
/*
void mexFunction(int nlhs, mxArray *plhs[],
int nrhs, const mxArray *prhs[])
{
int i, j, m, n;
double *data1, *data2;
float *data1f, *data2f;
float *data1f_gpu, *data2f_gpu;
mxClassID category;
if (nrhs != nlhs)
mexErrMsgTxt("The number of input and output arguments must be the same.");

//Create an mxArray for the output data by buyu 
//plhs[0] = mxCreateDoubleMatrix(1, 1, mxREAL);
//create a float host output variable for float
//data2f = (float *) mxMalloc(sizeof(float));

for (i = 0; i < nrhs; i++)
{
// Find the dimensions of the data 
m = mxGetM(prhs[i]);
n = mxGetN(prhs[i]);

//Create an mxArray for the output data 
//plhs[i] = mxCreateDoubleMatrix(m, n, mxREAL);
  plhs[i] = mxCreateDoubleMatrix(1, 1, mxREAL);


// Create an input and output data array on the GPU
cudaMalloc( (void **) &data1f_gpu,sizeof(float)*m*n);
//cudaMalloc( (void **) &data2f_gpu,sizeof(float)*m*n);
cudaMalloc( (void **) &data2f_gpu,sizeof(float));

// Retrieve the input data 
data1 = mxGetPr(prhs[i]);
// Check if the input array is single or double precision 
category = mxGetClassID(prhs[i]);
if( category == mxSINGLE_CLASS)
{
// The input array is single precision, it can be sent directly to the card
cudaMemcpy( data1f_gpu, data1, sizeof(float)*m*n,
cudaMemcpyHostToDevice);
}
if( category == mxDOUBLE_CLASS)
{
// The input array is in double precision, it needs to be converted t floats before being sent to the card 
data1f = (float *) mxMalloc(sizeof(float)*m*n);
for (j = 0; j < m*n; j++)
{
data1f[j] = (float) data1[j];
}
printf("before copyHost to device \n");
cudaMemcpy( data1f_gpu, data1f, sizeof(float)*n*m, cudaMemcpyHostToDevice);
}

//orginal output
//data2f = (float *) mxMalloc(sizeof(float)*m*n);
data2f = (float *) mxMalloc(sizeof(float));

// Compute execution configuration using 128 threads per block 
dim3 dimBlock(128);
dim3 dimGrid((m*n)/dimBlock.x);
if ( (n*m) % 128 !=0 ) dimGrid.x+=1;
    
printf("before calling GPU \n");
// Call function on GPU 
square_elements<<<dimGrid,dimBlock>>>(data1f_gpu, data2f_gpu, n*m);

printf("before copy result back \n");
// Copy result back to host 
//cudaMemcpy( data2f, data2f_gpu, sizeof(float)*n*m, cudaMemcpyDeviceToHost);
cudaMemcpy( data2f, data2f_gpu, sizeof(float), cudaMemcpyDeviceToHost);
// Create a pointer to the output data 
data2 = mxGetPr(plhs[i]);
// Convert from single to double before returning 

//for (j = 0; j < m*n; j++)
//{
//data2[j] = (double) data2f[j];
//}

printf("before return result to matlab \n");
data2[0] = 0;
data2[0] = (double) data2f[0];

// Clean-up memory on device and host 
mxFree(data1f);
mxFree(data2f);
cudaFree(data1f_gpu);
cudaFree(data2f_gpu);
}// for i

}
*/