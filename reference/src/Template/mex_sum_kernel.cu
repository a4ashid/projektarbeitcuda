
#include <hip/hip_runtime.h>

__global__ void summup(double* h, double* e, int N) 
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<N)
       h[idx] += e[idx];
    // __syncthreads();
}
