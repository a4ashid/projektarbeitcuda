#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"


__host__ size_t smat_size( int cnt_elements, int cnt_cols ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * cnt_elements
           + sizeof(t_mindex)  * (cnt_cols + 1);
}


__global__ void testsparseMatrixMul( t_FullMatrix pResultVector,t_SparseMatrix pSparseMatrix, t_FullMatrix b ) {

    t_mindex tix = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tix  < pSparseMatrix.m ) {
        //printf ( "\n block %u thread %u tix %u N %u", blockIdx.x, threadIdx.x, tix, pSparseMatrix.m );
        //printf("\n %u %f", tix, b.pElement[tix] );
        pResultVector.pElement[tix] = b.pElement[tix] - 1;
    }
}


__host__ void set_sparse_data( t_SparseMatrix A_in, t_SparseMatrix* A_out, void* mv ) {

    A_out->m     = A_in.m;
    A_out->n     = A_in.n;
    A_out->nzmax = A_in.nzmax;

    A_out->pCol       = (t_mindex *)  mv;
    A_out->pNZElement = (t_ve *)     (&A_out->pCol[A_out->nzmax] ) ;
    A_out->pRow       = (t_mindex *) (&A_out->pNZElement[A_out->nzmax]);

}

extern "C" void idrs_1st(

                     t_SparseMatrix A_in,    /* A Matrix in buyu-sparse-format */
                     t_ve*          b_in,    /* b as in A * b = x */

                     t_mindex N,

                     t_ve*          r_out,    /* the r from idrs.m line 6 : r = b - A*x; */

                     t_idrshandle*  ih_out  /* handle for haloding all the device pointers between matlab calls */

           ) {

    hipError_t e;
    size_t h_memblocksize;
    size_t d_memblocksize;

    t_SparseMatrix A_d;

    t_ve* d_tmpAb;
    t_ve* d_b;

    void *hostmem;
    void *devmem;

    h_memblocksize =   smat_size( A_in.nzmax, A_in.m )  /* A sparse     */
                     + N * sizeof( t_ve )             /* b full       */
                     ;

    d_memblocksize =  h_memblocksize
                    + N * sizeof( t_ve )            /* d_tmpAb         */
                    + N * sizeof( t_ve )            /* x             */
                    + N * sizeof( t_ve )            /* resvec        */
                      ;

    printf("\n using N = %u (full vector size )", N );
    printf("\n using %u bytes in Host   memory", h_memblocksize);
    printf("\n using %u bytes in Device memory", d_memblocksize);



    hostmem =   malloc( h_memblocksize );
    if ( hostmem == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you hostmem"); exit( -1 ); }

/*
      pcol       |  t_mindex  |  .nzmax
      pNZElement |  t_ve      |  .nzmax
      pRow       |  t_mindex  |  N
      b          |  t_ve      |  N
      d_tmpAb    |  t_ve      |  N
*/

    /* copy all parameter vectors to ony monoliythic block starting at hostmem */

    t_mindex *pcol = (t_mindex *) hostmem;
    memcpy( pcol, A_in.pCol, A_in.nzmax * sizeof(t_mindex) );

    t_ve* pNZElement =  (t_ve *) &pcol[A_in.nzmax] ;
    memcpy( pNZElement, A_in.pNZElement, A_in.nzmax *  sizeof(t_ve) );

    t_mindex* pRow = (t_mindex *) (&pNZElement[A_in.nzmax]);
    memcpy( pRow, A_in.pRow, ( A_in.m + 1 ) *  sizeof(t_mindex) );

    t_ve* b = (t_ve *) &pRow[A_in.m + 1];
    memcpy( b, b_in,  N *  sizeof(t_ve) );

    e = hipMalloc ( &devmem , d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMalloc")

    e = hipMemcpy( devmem, hostmem, h_memblocksize , hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpyHostToDevice");

    set_sparse_data(  A_in, &A_d, devmem );
    d_b     = (t_ve *) &A_d.pRow[A_in.m + 1];
    d_tmpAb = (t_ve *) &d_b[N];


    dim3 dimGrid ( 2 );
    dim3 dimBlock(32);

    /* --------------------------------------------------------------------- */

    t_FullMatrix mb;
    t_FullMatrix result;

    mb.m        = N;
    mb.n        = 1;
    mb.pElement = d_b;

    result.pElement = d_tmpAb;

    testsparseMatrixMul<<<dimGrid,dimBlock>>>( result, A_d, mb );
    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("testsparseMatrixMul");

    /* --------------------------------------------------------------------- */
    e = hipMemcpy( r_out, d_tmpAb, sizeof(t_ve) * N, hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK("hipMemcpyDeviceToHost");

    printf("\n*** IDRS.cu - unimplemented - doing nothing  *** \n");


    printf("\n first call of idrs_1st - unimplemented \n\n " );

}


extern "C" void idrs(

                     t_SparseMatrix A_h,

                     t_ve* b_h,
                     t_mindex s,
                     t_ve  tol,
                     t_mindex maxit,
                     t_ve* x0_h,
                     t_mindex N,


                     t_ve* x_h,  /* output vector */
                     t_ve* resvec_h,
                     t_mindex* piter
                  ) {
    hipError_t e;
    size_t h_memblocksize;
    size_t d_memblocksize;

    t_SparseMatrix A_d;

    void *hostmem;
    void *devmem;


    h_memblocksize =   smat_size( A_h.nzmax, A_h.m )  /* A sparse     */
                     + N * sizeof( t_ve )             /* b full       */
                     ;

    d_memblocksize =  h_memblocksize

                    + N * sizeof( t_ve )            /* x             */
                    + N * sizeof( t_ve )            /* resvec        */

                      ;

    printf("\n using N = %u (full vector size )", N );
    printf("\n using %u bytes in Host   memory", h_memblocksize);
    printf("\n using %u bytes in Device memory", d_memblocksize);



    hostmem =   malloc( h_memblocksize );
    if ( hostmem == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you hostmem"); exit( -1 ); }

/*
      pcol       |  t_mindex  |  .nzmax
      pNZElement |  t_ve      |  .nzmax
      pRow       |  t_mindex  |  N
      b          |  t_ve      |  N
*/

    /* copy all parameter vectors to ony monoliythic block starting at hostmem */

    t_mindex *pcol = (t_mindex *) hostmem;
    memcpy( pcol, A_h.pCol, A_h.nzmax * sizeof(t_mindex) );

    t_ve* pNZElement =  (t_ve *) &pcol[A_h.nzmax] ;
    memcpy( pNZElement, A_h.pNZElement, A_h.nzmax *  sizeof(t_ve) );

    t_mindex* pRow = (t_mindex *) (&pNZElement[A_h.nzmax]);
    memcpy( pRow, A_h.pRow, ( A_h.m + 1 ) *  sizeof(t_mindex) );

    t_ve* b = (t_ve *) &pRow[A_h.m + 1];
    memcpy( b, b_h,  N *  sizeof(t_ve) );

    e = hipMalloc ( &devmem , d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMalloc")

    e = hipMemcpy( devmem, hostmem, h_memblocksize , hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpyHostToDevice");

    A_d.m = A_h.m;
    A_d.n = A_h.n;
    A_d.nzmax = A_h.nzmax;



    A_d.pCol       = (t_mindex *) devmem;
    A_d.pNZElement = (t_ve *) (&A_d.pCol[A_d.nzmax] ) ;
    A_d.pRow       = (t_mindex *) (&A_d.pNZElement[A_d.nzmax]);


    printf("\n*** IDRS.cu - unimplemented - doing nothing  *** \n");


    e = hipFree(devmem);
    CUDA_UTIL_ERRORCHECK("hipFree")
    free( hostmem );

}

