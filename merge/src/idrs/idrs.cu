#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "kernels/sparseMatrixMul_kernel.h"
#include "kernels/dotMul_cuda_gpu.h"


typedef struct idrs_context {
    void*          devmem1stcall;
    t_SparseMatrix A;
    t_ve*          b;
    t_ve*          r;
    t_ve*          v;

    t_ve*          om1;
    t_ve*          om2;

} t_idrs_context;


static t_idrs_context ctxholder[4];

extern "C" size_t idrs_sizetve() {
  return sizeof(t_ve);
}


__global__ void sub_arrays_gpu( t_ve *in1, t_ve *in2, t_ve *out, t_mindex N)
{
    t_mindex i = threadIdx.y * blockDim.x + threadIdx.x;
    if ( i < N )
        out[i] = in1[i] - in2[i];
}

__host__ size_t smat_size( int cnt_elements, int cnt_cols ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * cnt_elements
           + sizeof(t_mindex)  * (cnt_cols + 1);
}


extern "C" void idrs2nd(
    t_FullMatrix P,
    t_ve tol,
    unsigned int s,
    unsigned int maxit,
    t_idrshandle ih_in, /* Context Handle we got from idrs_1st */
    t_ve* x,
    t_ve* resvec,
   unsigned int* piter
) {
    hipError_t e;
    t_idrshandle ctx;


    t_FullMatrix mv;
    t_FullMatrix mr;

    int cnt_multiprozessors;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    t_ve* om1;
    t_ve* om2;

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
        cnt_multiprozessors = deviceProp.multiProcessorCount;
    }

    printf("\n 2nd context handle %u", ih_in );
    printf("do nothing");

    ctx = ih_in;

    t_SparseMatrix A         = ctxholder[ctx].A ;

    mr.m        = A.m;
    mr.n        = 1;
    mr.pElement = ctxholder[ctx].r;

    mv.m        = A.m;
    mv.n        = 1;
    mv.pElement = ctxholder[ctx].v;

    om1 = ctxholder[ctx].om1;
    om2 = ctxholder[ctx].om2;

    dim3 dimGrid ( cnt_multiprozessors );
    dim3 dimBlock(512);
    dim3 dimGridsub( A.m / 512 + 1 );

    for ( int k = 1; k <= s; k++ ) {
        /* idrs.m line 23 */
        sparseMatrixMul<<<dimGrid,dimBlock>>>( mv, A, mr );
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("testsparseMatrixMul");

        kernel_dotmul<<<dimGridsub,dimBlock>>>( mv.pElement, mr.pElement, om1 ) ;
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("device_dotMul");


        kernel_dotmul<<<dimGridsub,dimBlock>>>( mv.pElement, mv.pElement, om2 ) ;
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("device_dotMul");

        e = hipStreamSynchronize(0);
        CUDA_UTIL_ERRORCHECK("hipStreamSynchronize(0)");
    }



    e = hipFree( ctxholder[ctx].devmem1stcall );
    CUDA_UTIL_ERRORCHECK("hipFree ctxholder[ctx].devmem1stcall ");
}


/*
__global__ void testsparseMatrixMul( t_FullMatrix pResultVector,t_SparseMatrix pSparseMatrix, t_FullMatrix b ) {

    t_mindex tix = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tix  < pSparseMatrix.m ) {
        //printf ( "\n block %u thread %u tix %u N %u", blockIdx.x, threadIdx.x, tix, pSparseMatrix.m );
        //printf("\n %u %f", tix, b.pElement[tix] );
        pResultVector.pElement[tix] = b.pElement[tix] - 1;
    }
    if ( tix == 0 ) {
        for ( t_mindex i = 0; i < pSparseMatrix.m + 1 ; i++ ) {
             printf("\n pRow[%u] =  %u", i, pSparseMatrix.pRow[i] );
        }
        for ( t_mindex i = 0; i < pSparseMatrix.nzmax ; i++ ) {
            printf("\n pNZElement[%u] =  %f", i, pSparseMatrix.pNZElement[i] );
        }
        for ( t_mindex i = 0; i < pSparseMatrix.nzmax ; i++ ) {
            printf("\n pCol[%u] =  %u", i, pSparseMatrix.pCol[i] );
        }
    }

}
*/

__host__ void set_sparse_data( t_SparseMatrix A_in, t_SparseMatrix* A_out, void* mv ) {

    A_out->m     = A_in.m;
    A_out->n     = A_in.n;
    A_out->nzmax = A_in.nzmax;

    A_out->pCol       = (t_mindex *)  mv;
    A_out->pNZElement = (t_ve *)     (&A_out->pCol[A_out->nzmax] ) ;
    A_out->pRow       = (t_mindex *) (&A_out->pNZElement[A_out->nzmax]);

}

extern "C" void idrs_1st(

                     t_SparseMatrix A_in,    /* A Matrix in buyu-sparse-format */
                     t_ve*          b_in,    /* b as in A * b = x */
                     t_ve*          xe_in,
                     t_mindex N,

                     t_ve*          r_out,    /* the r from idrs.m line 6 : r = b - A*x; */

                     t_idrshandle*  ih_out  /* handle for haloding all the device pointers between matlab calls */

           ) {



    t_idrshandle ctx;

    hipError_t e;
    size_t h_memblocksize;
    size_t d_memblocksize;

    t_SparseMatrix A_d;

    t_ve* d_tmpAb;
    t_ve* d_b;
    t_ve* d_xe;
    t_ve* d_r;
    t_ve* xe;

    void *hostmem;
    void *devmem;

    ctx = 0;

    int cnt_multiprozessors;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
        cnt_multiprozessors = deviceProp.multiProcessorCount;
    }


    h_memblocksize =   smat_size( A_in.nzmax, A_in.m )  /* A sparse     */
                     + N * sizeof( t_ve )               /* b full       */
                     + N * sizeof( t_ve )               /* xe        */
                     ;

    d_memblocksize =  h_memblocksize
                    + (N + 512) * sizeof( t_ve )            /* d_tmpAb         */
                    + (N + 512) * sizeof( t_ve )            /* d_r             */
                    + N * sizeof( t_ve )            /* om1             */
                    + N * sizeof( t_ve )            /* om2             */
                    + N * sizeof( t_ve )            /* x               */
                    + N * sizeof( t_ve )            /* resvec          */

                      ;

    printf("\n using N = %u (full vector size )", N );
    printf("\n using %u bytes in Host   memory", h_memblocksize);
    printf("\n using %u bytes in Device memory", d_memblocksize);



    hostmem =   malloc( h_memblocksize );
    if ( hostmem == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you hostmem"); exit( -1 ); }

/*
      pcol       |  t_mindex  |  .nzmax
      pNZElement |  t_ve      |  .nzmax
      pRow       |  t_mindex  |  N
      b          |  t_ve      |  N
      d_xe       |  t_ve      |  N
      d_tmpAb    |  t_ve      |  N
      d_r        |  t_ve      |  N
      d_om1      |  t_ve      |  N
      d_om2      |  t_ve      |  N

*/

    /* copy all parameter vectors to ony monoliythic block starting at hostmem */

    t_mindex *pcol = (t_mindex *) hostmem;
    memcpy( pcol, A_in.pCol, A_in.nzmax * sizeof(t_mindex) );

    t_ve* pNZElement =  (t_ve *) &pcol[A_in.nzmax] ;
    memcpy( pNZElement, A_in.pNZElement, A_in.nzmax *  sizeof(t_ve) );

    t_mindex* pRow = (t_mindex *) (&pNZElement[A_in.nzmax]);
    memcpy( pRow, A_in.pRow, ( A_in.m + 1 ) *  sizeof(t_mindex) );

    t_ve* b = (t_ve *) &pRow[A_in.m + 1];
    memcpy( b, b_in,  N *  sizeof(t_ve) );

    xe = (t_ve *) &b[N];
    memcpy( xe, xe_in,  N *  sizeof(t_ve) );

    e = hipMalloc ( &devmem , d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMalloc")

    e = hipMemcpy( devmem, hostmem, h_memblocksize , hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpyHostToDevice");

    free(hostmem);

    set_sparse_data(  A_in, &A_d, devmem );
    d_b     = (t_ve *) &A_d.pRow[A_in.m + 1];
    d_xe    = (t_ve *) &d_b[N];

    d_tmpAb = (t_ve *) &d_xe[N];
    d_r     = (t_ve *) &d_tmpAb[ N + 512 ];

    ctxholder[ctx].om1 = (t_ve *) &d_r[N + 512 ];
    ctxholder[ctx].om2 = (t_ve *) &ctxholder[ctx].om1[N];

    dim3 dimGrid ( cnt_multiprozessors );
    dim3 dimGridsub( N / 512 + 1 );
    dim3 dimBlock(512);

    /* --------------------------------------------------------------------- */

    t_FullMatrix mxe;
    t_FullMatrix result;

    mxe.m        = N;
    mxe.n        = 1;
    mxe.pElement = d_xe;

    result.pElement = d_tmpAb;
    result.m    = N ;
    result.n    = 1;
    //testsparseMatrixMul<<<dimGrid,dimBlock>>>( result, A_d, mb );
    sparseMatrixMul<<<dimGrid,dimBlock>>>( result, A_d, mxe );
    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("testsparseMatrixMul");


//   add_arrays_gpu( t_ve *in1, t_ve *in2, t_ve *out, t_mindex N)
    sub_arrays_gpu<<<dimGridsub,dimBlock>>>( d_b, d_tmpAb, d_r, N);
    CUDA_UTIL_ERRORCHECK("sub_arrays_gpu");
    /* --------------------------------------------------------------------- */
    e = hipMemcpy( r_out, d_r, sizeof(t_ve) * N, hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK("hipMemcpyDeviceToHost");


    ctxholder[ctx].devmem1stcall = devmem;
    ctxholder[ctx].A             = A_d;
    ctxholder[ctx].b             = d_b;
    ctxholder[ctx].r             = d_r;
    ctxholder[ctx].v             = d_tmpAb; /* memory reusage */

    *ih_out = ctx;  /* context handle for later use in later calls */

}


