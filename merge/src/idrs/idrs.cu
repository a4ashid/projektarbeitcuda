#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "kernels/sparseMatrixMul_kernel.h"


extern "C" size_t idrs_sizetve() {
  return sizeof(t_ve);
}


__host__ size_t smat_size( int cnt_elements, int cnt_cols ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * cnt_elements
           + sizeof(t_mindex)  * (cnt_cols + 1);
}


__global__ void testsparseMatrixMul( t_FullMatrix pResultVector,t_SparseMatrix pSparseMatrix, t_FullMatrix b ) {

    t_mindex tix = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tix  < pSparseMatrix.m ) {
        //printf ( "\n block %u thread %u tix %u N %u", blockIdx.x, threadIdx.x, tix, pSparseMatrix.m );
        //printf("\n %u %f", tix, b.pElement[tix] );
        pResultVector.pElement[tix] = b.pElement[tix] - 1;
    }
    if ( tix == 0 ) {
        for ( t_mindex i = 0; i < pSparseMatrix.m + 1 ; i++ ) {
             printf("\n pRow[%u] =  %u", i, pSparseMatrix.pRow[i] );
        }
        for ( t_mindex i = 0; i < pSparseMatrix.nzmax ; i++ ) {
            printf("\n pNZElement[%u] =  %f", i, pSparseMatrix.pNZElement[i] );
        }
        for ( t_mindex i = 0; i < pSparseMatrix.nzmax ; i++ ) {
            printf("\n pCol[%u] =  %u", i, pSparseMatrix.pCol[i] );
        }
    }

}


__host__ void set_sparse_data( t_SparseMatrix A_in, t_SparseMatrix* A_out, void* mv ) {

    A_out->m     = A_in.m;
    A_out->n     = A_in.n;
    A_out->nzmax = A_in.nzmax;

    A_out->pCol       = (t_mindex *)  mv;
    A_out->pNZElement = (t_ve *)     (&A_out->pCol[A_out->nzmax] ) ;
    A_out->pRow       = (t_mindex *) (&A_out->pNZElement[A_out->nzmax]);

}

extern "C" void idrs_1st(

                     t_SparseMatrix A_in,    /* A Matrix in buyu-sparse-format */
                     t_ve*          b_in,    /* b as in A * b = x */

                     t_mindex N,

                     t_ve*          r_out,    /* the r from idrs.m line 6 : r = b - A*x; */

                     t_idrshandle*  ih_out  /* handle for haloding all the device pointers between matlab calls */

           ) {

    hipError_t e;
    size_t h_memblocksize;
    size_t d_memblocksize;

    t_SparseMatrix A_d;

    t_ve* d_tmpAb;
    t_ve* d_b;

    void *hostmem;
    void *devmem;

    h_memblocksize =   smat_size( A_in.nzmax, A_in.m )  /* A sparse     */
                     + N * sizeof( t_ve )             /* b full       */
                     ;

    d_memblocksize =  h_memblocksize
                    + N * sizeof( t_ve )            /* d_tmpAb         */
                    + N * sizeof( t_ve )            /* x             */
                    + N * sizeof( t_ve )            /* resvec        */
                      ;

    printf("\n using N = %u (full vector size )", N );
    printf("\n using %u bytes in Host   memory", h_memblocksize);
    printf("\n using %u bytes in Device memory", d_memblocksize);



    hostmem =   malloc( h_memblocksize );
    if ( hostmem == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you hostmem"); exit( -1 ); }

/*
      pcol       |  t_mindex  |  .nzmax
      pNZElement |  t_ve      |  .nzmax
      pRow       |  t_mindex  |  N
      b          |  t_ve      |  N
      d_tmpAb    |  t_ve      |  N
*/

    /* copy all parameter vectors to ony monoliythic block starting at hostmem */

    t_mindex *pcol = (t_mindex *) hostmem;
    memcpy( pcol, A_in.pCol, A_in.nzmax * sizeof(t_mindex) );

    t_ve* pNZElement =  (t_ve *) &pcol[A_in.nzmax] ;
    memcpy( pNZElement, A_in.pNZElement, A_in.nzmax *  sizeof(t_ve) );

    t_mindex* pRow = (t_mindex *) (&pNZElement[A_in.nzmax]);
    memcpy( pRow, A_in.pRow, ( A_in.m + 1 ) *  sizeof(t_mindex) );

    t_ve* b = (t_ve *) &pRow[A_in.m + 1];
    memcpy( b, b_in,  N *  sizeof(t_ve) );

    e = hipMalloc ( &devmem , d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMalloc")

    e = hipMemcpy( devmem, hostmem, h_memblocksize , hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpyHostToDevice");

    set_sparse_data(  A_in, &A_d, devmem );
    d_b     = (t_ve *) &A_d.pRow[A_in.m + 1];
    d_tmpAb = (t_ve *) &d_b[N];


    dim3 dimGrid ( 10 );
    dim3 dimBlock(512);

    /* --------------------------------------------------------------------- */

    t_FullMatrix mb;
    t_FullMatrix result;

    mb.m        = N;
    mb.n        = 1;
    mb.pElement = d_b;

    result.pElement = d_tmpAb;
    result.m    = N ;
    result.n    = 1;
    //testsparseMatrixMul<<<dimGrid,dimBlock>>>( result, A_d, mb );
    sparseMatrixMul<<<dimGrid,dimBlock>>>( result, A_d, mb );
    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("testsparseMatrixMul");

    /* --------------------------------------------------------------------- */
    e = hipMemcpy( r_out, d_tmpAb, sizeof(t_ve) * N, hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK("hipMemcpyDeviceToHost");

    printf("\n*** IDRS.cu - unimplemented - doing nothing  *** \n");


    printf("\n first call of idrs_1st - unimplemented \n\n " );

}


