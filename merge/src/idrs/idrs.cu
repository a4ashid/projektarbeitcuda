#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "kernels/sparseMatrixMul_kernel.h"
#include "kernels/dotMul_cuda_gpu.h"
#include "kernels/norm_cuda_gpu.h"
#include "kernels/gausskernel.h"

#include "bastianortho.h"

#include "kernels/matrixMul_kernel.h"

typedef struct idrs_context {
    void*          devmem1stcall;
    t_SparseMatrix A;
    t_ve*          b;
    t_ve*          r;
    t_ve*          v;
    t_ve*          x;

    t_ve*          om1;
    t_ve*          om2;

} t_idrs_context;


static t_idrs_context ctxholder[4];

extern "C" size_t idrs_sizetve() {
  return sizeof(t_ve);
}


__host__  void testortholinkcompileonly() {

    t_ve dummyRes;
    t_ve dummyP;
    orthogonalize( &dummyP, &dummyRes, 12345, 6 );
}

__global__ void kernel_vec_mul_skalar( t_ve *invec, t_ve scalar, t_ve *out, t_mindex N )
{
    t_mindex i = threadIdx.y * blockDim.x + threadIdx.x;
    if ( i < N )
        out[i] = invec[i] * scalar;
}


__global__ void sub_arrays_gpu( t_ve *in1, t_ve *in2, t_ve *out, t_mindex N)
{
    t_mindex i = threadIdx.y * blockDim.x + threadIdx.x;
    if ( i < N )
        out[i] = in1[i] - in2[i];

}

__global__ void sub_and_mul_arrays_gpu(
                                         t_ve *in1,
                                         t_ve *in2,
                                         t_ve coefficient,
                                         t_ve *out,
                                         t_mindex N
                                        )
{
    t_mindex i = threadIdx.y * blockDim.x + threadIdx.x;
    if ( i < N )
        out[i] = in1[i] - coefficient * in2[i];

}


__global__ void add_and_mul_arrays_gpu(
                                         t_ve *in1,
                                         t_ve *in2,
                                         t_ve coefficient,
                                         t_ve *out,
                                         t_mindex N
                                        )
{
    t_mindex i = threadIdx.y * blockDim.x + threadIdx.x;
    if ( i < N )
        out[i] = in1[i] + coefficient * in2[i];

}



__global__ void add_arrays_gpu( t_ve *in1, t_ve *in2, t_ve *out, t_mindex N)
{
    t_mindex i = threadIdx.y * blockDim.x + threadIdx.x;
    if ( i < N )
        out[i] = in1[i] + in2[i];
}

__host__ size_t smat_size( int cnt_elements, int cnt_cols ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * cnt_elements
           + sizeof(t_mindex)  * (cnt_cols + 1);
}


extern "C" void idrs2nd(
    t_FullMatrix P_in,
    t_ve tol,
    unsigned int s,
    unsigned int maxit,
    t_idrshandle ih_in, /* Context Handle we got from idrs_1st */
    t_ve* x,
    t_ve* resvec,
   unsigned int* piter
) {
    hipError_t e;
    t_idrshandle ctx;


    t_FullMatrix mv;
    t_FullMatrix mr;
    t_FullMatrix mt;

    int cnt_multiprozessors;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    t_ve* om1;
    t_ve* om2;
    t_ve* v;

    t_mindex resveci  = 1;
    void* devmem;

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
        cnt_multiprozessors = deviceProp.multiProcessorCount;
    }

    printf("\n 2nd context handle %u", ih_in );
    printf("do nothing");

    ctx = ih_in;

    t_SparseMatrix A         = ctxholder[ctx].A ;

    t_mindex N = A.m;

    size_t h_memblocksize =   N * sizeof( t_ve )            /* om1             */
                            + N * sizeof( t_ve )            /* om2             */
                            + N * sizeof( t_ve )            /* debugbuffer1    */
                            + N * sizeof( t_ve )            /* h_norm    */
                            ;

    size_t d_memblocksize =  (N*s )       * sizeof( t_ve )           /* P      */
                           + s * (s + 1 + 1 )       * sizeof( t_ve )           /* M m c    */
                           + ( N + 512 )  * sizeof( t_ve )            /* v      */
                           + (N*s )       * sizeof( t_ve )            /* dR     */
                           + (N*s )       * sizeof( t_ve )            /* dX     */
                           + (N )         * sizeof( t_ve )            /* dR_k   */
                           + (N )         * sizeof( t_ve )            /* dX_k   */
                           + (N )         * sizeof( t_ve )            /* dnormv   */
                           + (N )         * sizeof( t_ve )            /* q   */
                           + (N + 512 )   * sizeof( t_ve )            /* t   */
                           + (N + 512    ) * sizeof( t_ve )           /* buffer1   */
                           + (N + 512    ) * sizeof( t_ve )           /* dm   */
//                           + (N ) * sizeof( t_ve )                  /* x   */
                      ;

    e = hipMalloc ( &devmem , d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMalloc");

    e = hipMemset (devmem, 0, d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMalloc");

    printf("\n additional using %u bytes in Device memory", d_memblocksize);

    t_ve* P      = (t_ve*) devmem ;
    t_ve* M      = &P[ N * s ];
    t_ve* m      = &M[ s * s ];
    t_ve* c      = &M[ s  ];
    v            = &c[ s  ];
    t_ve* dR     = &v[N + 512 ];
    t_ve* dX     = &dR[ N * s ];
    t_ve* dR_k   = &dX[ N * s ];
    t_ve* dX_k   = &dR_k[ N  ];
    t_ve* dnormv = &dX_k[ N  ];
    t_ve* q      = &dnormv[ N  ];
    t_ve* t      = &q[ N  ];
    t_ve* buffer1 = &t[N + 512 ];
    t_ve* dm      = &buffer1[N + 512 ];

    x          = ctxholder[ctx].x;

    void* hostmem =   malloc( h_memblocksize );
    if ( hostmem == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you hostmem"); exit( -1 ); }

    t_ve*  h_om1        = (t_ve*) hostmem;
    t_ve*  h_om2        = &h_om1[N];
    t_ve*  debugbuffer1 = &h_om2[N];
    t_ve*  h_norm        = &debugbuffer1[N];

    t_ve norm;


    mr.m        = A.m;
    mr.n        = 1;
    mr.pElement = ctxholder[ctx].r;

    mt.m        = A.m;
    mt.n        = 1;
    mt.pElement = t;


    t_ve* r = mr.pElement;

    mv.m        = A.m;
    mv.n        = 1;
    mv.pElement = v ;

    om1 = ctxholder[ctx].om1;
    om2 = ctxholder[ctx].om2;

    dim3 dimGrid ( cnt_multiprozessors );
    dim3 dimBlock(512);
    dim3 dimGridsub( A.m / 512 + 1 );

    dim3 dimGridgauss( 1 );
    dim3 dimBlockgauss(512);

    t_ve som ;

    for ( int k = 1; k <= s; k++ ) {

        dR_k = &dR[ N * (k-1) ];
        dX_k = &dX[ N * (k-1) ];

        /* idrs.m line 23 */
        sparseMatrixMul<<<dimGrid,dimBlock>>>( mv, A, mr );
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("testsparseMatrixMul");


        e = hipStreamSynchronize(0);
        CUDA_UTIL_ERRORCHECK("hipStreamSynchronize(0)");

/*
        if ( N < 200 ) {
            e = hipMemcpy( debugbuffer1, mv.pElement, sizeof(t_ve) * N , hipMemcpyDeviceToHost);
            CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");
            if ( k == 1 ) {
               for ( t_mindex i = 0; i < N; i++ )
               printf("\n k = 1, mv.pElement[%u] = %f", i, debugbuffer1[i]);
            }
        }
*/
        kernel_dotmul<<<dimGridsub,dimBlock>>>( v, r, om1 ) ;
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("device_dotMul");

        e = hipStreamSynchronize(0);
        CUDA_UTIL_ERRORCHECK("hipStreamSynchronize(0)");

/*
        if ( N < 200 ) {
            e = hipMemcpy( debugbuffer1, om1, sizeof(t_ve) * N , hipMemcpyDeviceToHost);
            CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");
            if ( k == 1 ) {
               for ( t_mindex i = 0; i < N; i++ )
               printf("\n k = 1, om1[%u] = %f", i, debugbuffer1[i]);
            }
        }
*/
        kernel_dotmul<<<dimGridsub,dimBlock>>>( mv.pElement, mv.pElement, om2 ) ;
        //kernel_dotmul<<<vdimGridsub,dimBlock>>>( ctxholder[ctx].b, ctxholder[ctx].b, om2 ) ;
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("device_dotMul");

        e = hipStreamSynchronize(0);
        CUDA_UTIL_ERRORCHECK("hipStreamSynchronize(0)");

        e = hipMemcpy( h_om1, om1, sizeof(t_ve) * N * 2, hipMemcpyDeviceToHost);
        CUDA_UTIL_ERRORCHECK("hipMemcpy( h_om1, om1, sizeof(t_ve) * N * 2, hipMemcpyDeviceToHost)");

        t_ve  som1 = 0;
        t_ve  som2 = 0;
        for ( t_mindex blockidx = 0; blockidx < A.m / 512 + 1; blockidx++ ) {
            som1 += h_om1[blockidx];
            som2 += h_om2[blockidx];
        }
        som = som1 / som2;



        kernel_vec_mul_skalar<<<dimGridsub,dimBlock>>>( mr.pElement,   som , dX_k, N );
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("kernel_vec_mul_skalar<<<dimGridsub,dimBlock>>>( mr.pElement,   som , dX_k, N )");

        kernel_vec_mul_skalar<<<dimGridsub,dimBlock>>>( mv.pElement, - som , dR_k, N );
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("kernel_vec_mul_skalar<<<dimGridsub,dimBlock>>>( mv.pElement, - som , dR_k, N )");


        e = hipStreamSynchronize(0);
        CUDA_UTIL_ERRORCHECK("hipStreamSynchronize(0)");

        add_arrays_gpu<<<dimGridsub,dimBlock>>>( x, dX_k, x, N );
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("add_arrays_gpu<<<dimGridsub,dimBlock>>>( x, dX_k, x, N )");

        add_arrays_gpu<<<dimGridsub,dimBlock>>>( mr.pElement, dR_k, mr.pElement, N );
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("add_arrays_gpu<<<dimGridsub,dimBlock>>>( mr.pElement, dR_k, mr.pElement, N );");


        kernel_norm<<<dimGridsub,dimBlock>>>( mr.pElement, dnormv );
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("kernel_norm<<<dimGridsub,dimBlock>>>( mr.pElement, dnormv )");


        e = hipMemcpy( h_norm, dnormv, sizeof(t_ve) * N , hipMemcpyDeviceToHost);
        CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

        t_ve snorm = 0;
        for ( t_mindex i = 0; i < N / 512 + 1 ; i++ ) {
             snorm +=  h_norm[i];
        }
        norm = snorm;
        resvec[ resveci++ ]  = sqrt( norm );

        /* 28    M(:,k) = P*dR(:,k); */

        t_ve* Mk = &M[ s * (k-1) ];
        matrixMul<<<dimGrid,dimBlock>>>( P, dR_k , Mk, s, 1 ); /*  :-/  :-/ :-/ :-( */
        e = hipGetLastError();
        CUDA_UTIL_ERRORCHECK("matrixMul<<<dimGrid,dimBlock>>>( P, r , m, s, 1 )");


        printf("\n iteration %u,    1 %f   2 %f", k , som1, som2 );

        e = hipStreamSynchronize(0);
        CUDA_UTIL_ERRORCHECK("hipStreamSynchronize(0)");
    }


    t_mindex iter   = s; /* iter.m line 31 */
    t_mindex oldest = 0; /* iter.m line 32 */


    /*
     *
     *  33   m = P* r   - we need a Matrixmul that can deal with P(s,N)   (mA = s = 6 << N) :-/
     *
     *
     */

    matrixMul<<<dimGrid,dimBlock>>>( P, r , m, s, 1 ); /*  */
    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("matrixMul<<<dimGrid,dimBlock>>>( P, r , m, s, 1 )");


    while (  (norm > tol ) && ( iter < maxit )  ) {
        for ( t_mindex k = 0; k <= s; k++ ) {

           t_ve* dRoldest = &dR[ oldest  * N ];
           t_ve* dXoldest = &dX[ oldest  * N ];

           /* c = M\n  iter.m line 36 */
           device_gauss_solver<<<dimGridgauss,dimBlockgauss>>>( M, s, c ); /* vec m is s+1 column of M - see memory allocation plan  */
           e = hipGetLastError();
           CUDA_UTIL_ERRORCHECK("device_gauss_solver<<<dimGridgauss,dimBlockgauss>>>( M, s, c )");


           /* q = -dR * c */
           matrixMul<<<dimGrid,dimBlock>>>( q, dR , c, N, 1 ); /* the - is missing */
           e = hipGetLastError();
           CUDA_UTIL_ERRORCHECK("matrixMul<<<dimGridgauss,dimBlockgauss>>>( q, dR , c, N, 1 )");

           add_arrays_gpu<<<dimGridsub,dimBlock>>>( r, q, v, N );
           e = hipGetLastError();
           CUDA_UTIL_ERRORCHECK("add_arrays_gpu<<<dimGridsub,dimBlock>>>( x, dX_k, x, N )");

           if ( k == 0 ) {
               /* t = A*v  idrs.m line 40 */
               sparseMatrixMul<<<dimGrid,dimBlock>>>( mt, A, mv );
               e = hipGetLastError();
               CUDA_UTIL_ERRORCHECK("sparseMatrixMul<<<dimGrid,dimBlock>>>( mt, A, mv )");




               kernel_dotmul<<<dimGridsub,dimBlock>>>( t, v, om1 ) ;
                //kernel_dotmul<<<dimGridsub,dimBlock>>>( ctxholder[ctx].b, ctxholder[ctx].b, om2 ) ;
               e = hipGetLastError();
               CUDA_UTIL_ERRORCHECK("device_dotMul");

               kernel_dotmul<<<dimGridsub,dimBlock>>>( t, t, om2 ) ;
                //kernel_dotmul<<<dimGridsub,dimBlock>>>( ctxholder[ctx].b, ctxholder[ctx].b, om2 ) ;
               e = hipGetLastError();
               CUDA_UTIL_ERRORCHECK("device_dotMul");

               e = hipStreamSynchronize(0);
               CUDA_UTIL_ERRORCHECK("hipStreamSynchronize(0)");

               e = hipMemcpy( h_om1, om1, sizeof(t_ve) * N * 2, hipMemcpyDeviceToHost);
               CUDA_UTIL_ERRORCHECK("hipMemcpy( h_om1, om1, sizeof(t_ve) * N * 2, hipMemcpyDeviceToHost)");

               t_ve  som1 = 0;
               t_ve  som2 = 0;
               for ( t_mindex blockidx = 0; blockidx < A.m / 512 + 1; blockidx++ ) {
                    som1 += h_om1[blockidx];
                    som2 += h_om2[blockidx];
               }
               t_ve som = som1 / som2;

               /*  42            dR(:,oldest) = q - om*t; % 1 update */
               sub_and_mul_arrays_gpu<<<dimGridsub,dimBlock>>>( q, t, som, dRoldest , N);
               e = hipGetLastError();
               CUDA_UTIL_ERRORCHECK("sub_and_mul_arrays_gpu");


               /*  43    dX(:,oldest) = -dX*c + om*v; % s updates + 1 scaling */
               matrixMul<<<dimGrid,dimBlock>>>( dX, c , buffer1, N, 1 ); /* the - is missing */
               e = hipGetLastError();
               CUDA_UTIL_ERRORCHECK("matrixMul<<<dimGrid,dimBlock>>>( dX, c , dXoldest, N, 1 )");

               add_and_mul_arrays_gpu<<<dimGridsub,dimBlock>>>( buffer1, v, som, dXoldest , N);
               e = hipGetLastError();
               CUDA_UTIL_ERRORCHECK("add_and_mul_arrays_gpu");

               printf("\n k = %u om = %f  %f %f", k, som, som1, som2   );

           }
           else {

               t_FullMatrix mdRoldest;
               t_FullMatrix mdXoldest;

               mdRoldest.m        = 1;
               mdRoldest.n        = N;
               mdRoldest.pElement = dRoldest;

               mdXoldest.m        = 1;
               mdXoldest.n        = N;
               mdXoldest.pElement = dXoldest;

              /* 46  dR(:,oldest) = -A*dX(:,oldest); % 1 matmul */
               sparseMatrixMul<<<dimGrid,dimBlock>>>( mdRoldest, A, mdXoldest );
               e = hipGetLastError();
               CUDA_UTIL_ERRORCHECK("sparseMatrixMul<<<dimGrid,dimBlock>>>( mt, A, mv )");

               /*  45    dX(:,oldest) = -dX*c + om*v; % s updates + 1 scaling */
               matrixMul<<<dimGrid,dimBlock>>>( dX, c , buffer1, N, 1 ); /* the - is missing */
               e = hipGetLastError();
               CUDA_UTIL_ERRORCHECK("matrixMul<<<dimGrid,dimBlock>>>( dX, c , dXoldest, N, 1 )");

               add_and_mul_arrays_gpu<<<dimGridsub,dimBlock>>>( buffer1, v, som, dXoldest , N);
               e = hipGetLastError();
               CUDA_UTIL_ERRORCHECK("add_and_mul_arrays_gpu");
           }

           /*        r = r + dR(:,oldest); % simple addition */

           add_arrays_gpu<<<dimGridsub,dimBlock>>>( r, dRoldest, r, N );
           e = hipGetLastError();
           CUDA_UTIL_ERRORCHECK("add_arrays_gpu<<<dimGridsub,dimBlock>>>( r, dRoldest, r, N )");

           /* x = x + dX(:,oldest); % simple addition */

           add_arrays_gpu<<<dimGridsub,dimBlock>>>( x, dXoldest, x, N );
           e = hipGetLastError();
           CUDA_UTIL_ERRORCHECK("add_arrays_gpu<<<dimGridsub,dimBlock>>>( r, dRoldest, r, N )");



           //printf( "\n iterartion %u", iter );

           iter++;

           kernel_norm<<<dimGridsub,dimBlock>>>( mr.pElement, dnormv );
           e = hipGetLastError();
           CUDA_UTIL_ERRORCHECK("kernel_norm<<<dimGridsub,dimBlock>>>( mr.pElement, dnormv )");


           e = hipMemcpy( h_norm, dnormv, sizeof(t_ve) * N , hipMemcpyDeviceToHost);
           CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

            t_ve snorm = 0;
            for ( t_mindex i = 0; i < N / 512 + 1 ; i++ ) {
                 snorm +=  h_norm[i];
            }
            norm = snorm;
            resvec[ resveci++ ]  = sqrt( norm );
            printf( "\n iterartion %u norm %f", iter, norm );

            t_ve* Moldest = &M[ s * oldest ];


            /* 53 dm = P*dR(:,oldest); % s inner products */
            dm = Moldest;
            matrixMul<<<dimGrid,dimBlock>>>( P, dRoldest , Moldest, s, 1 ); /* the - is missing */
            e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("matrixMul<<<dimGrid,dimBlock>>>( P, dRoldest , Moldest, s, 1 )");

            /* 55  m = m + dm; */
            add_arrays_gpu<<<dimGridgauss,dimBlock>>>( m, dm, m, s );
            e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("add_arrays_gpu<<<dimGridsub,dimBlock>>>( r, dRoldest, r, N )");

            oldest++;
            if ( oldest > s - 1 ) {
               oldest = 1 ;
            }
        }

    }
    *piter = iter;

    e = hipFree( devmem );
    CUDA_UTIL_ERRORCHECK("e = hipFree( devmem );");

    e = hipFree( ctxholder[ctx].devmem1stcall );
    CUDA_UTIL_ERRORCHECK("hipFree ctxholder[ctx].devmem1stcall ");
}


/*
__global__ void testsparseMatrixMul( t_FullMatrix pResultVector,t_SparseMatrix pSparseMatrix, t_FullMatrix b ) {

    t_mindex tix = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tix  < pSparseMatrix.m ) {
        //printf ( "\n block %u thread %u tix %u N %u", blockIdx.x, threadIdx.x, tix, pSparseMatrix.m );
        //printf("\n %u %f", tix, b.pElement[tix] );
        pResultVector.pElement[tix] = b.pElement[tix] - 1;
    }
    if ( tix == 0 ) {
        for ( t_mindex i = 0; i < pSparseMatrix.m + 1 ; i++ ) {
             printf("\n pRow[%u] =  %u", i, pSparseMatrix.pRow[i] );
        }
        for ( t_mindex i = 0; i < pSparseMatrix.nzmax ; i++ ) {
            printf("\n pNZElement[%u] =  %f", i, pSparseMatrix.pNZElement[i] );
        }
        for ( t_mindex i = 0; i < pSparseMatrix.nzmax ; i++ ) {
            printf("\n pCol[%u] =  %u", i, pSparseMatrix.pCol[i] );
        }
    }

}
*/

__host__ void set_sparse_data( t_SparseMatrix A_in, t_SparseMatrix* A_out, void* mv ) {

    A_out->m     = A_in.m;
    A_out->n     = A_in.n;
    A_out->nzmax = A_in.nzmax;

    A_out->pCol       = (t_mindex *)  mv;
    A_out->pNZElement = (t_ve *)     (&A_out->pCol[A_out->nzmax] ) ;
    A_out->pRow       = (t_mindex *) (&A_out->pNZElement[A_out->nzmax]);

}

extern "C" void idrs_1st(

                     t_SparseMatrix A_in,    /* A Matrix in buyu-sparse-format */
                     t_ve*          b_in,    /* b as in A * b = x */
                     t_ve*          xe_in,
                     t_mindex N,

                     t_ve*          r_out,    /* the r from idrs.m line 6 : r = b - A*x; */

                     t_idrshandle*  ih_out  /* handle for haloding all the device pointers between matlab calls */

           ) {



    t_idrshandle ctx;

    hipError_t e;
    size_t h_memblocksize;
    size_t d_memblocksize;

    t_SparseMatrix A_d;

    t_ve* d_tmpAb;
    t_ve* d_b;
    t_ve* d_xe;
    t_ve* d_r;
    t_ve* xe;

    void *hostmem;
    void *devmem;

    ctx = 0;

    int cnt_multiprozessors;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
        cnt_multiprozessors = deviceProp.multiProcessorCount;
    }


    h_memblocksize =   smat_size( A_in.nzmax, A_in.m )  /* A sparse     */
                     + N * sizeof( t_ve )               /* b full       */
                     + N * sizeof( t_ve )               /* xe        */
                     ;

    d_memblocksize =  h_memblocksize
                    + (N + 512) * sizeof( t_ve )            /* d_tmpAb         */
                    + (N + 512) * sizeof( t_ve )            /* d_r             */
                    + N * sizeof( t_ve )            /* om1             */
                    + N * sizeof( t_ve )            /* om2             */
                    + N * sizeof( t_ve )            /* x               */
                    + N * sizeof( t_ve )            /* resvec          */

                      ;

    printf("\n using N = %u (full vector size )", N );
    printf("\n using %u bytes in Host   memory", h_memblocksize);
    printf("\n using %u bytes in Device memory", d_memblocksize);



    hostmem =   malloc( h_memblocksize );
    if ( hostmem == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you hostmem"); exit( -1 ); }

/*
      pcol       |  t_mindex  |  .nzmax
      pNZElement |  t_ve      |  .nzmax
      pRow       |  t_mindex  |  N
      b          |  t_ve      |  N
      d_xe       |  t_ve      |  N
      d_tmpAb    |  t_ve      |  N
      d_r        |  t_ve      |  N
      d_om1      |  t_ve      |  N
      d_om2      |  t_ve      |  N

*/

    /* copy all parameter vectors to ony monoliythic block starting at hostmem */

    t_mindex *pcol = (t_mindex *) hostmem;
    memcpy( pcol, A_in.pCol, A_in.nzmax * sizeof(t_mindex) );

    t_ve* pNZElement =  (t_ve *) &pcol[A_in.nzmax] ;
    memcpy( pNZElement, A_in.pNZElement, A_in.nzmax *  sizeof(t_ve) );

    t_mindex* pRow = (t_mindex *) (&pNZElement[A_in.nzmax]);
    memcpy( pRow, A_in.pRow, ( A_in.m + 1 ) *  sizeof(t_mindex) );

    t_ve* b = (t_ve *) &pRow[A_in.m + 1];
    memcpy( b, b_in,  N *  sizeof(t_ve) );

    xe = (t_ve *) &b[N];
    memcpy( xe, xe_in,  N *  sizeof(t_ve) );

    e = hipMalloc ( &devmem , d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMalloc")

    e = hipMemset (devmem, 0, d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMemset");

    e = hipMemcpy( devmem, hostmem, h_memblocksize , hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpyHostToDevice");

    free(hostmem);

    set_sparse_data(  A_in, &A_d, devmem );
    d_b     = (t_ve *) &A_d.pRow[A_in.m + 1];
    d_xe    = (t_ve *) &d_b[N];

    d_tmpAb = (t_ve *) &d_xe[N];
    d_r     = (t_ve *) &d_tmpAb[ N + 512 ];

    ctxholder[ctx].om1 = (t_ve *) &d_r[N + 512 ];
    ctxholder[ctx].om2 = (t_ve *) &ctxholder[ctx].om1[N];

    dim3 dimGrid ( cnt_multiprozessors );
    dim3 dimGridsub( N / 512 + 1 );
    dim3 dimBlock(512);

    /* --------------------------------------------------------------------- */

    t_FullMatrix mxe;
    t_FullMatrix result;

    mxe.m        = N;
    mxe.n        = 1;
    mxe.pElement = d_xe;

    result.pElement = d_tmpAb;
    result.m    = N ;
    result.n    = 1;
    //testsparseMatrixMul<<<dimGrid,dimBlock>>>( result, A_d, mb );
    sparseMatrixMul<<<dimGrid,dimBlock>>>( result, A_d, mxe );
    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("testsparseMatrixMul");


//   add_arrays_gpu( t_ve *in1, t_ve *in2, t_ve *out, t_mindex N)
    sub_arrays_gpu<<<dimGridsub,dimBlock>>>( d_b, d_tmpAb, d_r, N);
    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("sub_arrays_gpu");
    /* --------------------------------------------------------------------- */
    e = hipMemcpy( r_out, d_r, sizeof(t_ve) * N, hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK("hipMemcpyDeviceToHost");


    ctxholder[ctx].devmem1stcall = devmem;
    ctxholder[ctx].A             = A_d;
    ctxholder[ctx].b             = d_b;
    ctxholder[ctx].r             = d_r;
    ctxholder[ctx].v             = d_tmpAb; /* memory reusage */
    ctxholder[ctx].x             = d_xe;

    *ih_out = ctx;  /* context handle for later use in later calls */

}


