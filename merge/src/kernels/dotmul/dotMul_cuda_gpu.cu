#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"
//#include "mex.h"
/*
 release :dotMul_cuda_gpu05
 Kernel to square elements of the array on the GPU 
*/

__global__ void device_dotMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
	__shared__ float Cs[VECTOR_BLOCK_SIZE];
	
	int tid = threadIdx.x;
	int idx = blockIdx.x*blockDim.x + tid;
	
	Cs[threadIdx.x] = 0;

	if ( idx < N ) {
	    Cs[threadIdx.x] = in1[ idx ] * in2[ idx ];
	}
	__syncthreads();
	
	t_ve blocksum = 0;
	
	int offset; 
	offset = VECTOR_BLOCK_SIZE/2;
	while (offset > 0) {
		if(tid < offset) {
			Cs[tid] += Cs[tid + offset];
		}
		offset >>= 1;
		__syncthreads();
	}
	/*
	if(tid < 256) {Cs[tid] += Cs[tid + 256];}
	__syncthreads();
	if(tid < 128) {Cs[tid] += Cs[tid + 128];}
	__syncthreads();
	if(tid < 64) {Cs[tid] += Cs[tid + 64];}
	__syncthreads();
	if(tid < 32) {Cs[tid] += Cs[tid + 32];}
	__syncthreads();
	if(tid < 16) {Cs[tid] += Cs[tid + 16];}
	__syncthreads();
	if(tid < 8) {Cs[tid] += Cs[tid + 8];}
	__syncthreads();
	if(tid < 4) {Cs[tid] += Cs[tid + 4];}
	__syncthreads();
	if(tid < 2) {Cs[tid] += Cs[tid + 2];}
	__syncthreads();
	if(tid < 1) {Cs[tid] += Cs[tid + 1];}
	__syncthreads();
	*/
	
	out[blockIdx.x]=0;
	out[blockIdx.x]=Cs[0];
	__syncthreads();
	
	////block summe in cpu
	/*
	if ( idx == 0 ) {
	     for ( int i = 1; i < gridDim.x; i++ ) {
		     out[0] += out[i];
		 }
	}
	*/
}