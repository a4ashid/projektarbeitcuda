#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "projektcuda.h"



__global__ void kernel_dotmul( t_ve *in1,
                               t_ve *in2,
                               t_ve *out
                             ) {
    __shared__ t_ve Vs [DEF_BLOCKSIZE];


    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    Vs[threadIdx.x] = in1[idx] * in2[idx];


    __syncthreads();
    if ( threadIdx.x < 256 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  + 256 ]; }
    __syncthreads();

    if ( threadIdx.x < 128 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  + 128 ];}
    __syncthreads();

    if ( threadIdx.x <  64 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  +  64 ];}
    __syncthreads();

#ifndef PRJCUDAEMU

    if ( threadIdx.x <  32 ) {
        Vs[threadIdx.x] += Vs[ threadIdx.x + 32 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x + 16 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  8 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  4 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  2 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  1 ];

        if ( threadIdx.x == 0 ) {
            //out[blockIdx.x] =  Vs[0]  ;
            out[blockIdx.x] =  Vs[0]  ;
        }
    }

#endif

#ifdef PRJCUDAEMU

    if ( threadIdx.x <  32 )
        Vs[threadIdx.x] += Vs[ threadIdx.x + 32 ];
    __syncthreads();
    if ( threadIdx.x <  16 )
        Vs[threadIdx.x] += Vs[ threadIdx.x + 16 ];
    __syncthreads();
    if ( threadIdx.x <  8 )
        Vs[threadIdx.x] += Vs[ threadIdx.x +  8 ];
    __syncthreads();
    if ( threadIdx.x <  4 )
        Vs[threadIdx.x] += Vs[ threadIdx.x +  4 ];
    __syncthreads();
    if ( threadIdx.x <  2 )
        Vs[threadIdx.x] += Vs[ threadIdx.x +  2 ];
    __syncthreads();
    if ( threadIdx.x <  1 )
        Vs[threadIdx.x] += Vs[ threadIdx.x +  1 ];
    __syncthreads();
        if ( threadIdx.x == 0 ) {
            //out[blockIdx.x] =  Vs[0]  ;
            out[blockIdx.x] =  Vs[0]  ;
        }


#endif

}


__host__ void dbg_dotmul_checkresult ( t_ve *in1,
                                       t_ve *in2,
                                       t_ve tobeckecked,
                                       t_mindex N ,
                                       char* debugname
                                      )

                          {

    hipError_t e;


    t_ve* v1 = (t_ve*) malloc( sizeof( t_ve* ) * N );
    if (  v1 == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you C"); exit( -1 ); }

    t_ve* v2 = (t_ve*) malloc( sizeof( t_ve* ) * N );
    if (  v2 == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you C"); exit( -1 ); }

    e = hipMemcpy( v1, in1, sizeof(t_ve) * N , hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

    e = hipMemcpy( v2, in2, sizeof(t_ve) * N, hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

    t_ve calresult = 0;

    for( t_mindex i = 0; i < N; i++ ) {
        calresult += v1[i] * v2[i];
    }
    if ( abs( calresult - tobeckecked ) > 0.001 ) {
//        printf("\n Dotmul %s OK", debugname );
//    }
//    else {
        printf("\n Dotmul %s *not� OK :  expected %f, got %f", debugname , calresult, tobeckecked );
        exit( - 1 );
    }

    free(v1);
    free(v2);

}

