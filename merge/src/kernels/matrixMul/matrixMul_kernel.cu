#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"

#define AZeile(i,j,w) A[i*w+j]
#define ASpalte(i,j,w) A[j*w+i]

__global__ void matrixMul(
                            t_ve* C_out,
                            t_ve* A,
                            t_ve* B,
                            int mA,
                            int mB
                           ) {

   // blockIdx.x is current row of C and B

    if ( threadIdx.x == 0 ) {
       t_ve c = 0;

       for ( int as = 0; as < mB; as++ ) {
            c += A[ blockIdx.x + as * mA] * B[as];
       }

       C_out[blockIdx.x] = c;
    }


}

/* Kernel to square elements of the array on the GPU */
/*
	Matrix A is mA x nB  , Vector B is nB
	Vector C output vector in size of mA
	C=A*B
	matrixMul_kernel03.cu
description:
	each row of A occuppy one block. if gridDim is smaller than the row number of A
*/

__global__ void matrixMul_long_mA( t_ve* C, t_ve* A, t_ve* B, int mA, int nB) {

	//define a Result Vector for each block
	__shared__ float Cs[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 512

	//define gridIndex, if gridDim < mA, gridIndex > 0;
	int gridIndex = 0;
	// get a thread indentifier
	//int idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
	int aBegin = 0;
	int bBegin = 0;
	int aStep = gridDim.x;
	int bStep = VECTOR_BLOCK_SIZE; // blockDim.x
	int aEnd = mA;
	int bEnd = nB;
	int tx;
	tx = threadIdx.x;

		//initialise Cs
		Cs[tx] = 0;
		__syncthreads();
		//initialize output vector for each block
	if(tx==0){
		C[gridIndex*gridDim.x+blockIdx.x]=0;
	}
		__syncthreads();
	// if nB > gridDim???????
	//idx < (gridIndex*gridDim.x+mA%VECTOR_BLOCK_SIZE)*()
	for(int a = aBegin; (a < aEnd)&&((gridIndex*gridDim.x+blockIdx.x)<aEnd); a += aStep, gridIndex++){
		//initialize output vector for each block
		if(threadIdx.x==0){
			C[gridIndex*gridDim.x+blockIdx.x]=0;
		}
		__syncthreads();

		//following is operations within one block
		// initialize the dot product for each row in A and vector B
		t_ve blocksum = 0;
		//if nB> blockDim, split repeat the
		//for(int b = bBegin; (b < bEnd)&&((threadIdx.x+b) < bEnd); b += bStep ) {
		for(int b = bBegin; b < bEnd; b += bStep ) {

		//initialise Cs#include "project_comm.h"
			Cs[tx] = 0;
			__syncthreads();
			// compute scalar product
			if (( (gridIndex*gridDim.x+blockIdx.x)<aEnd)&&((b+tx) < bEnd)) {
				//Cs[threadIdx.x] = A[a + blockIdx.x ][b + threadIdx.x] * B[b + threadIdx.x ];
				//Cs[threadIdx.x] = A[(a + blockIdx.x)* nB+b + tx] * B[b + tx ];
				//30,Jan.2010
				Cs[threadIdx.x] = ASpalte(a + blockIdx.x,b + tx,mA) * B[b + tx ];
			}
			__syncthreads();

			if(tx == 0){
				//30.Nov.2009 fixeded for Cs summe
				int kEnd = bEnd-b;
				if(kEnd > VECTOR_BLOCK_SIZE)kEnd = VECTOR_BLOCK_SIZE;
				//Because I add Cs[0...k], if blockSize and Matrix does not fit, Parts of Cs[k] are not initialized as 0.

				for (int k = 1; k < kEnd; k++) Cs[0] += Cs[k];
				blocksum += Cs[0];
			}
			__syncthreads();
			/*
			int offset;
			offset = VECTOR_BLOCK_SIZE/2;
			while (offset > 0) {
				if(tx < offset) {
					Cs[tx] += Cs[tx + offset];
				}
				offset >>= 1;
				__syncthreads();
			}
			__syncthreads();
			if(threadIdx.x == 0)
			blocksum += Cs[0]; //??? blocksum = Cs[0];
		*/
		}//for b
		__syncthreads();

		if(threadIdx.x == 0) C[gridIndex*gridDim.x+blockIdx.x] = blocksum;
		__syncthreads();
		// summe all block, need test for mA bigger than one Grid
		//idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;

	}//for a


}

__host__ void dbg_matrixMul_checkresult(
                                          t_ve* C_in,
                                          t_ve* A_in,
                                          t_ve* B_in,
                                          t_mindex mA,
                                          t_mindex mB,
                                          char* debugname
                                        ) {
    hipError_t e;

    t_ve* C = (t_ve*) malloc( sizeof( t_ve* ) * mA );
    if (  C == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you C"); exit( -1 ); }

    t_ve* Co = (t_ve*) malloc( sizeof( t_ve* ) * mA );
    if (  Co == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you C"); exit( -1 ); }

    t_ve* A = (t_ve*) malloc( sizeof( t_ve* ) * mA  * mB );
    if (  A == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you A"); exit( -1 ); }

    t_ve* B = (t_ve*) malloc( sizeof( t_ve* ) * mB );
    if (  B == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you B"); exit( -1 ); }


    e = hipMemcpy( A, A_in, sizeof(t_ve) * mA  * mB , hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

    e = hipMemcpy( C, C_in, sizeof(t_ve) * mA, hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

    e = hipMemcpy( B, B_in, sizeof(t_ve) * mB, hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");



    for ( t_mindex cr = 0; cr < mA; cr++ ) {
       t_ve Celement = 0;
       for ( t_mindex br = 0; br < mB; br++ ) {
           t_mindex as = br;
           Celement += A[ cr + as * mA ] * B[ br ];
       }
       Co[cr] = Celement;
//       t_ve tolerance = abs( Celement / 100 );
       t_ve tolerance;
       if ( abs(Celement) > 1 ) {
            tolerance = abs(  Celement / 10 * mA ) ;
       }
       else {
          tolerance = 0.05 * mA;
       }
      //t_ve tolerance = Celement = 0;
       t_ve diff = Celement - C[cr];
       if ( abs( Celement - C[cr] ) > tolerance ) {

           printf( "\n Matmul '%s' not OK ( sum is C[%u]%f, should be %f", debugname , cr, C[cr], Celement  );
           for ( t_mindex i = 0; i < mB; i++ ) {
              printf("\n C[%u]=%f", i, C[i] );
           }
           printf( "\n" );

           t_ve cumm = 0;
           for ( t_mindex i = 0; i < mB; i++ ) {
              t_mindex ai = i * mA ;
              t_ve prod = B[i] * A[ai];
              cumm += prod;
              printf("\n B[%u] = %f A[%u]= %f  .>  control calculation:  a*b = %f   -> c = %f ", i, B[i], ai, A[ai] , prod, cumm );
           }
           printf( "\n" );

           for ( t_mindex s = 0; s < mB ; s++ ) {
               for ( t_mindex r = 0; r < mA ; r++ ) {
                   t_mindex i = s * mA + r;
                   printf("\n A(%u,%u) = A[%u]=%f", r+1, s+1 , i, A[i] );
               }
           }
           printf( "\n Matmul '%s' not OK ( sum is C[%u]%f, should be %f (tolerance %f, diff %f)", debugname , cr, C[cr], Celement,tolerance, diff);
           printf( "\n mA = %u; mB = %u \n ", mA, mB );
           exit(-1);
       }

    }

//    e = hipMemcpy( C_in, Co, sizeof(t_ve) * mA, hipMemcpyHostToDevice);
//    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

    free(A);
    free(B);
    free(C);
    free(Co);




}
