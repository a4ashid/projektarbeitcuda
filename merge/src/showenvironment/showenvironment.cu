#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"
#include "measurehelp.h"

int main()
{
    printf( "Build configuration: sizeof(t_ve) = %u \n", sizeof(t_ve));
    printf("\n detecting environment... \n");

    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);



        printf("\nDevice %d: \"%s\"\n \n", dev, deviceProp.name);
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
        printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
        printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    }

}
