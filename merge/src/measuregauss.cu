#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "measurehelp.h"
#include "kernels/gausskernel.h"

__host__ void eleminate ( t_ve* Ab, t_ve* x, t_mindex N ) {
    unsigned int i;   // columns
    unsigned int j;   // rows, equitations
    unsigned int k, max;
    t_ve t;

    for ( i = 1; i <= N ; i++ ) {


       max = i;
       for( j = i + 1; j <= N; j++ ) {
           if ( abs( Ab[ ab(j,i) ] ) > abs( Ab[ ab(max,i) ] )  ) {
              max = j;
           }
       }

       for ( k = i; k <= N + 1; k++ ) {
          t              = Ab[ ab(i,k) ];
          Ab[ ab(i,k)   ] = Ab[ ab(max,k) ];
          Ab[ ab(max,k) ] = t;
       }

       for ( j = i +1; j <= N ; j++ ) {
          for ( k = N + 1; k >= i ; k-- ) {
             Ab[ ab(j,k) ] -= Ab[ ab(i,k) ] * Ab[ ab(j,i) ] /  Ab[ ab(i,i) ];
          }
       }


      // substitute ...

        for (j = N; j >= 1; j-- ) {
            t_ve t = 0.0;
            for ( k = j + 1; k <= N; k++ ) {
                    t +=  Ab[ ab(j,k) ] * x[ k - 1 ];
            }
            x[ j - 1 ] = ( Ab[ ab(j,N+1) ] - t ) / Ab[ ab(j,j) ] ;
        }

    }
}

__host__ void dosolver (  t_mindex N_in ) {


    //printf("\n%u\t%f\t%f\t%f", N_in, gpudot_ms, cpudot_ms, gpudotwom_ms  );
    hipError_t e;
    size_t devsize = SIZE_VE * N_in * ( N_in + 2 )   ;

    pt_ve abx, x, abx_d, x_d;

    abx = ( pt_ve ) malloc( devsize );
    if (  abx == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you P.pElement"); exit( -1 ); }

    t_mindex N =  N_in;

    x = &abx[ N_in * ( N_in + 1 ) ];

    for ( t_mindex m = 1; m <= N_in; m++ ) {
        for ( t_mindex n = 1; n <= N_in + 1 ; n++ ) {
            abx[ ab( m, n ) ] =  ((t_ve) rand()) / RAND_MAX - 0.5;
        }
        x[m-1] = 0;
    }

    e = hipMalloc ( &abx_d, devsize );
    CUDA_UTIL_ERRORCHECK("hipMalloc");

    e = hipMemcpy( abx_d, abx, devsize, hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpy");

    x_d =  &abx_d[ N_in * ( N_in + 1 ) ];

    dim3 dimGridgauss( 1 );
    dim3 dimBlockgauss(512);

   float gpugauss_ms;

    {
           START_CUDA_TIMER

           device_gauss_solver<<<dimGridgauss,dimBlockgauss>>>( abx_d, N_in, x_d ); /* vec m is s+1 column of M - see memory allocation plan  */
           e = hipGetLastError();  CUDA_UTIL_ERRORCHECK("device_gauss_solver<<<dimGridgauss,dimBlockgauss>>>( M, s, c )");

           STOP_CUDA_TIMER( &gpugauss_ms )

    }

   float cpugauss_ms;

    {
        START_CUDA_TIMER
        eleminate ( abx, x, N_in );

        STOP_CUDA_TIMER( &cpugauss_ms )
    }
    printf("\n%u\t%f\t%f", N_in, gpugauss_ms, cpugauss_ms  );
}


int main( int argc, char *argv[] )
{
   printf("\n measure gauss");
   printf( "\n Build configuration: sizeof(t_ve) = %u \n", sizeof(t_ve));



    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);



        printf("\nDevice %d: \"%s\"\n \n", dev, deviceProp.name);
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
        printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
        printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    }

   t_mindex maxn = 10;

   if ( argc > 1 ) {
      maxn = atoi( argv[1] );
   }

    for ( t_mindex n = 3; n < maxn ; n++ ) {
        dosolver( n );
    }
}

