
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define CUDA_UTIL_ERRORCHECK(MSG)        if( e != hipSuccess ) \
        {\
            fprintf(stderr, "*** Error on CUDA operation '%s': '%s'*** \n\n", MSG, hipGetErrorString(e));\
            exit(-3);\
        }\

#ifndef PRJACUDADOUBLE
typedef float        t_ve; /* base type of Matrizes: 'float' or 'double' */
#endif

#ifdef PRJACUDADOUBLE
typedef double       t_ve; /* base type of Matrizes: 'float' or 'double' */
#endif

int N = 10;

__global__ void minikernel(  int N_in, t_ve* out  ) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N ) {
       out[i] = 1234;
    }
}


int  main () {
   printf("\n the precision- and compile-option checker \n");
   printf("\n sizeof(t_ve) = %u", sizeof(t_ve));


    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\n\n\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
        printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
    }

    t_ve* hostmem = (t_ve*) malloc( sizeof(t_ve) * N  );

    if ( hostmem == NULL ) { printf("sorry, can not allocate memory for you"); exit(-1); }

    t_ve* devmem;
    hipError_t e;

    e = hipMalloc ( &devmem , sizeof(t_ve) * N );
    CUDA_UTIL_ERRORCHECK("hipMalloc");

//    e = cudaMemset (devmem, 0, sizeof(t_ve) * N );
//    CUDA_UTIL_ERRORCHECK("cudaMalloc");

    dim3 dimGrid( 1 );
    dim3 dimBlock(512);

    minikernel<<<dimGrid,dimBlock>>>( N,  devmem );

    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("minikernel");

    e = hipMemcpy( hostmem, devmem, sizeof(t_ve) * N , hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

    for  ( int i = 0; i < N; i++ ) {
        printf("\nout[%u] = %f", i, hostmem[i]  );
    }

    free(hostmem);
    e = hipFree( devmem);
    CUDA_UTIL_ERRORCHECK("hipMalloc");
}

