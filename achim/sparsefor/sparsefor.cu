#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>


#include "projektcuda.h"


__global__ void kernel_sparsemul( int N, int row_step ) {


	int i = 0;

	if ( blockIdx.x == gridDim.x - 1 ) {
		printf("\n %u special case", threadIdx.x );
	}
	else {
		for ( int rs = 0; rs < row_step ; rs++ ) {
			int row = rs + row_step * blockIdx.x ;
			//printf ( "\n row %u rowblock %u block %u ", row, rs, blockIdx.x );

			// start calculation of row * b-vector here
	    }
	}


}

int main()
{
    t_ve* hostmem;

    printf("\n measure CUBLAS dotmul\n");

    int N = 2000000;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int gridsize;

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        gridsize = deviceProp.multiProcessorCount;

        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
        printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    }

    int row_step = N / gridsize + 1;

    int n_calc = row_step * gridsize;

    printf("\n Initial N %d ", N );

    printf("\n using gridsize %d ", gridsize );
    printf("\n using row_step %d ", row_step );
    printf("\n using N_calc %d "  , n_calc );
    printf("\n \n");

    dim3 dimGrid ( gridsize );

    dim3 dimBlock( 8 );

    kernel_sparsemul<<<dimGrid,dimBlock>>>( N , row_step );

        printf("\n\n Initial N %d ", N );

	    printf("\n using gridsize %d ", gridsize );
	    printf("\n using row_step %d ", row_step );
	    printf("\n using N_calc %d "  , n_calc );
	    printf("\n \n");


}