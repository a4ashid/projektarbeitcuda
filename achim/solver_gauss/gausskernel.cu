#include "hip/hip_runtime.h"

#include "projektcuda.h"

#include "gausskernel.h"

__global__ void device_gauss_solver( t_ve* p_Ab, unsigned int N, t_ve* p_x )
{


    __shared__ unsigned int i;
    __shared__ unsigned int max;

    __shared__ t_ve Ab[ GAUSSNMAX * ( GAUSSNMAX + 1) ];
     t_ve x[ GAUSSNMAX ];

    unsigned int tidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int n;

    t_ve t ;

    if ( tidx  <  N * (N+1) ) {
         Ab[tidx] = p_Ab[tidx];
    }

    if ( tidx == 0 ) { i = 1; }

    __syncthreads();

    while ( i <= N ) {                  /* for ( i = 1; i <= N ; i++ ) */
        if ( tidx == 0 ) {
            unsigned int j;
            max = i;
            for( j = i + 1; j <= N; j++ ) {
                if ( abs( Ab[ a(j,i) ] ) > abs( Ab[ a(max,i) ] )  ) {
                    max = j;
                }
            }
       }
       __syncthreads();


//       if ( threadIdx.y == 0 ) {
           unsigned int k = tidx + 1;
         if ( tidx == 0 ) { /* does not work in parallel on device (don't not know why :-/ ) */
//           if ( ( k >= i ) && ( k <= N + 1 ) ) {
            for ( k = i; k <= N + 1; k++ ) {
               t              = Ab[ a(i  ,k) ];
               Ab[ a(i,k)   ] = Ab[ a(max,k) ];
               Ab[ a(max,k) ] = t;
           }
        }

      __syncthreads();

      {
          unsigned int j = threadIdx.x + 1;
          if (  ( j >= i +1 ) && ( j <= N ) && threadIdx.y == 0 ) {       /*   for ( j = i +1; j <= N ; j++ ) */
              unsigned int  k ;
              for ( k = N + 1; k >= i ; k-- ) {
                 Ab[ a(j,k) ] -= Ab[ a(i,k) ] * Ab[ a(j,i) ] /  Ab[ a(i, i) ];
              }
           }
       }
       __syncthreads();
       if ( tidx == 0 ) { i++; }
    }
    __syncthreads();

    if ( tidx == 0 ) {

        /* the substitute part */
        unsigned int j,k;
        for (j = N; j >= 1; j-- ) {
            t_ve t = 0.0;
            for ( k = j + 1; k <= N; k++ ) {
                    t +=  Ab[ a(j,k) ] * x[ k - 1 ];
            }
            x[ j - 1 ] = ( Ab[ a(j,N+1) ] - t ) / Ab[ a(j,j) ] ;
        }
        /* copy result back to global memory */

        for  ( n = 0; n <  N * (N+1); n++ ) {
            p_Ab[n] = Ab[n];
        }
        for  ( n = 0; n < N; n++ ) {
            p_x[n] = x[n];
        }
    }
   __syncthreads();
}
