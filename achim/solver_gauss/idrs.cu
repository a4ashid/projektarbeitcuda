

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"
#include "idrs.h"

__host__ void push_vector_2_device( t_ve* V,  t_ve** pV_d, unsigned int L ) {
	hipError_t e;

	t_ve* Vd;

    e = hipMalloc ((void **) pV_d, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMemcpy(  *pV_d, V, sizeof(t_ve) * L , hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
}

__host__ void malloc_vector_on_device(  t_ve** pV_d, unsigned int L ) {
	hipError_t e;

    e = hipMalloc ((void **) pV_d, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    hipMemset ( *pV_d, 0, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on memset 0: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}




__host__ void idrs(
                     t_ve* A_h,
                     t_ve* b_h,
                     unsigned int s,
                     t_ve  tol,
                     unsigned int maxit,
                     t_ve* x0_h,

                     unsigned int N,

                     t_ve* x_h,  /* output vector */
                     t_ve* resvec_h,
                     unsigned int* piter
                  ) {


   t_ve* A;    /* A in device , A_h is Host */
   t_ve* b;
   t_ve* x;

   t_ve* dR;
   t_ve* dX;

   printf("\n empty IDRS, malloc \n");

   push_vector_2_device( A_h, &A, N * N );
   push_vector_2_device( b_h, &b, N );
   push_vector_2_device( x_h, &x, N );

   /* m20:  dR = zeros(N,s); dX = zeros(N,s); */
   malloc_vector_on_device( &dR, N * s );
   malloc_vector_on_device( &dX, N * s );

}

