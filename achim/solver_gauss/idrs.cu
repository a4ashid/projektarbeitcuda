

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"
#include "idrs.h"
#include "dotMul_cuda_gpu.h"

__host__ void push_vector_2_device( t_ve* V,  t_ve** pV_d, unsigned int L ) {
	hipError_t e;

	t_ve* Vd;

    e = hipMalloc ((void **) pV_d, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMemcpy(  *pV_d, V, sizeof(t_ve) * L , hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
}

__host__ void malloc_vector_on_device(  t_ve** pV_d, unsigned int L ) {
	hipError_t e;

    e = hipMalloc ((void **) pV_d, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    hipMemset ( *pV_d, 0, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on memset 0: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}




__host__ void idrs(
                     t_ve* A_h,
                     t_ve* b_h,
                     unsigned int s,
                     t_ve  tol,
                     unsigned int maxit,
                     t_ve* x0_h,

                     unsigned int N,

                     t_ve* x_h,  /* output vector */
                     t_ve* resvec_h,
                     unsigned int* piter
                  ) {


   t_ve* A;    /* A in device , A_h is Host */
   t_ve* b;
   t_ve* x;

   t_ve* dR;
   t_ve* dX;
   t_ve* r;
   t_ve* v;
   t_ve*  om1;
   t_ve*  om2;

   int block_size = 512;

   dim3 dimBlock(block_size);
   dim3 dimGrid ( N / block_size );

   printf("\n empty IDRS, malloc \n");

   push_vector_2_device( A_h, &A, N * N );
   push_vector_2_device( b_h, &b, N );
   push_vector_2_device( x_h, &x, N );

   /* m20:  dR = zeros(N,s); dX = zeros(N,s); */
   malloc_vector_on_device( &dR, N * s );
   malloc_vector_on_device( &dX, N * s );
   malloc_vector_on_device( &r , N  );
   malloc_vector_on_device( &v , N  );
   malloc_vector_on_device( &om1, N / block_size );
   malloc_vector_on_device( &om2, N / block_size );

   hipError_t e;
   for ( int k = 1; k < s; k ++ )
   {
	    /*  v = A*r;   */
	   printf("\n %u this loop is currently not implemented");


	   device_dotMul<<<dimGrid,dimBlock>>>( v, r, om1, N );
	   e = hipGetLastError();
	   if( e != hipSuccess )
	   {
	       fprintf(stderr, "CUDA Error on add_arrays_gpu: '%s' \n", hipGetErrorString(e));
	       exit(-3);
       }
	   device_dotMul<<<dimGrid,dimBlock>>>( v, v, om2, N );
	   e = hipGetLastError();
	   if( e != hipSuccess )
	   {
	       fprintf(stderr, "CUDA Error on add_arrays_gpu: '%s' \n", hipGetErrorString(e));
	       exit(-3);

       }

   }

}

