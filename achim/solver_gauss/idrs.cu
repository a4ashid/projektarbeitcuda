

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"
#include "idrs.h"

__host__ void idrs(
                     t_ve* A,
                     t_ve* b,
                     t_ve* s,
                     t_ve  tol,
                     unsigned int maxit,
                     t_ve* x0,

                     unsigned int N,

                     t_ve* x,  /* output vector */
                     t_ve* resvec,
                     unsigned int* piter
                  ) {

   int bla;
   bla = 0;
   t_ve* A_d; /* A in device */

   printf("\n empty IDRS, malloc \n");

    hipError_t e;
    e = hipMalloc ((void **) &A_d, sizeof(t_ve) * N * N );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}

