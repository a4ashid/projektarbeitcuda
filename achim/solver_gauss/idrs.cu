

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"
#include "idrs.h"

__host__ void push_vector_2_device( t_ve* V,  t_ve** pV_d, unsigned int L ) {
	hipError_t e;

    e = hipMalloc ((void **) pV_d, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMemcpy(  V, *pV_d, sizeof(t_ve) * L , hipMemcpyDeviceToHost);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
}

__host__ void idrs(
                     t_ve* A_h,
                     t_ve* b_h,
                     t_ve* s_h,
                     t_ve  tol,
                     unsigned int maxit,
                     t_ve* x0_h,

                     unsigned int N,

                     t_ve* x_h,  /* output vector */
                     t_ve* resvec_h,
                     unsigned int* piter
                  ) {


   t_ve* A;    /* A in device , A_h is Host */
   t_ve* b;
   t_ve* x;

   printf("\n empty IDRS, malloc \n");

   push_vector_2_device( A_h, &A, N * N );
   push_vector_2_device( b_h, &b, N );
   push_vector_2_device( x_h, &x, N );

}

