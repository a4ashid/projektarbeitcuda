#include <stdlib.h>
#include <stdio.h>

#include "hipblas.h"

#include "projektcuda.h"

#define NMAX 1000


__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    t_ve* v =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( v == NULL) {
           fprintf(stderr, "sorry, can not allocate memory for you");
           exit( -1 );
    }
    *M = v;
}
int main()
{
    t_ve* hostmem;

    printf("\n measure CUBLAS dotmul\n");

    malloc_N( NMAX * 2 , &hostmem );

    t_ve* in1 = &hostmem[0];
    t_ve* in2 = &hostmem[NMAX];

    for ( int i = 0; i < NMAX; i++ ) {
       in1[i] = 1;
       in2[i] = 2;
    }
    hipblasStatus_t ce;
    ce = cublasInit();
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS error>>>"); exit( -3); }

    t_ve* d_in1;
    ce = cublasAlloc( NMAX, sizeof(t_ve), (void**)&d_in1 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc>>>"); exit( -3); }

    t_ve* d_in2;
    ce = cublasAlloc( NMAX, sizeof(t_ve), (void**)&d_in2 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc>>>"); exit( -3); }

    ce = hipblasSetVector( NMAX, sizeof(t_ve), in1, 1, d_in1, 1 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc hipblasSetVector>>>"); exit( -3); }

    ce = hipblasSetVector( NMAX, sizeof(t_ve), in2, 1, d_in2, 1 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc hipblasSetVector>>>"); exit( -3); }

    float dm = hipblasSdot( NMAX, d_in1, 1, d_in2, 1 );

    printf("\nresult dotmul: %f", dm );

    ce = cublasFree( d_in2 );
    ce = cublasFree( d_in1 );
    ce = cublasShutdown();
}
