#include <stdlib.h>
#include <stdio.h>

#include "hipblas.h"

#include "projektcuda.h"
#include "measurehelp.h"

#include "dotMul_cpu.h"


__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    t_ve* v =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( v == NULL) {
           fprintf(stderr, "sorry, can not allocate memory for you");
           exit( -1 );
    }
    *M = v;
}
int main()
{
    t_ve* hostmem;

    printf("\n measure CUBLAS dotmul\n");

    hipblasStatus_t ce;
    ce = cublasInit();

    for ( int N = 1; N < 10000000; N *= 10 ) {

    malloc_N( N * 3 , &hostmem );

    t_ve* in1 = &hostmem[0];
    t_ve* in2 = &hostmem[N];
    t_ve* out = &hostmem[N*2];

    for ( int i = 0; i < N; i++ ) {
       in1[i] = 1;
       in2[i] = 2;
    }

    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS error>>>"); exit( -3); }

    t_ve* d_in1;
    ce = cublasAlloc( N, sizeof(t_ve), (void**)&d_in1 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc>>>"); exit( -3); }

    t_ve* d_in2;
    ce = cublasAlloc( N, sizeof(t_ve), (void**)&d_in2 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc>>>"); exit( -3); }

    ce = hipblasSetVector( N, sizeof(t_ve), in1, 1, d_in1, 1 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc hipblasSetVector>>>"); exit( -3); }

    ce = hipblasSetVector( N, sizeof(t_ve), in2, 1, d_in2, 1 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc hipblasSetVector>>>"); exit( -3); }

    hipError_t e;
    float cublas_ms, cpu_ms;
    float dm;
    {
        START_CUDA_TIMER
        for ( int i = 0; i < 20; i++ ) {
            dm = hipblasSdot( N, d_in1, 1, d_in2, 1 );
        }
        STOP_CUDA_TIMER( &cublas_ms )
    }

    {
        START_CUDA_TIMER
        for ( int i = 0; i < 20; i++ ) {
            dotMul_cpu(in1, in2, out, N );
        }
        STOP_CUDA_TIMER( &cpu_ms )
    }



    printf( "\n N = %u, time spent cublas: %f ms time CPU %f", N ,cublas_ms, cpu_ms);

    ce = cublasFree( d_in2 );
    ce = cublasFree( d_in1 );

    free( hostmem );

    }
    ce = cublasShutdown();
}
