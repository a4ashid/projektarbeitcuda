#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "hipblas.h"

#include "projektcuda.h"
#include "measurehelp.h"

#include "dotMul_cpu.h"


__global__ void kernel_dotmul( t_ve *in1, t_ve *in2, t_ve *out ) {
    __shared__ t_ve Vs [DEF_BLOCKSIZE];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    Vs[threadIdx.x] = in1[idx] * in2[idx];


    __syncthreads();
    if ( threadIdx.x < 256 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  + 256 ]; }
    __syncthreads();

    if ( threadIdx.x < 128 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  + 128 ];}
    __syncthreads();

    if ( threadIdx.x <  64 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  +  64 ];}
    __syncthreads();


    if ( threadIdx.x <  32 ) {
        Vs[threadIdx.x] += Vs[ threadIdx.x + 32 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x + 16 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  8 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  4 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  2 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  1 ];

        if ( threadIdx.x == 0 ) {
            out[blockIdx.x] =  Vs[0]  ;
        }
    }
}



__host__ void own_dotmul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N) {

     int gridsize =  ( N / (DEF_BLOCKSIZE) ) + 1;

     //printf("\n N is %u, gridsize %u", N, gridsize);

     dim3 dimGrid ( gridsize );
     dim3 dimBlock(DEF_BLOCKSIZE);

     hipError_t e;


     kernel_dotmul<<<dimGrid,dimBlock>>>( in1, in2, out );
     e = hipGetLastError();
     CUDA_UTIL_ERRORCHECK("summup_kernel_kernel_dotmul");

     int bla = 0;
}

__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    t_ve* v =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( v == NULL) {
           fprintf(stderr, "sorry, can not allocate memory for you");
           exit( -1 );
    }
    *M = v;
}
int main()
{
    t_ve* hostmem;

    printf("\n measure CUBLAS dotmul\n");


    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
        printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    }

    hipblasStatus_t ce;

    ce = cublasInit();

    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<error on cublasInit>>>"); exit( -3); }

    for ( int N = 10; N < 100000000; N *= 10 ) {

    malloc_N( ( N + 512) * 3 , &hostmem );

    t_ve* in1 = &hostmem[0];
    t_ve* in2 = &hostmem[N + 512];
    t_ve* out = &hostmem[ (N + 512 ) *2  ];

    for ( int i = 0; i < N; i++ ) {
       in1[i] = 1;
       in2[i] = 2;
    }



    t_ve* d_in1;
    ce = cublasAlloc( N, sizeof(t_ve), (void**)&d_in1 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc>>>"); exit( -3); }

    t_ve* d_in2;
    ce = cublasAlloc( N, sizeof(t_ve), (void**)&d_in2 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc>>>"); exit( -3); }

    ce = hipblasSetVector( N, sizeof(t_ve), in1, 1, d_in1, 1 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc hipblasSetVector>>>"); exit( -3); }

    ce = hipblasSetVector( N, sizeof(t_ve), in2, 1, d_in2, 1 );
    if ( ce != HIPBLAS_STATUS_SUCCESS ) { printf("<<<CUBLAS erroralloc hipblasSetVector>>>"); exit( -3); }

    hipError_t e;
    float cublas_ms, cpu_ms, gpu_ms;
    float dm;
    {
        START_CUDA_TIMER
        for ( int i = 0; i < 2; i++ ) {
            dm = hipblasSdot( N, d_in1, 1, d_in2, 1 );
        }
        STOP_CUDA_TIMER( &cublas_ms )
    }

    {
        START_CUDA_TIMER
        for ( int i = 0; i < 2; i++ ) {
            dotMul_cpu(in1, in2, out, N );
        }
        STOP_CUDA_TIMER( &cpu_ms )
    }

    ce = cublasFree( d_in2 );
    ce = cublasFree( d_in1 );

/* -------------------------------------------------------- */
    {
        t_ve* devicemem;
        e = hipMalloc ((void **) &devicemem, sizeof(t_ve) * ( N + 512 ) * 3 );
        CUDA_UTIL_ERRORCHECK("hipMalloc")

        e = hipMemcpy(  devicemem, hostmem, sizeof(t_ve) * ( N + 512) * 2 , hipMemcpyHostToDevice);
        CUDA_UTIL_ERRORCHECK("hipMemcpy")

        t_ve* din1 = &devicemem[0];
        t_ve* din2 = &devicemem[N + 512 ];
        t_ve* dout = &devicemem[(N + 512) * 2];

        {
            START_CUDA_TIMER
            for ( int i = 0; i < 2; i++ ) {
                own_dotmul(din1, din2, dout, N );
            }
            STOP_CUDA_TIMER( &gpu_ms )
        }

        e = hipFree(devicemem);
        CUDA_UTIL_ERRORCHECK("hipFree")
    }

/* -------------------------------------------------------- */

    printf( "\n N = %u, time spent cublas: %f ms time CPU %f ms -- GPU %f ms", N ,cublas_ms, cpu_ms, gpu_ms);



    free( hostmem );

    }
    ce = cublasShutdown();
}
