#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"
#include "measurehelp.h"

#define ITERATIONS 5

__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    t_ve* v =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( v == NULL) {
           fprintf(stderr, "sorry, can not allocate memory for you");
           exit( -1 );
    }
    *M = v;
}

/* ----------------------------------------------------------------- */
__global__ void summup_kernel_triangle( t_ve *in, t_ve *out, unsigned int N ) {
    __shared__ t_ve Vs [DEF_BLOCKSIZE];

    Vs[threadIdx.x] = in[threadIdx.x];
    __syncthreads();

/*
    unsigned short t = 1;
    for ( t = 1 << BLOCK_EXP - 1; t > 1; t >>= 1 ) {
        __syncthreads();
        if ( threadIdx.x < t ) {
            Vs[threadIdx.x] += Vs[ threadIdx.x + t ];
        }
    }
    __syncthreads();
    if ( threadIdx.x == 0 ) { out[0] = Vs[0] + Vs[t]; }
*/

    int offset = 1;
    for (  int i = 1; i < BLOCK_EXP ; i++ ) {
         int old = offset;
        offset <<= 1;
        if ( threadIdx.x % offset == 0 ) {

           Vs[threadIdx.x] += Vs[ threadIdx.x + old ];
        }
        __syncthreads();
    }
    if ( threadIdx.x == 0 ) { out[0] = Vs[0] + Vs[offset]; }

}
/* ----------------------------------------------------------------- */
__global__ void summup_kernel_triangle_warpop( t_ve *in, t_ve *out, unsigned int N ) {
    __shared__ t_ve Vs [DEF_BLOCKSIZE];


    Vs[threadIdx.x] = in[threadIdx.x];


    __syncthreads();
    if ( threadIdx.x < 256 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  | 256 ]; }
    //if ( threadIdx.x < 256 ) { Vs[threadIdx.x] = in[threadIdx.x] + in[threadIdx.x + 256]; }
    __syncthreads();

    if ( threadIdx.x < 128 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  | 128 ];}
    __syncthreads();

    if ( threadIdx.x <  64 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  |  64 ];}
    __syncthreads();

    if ( threadIdx.x <  32 ) { Vs[threadIdx.x ] += Vs[ threadIdx.x  |  32 ];}
    __syncthreads();

    if ( threadIdx.x <  16 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  | 16 ]; }
    __syncthreads();

    if ( threadIdx.x <   8 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  | 8 ]; }
    __syncthreads();

//    if ( threadIdx.x <   4 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  | 4 ]; }
//    __syncthreads();

//    if ( threadIdx.x <   2 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  | 2 ]; }
//    __syncthreads();


    if ( threadIdx.x == 0 ) {
        t_ve sum = 0;
        for ( short int i = 0; i < 8; i++ ) {
            sum += Vs[i];
        }
        out[0] =  sum;
    }

}
/* ----------------------------------------------------------------- */

__global__ void summup_kernel_for( t_ve *in, t_ve *out, unsigned int N ) {
    __shared__ t_ve v [DEF_BLOCKSIZE];
    t_ve blocksum = 0;

    v[threadIdx.x] = in[threadIdx.x];
    __syncthreads();

    if ( threadIdx.x == 0 ) {
        for ( int i = 0; i < N; i++ ) {
           blocksum += v[i];
        }
        out[0] = blocksum;
    }
}
/* ----------------------------------------------------------------- */

__host__ void summup_cpu( t_ve *in, t_ve *out, unsigned int N ) {
    t_ve sum = 0;
    for ( int i = 0; i < N; i++ ) {
       sum += in[i];
    }
    out[0] = sum;
}
/* ----------------------------------------------------------------- */

int main()
{
    hipError_t e;



    printf("\n triangle adder,  \nrunning with %u iterations per kernel call \n\n", ITERATIONS);

    t_ve* in;

    t_ve  out;

    t_ve  outgpu_for;
    t_ve  outgpu_triangle;
    t_ve*  outgpu_d;

/*
    short offset =  1;
    for ( short i = 0; i < BLOCK_EXP ; i++ ) {
        short old = offset;
        offset <<= 1;
        printf("\n step: %u. offset %u old %u", i, offset, old );
    }
*/
    short offset = 1;
    for ( short t = 1 << BLOCK_EXP - 1; t > 1; t >>= 1 ) {
        printf("\n threadlimit %u, offset %u", t, offset );
        offset <<= 1;
    }

    malloc_N( DEF_BLOCKSIZE , &in );

    for ( int i = 0; i < DEF_BLOCKSIZE; i++ ) {
       in[i] = 10;
    }

    float cpu_ms;

    {
       START_CUDA_TIMER
       for  ( int i = 0; i < ITERATIONS; i++ ) {
           summup_cpu( in, &out, DEF_BLOCKSIZE );
       }
       STOP_CUDA_TIMER( &cpu_ms )

    }
    //printf("\n\n got from CPU calc: %f", out);

/*  --------------------------------------------------  */
    t_ve* devicemem;
    e = hipMalloc ((void **) &devicemem, sizeof(t_ve) * (DEF_BLOCKSIZE + 1) );
    CUDA_UTIL_ERRORCHECK("hipMalloc &devicemem");

    outgpu_d = &devicemem[DEF_BLOCKSIZE];

    e = hipMemcpy(  devicemem, in, sizeof(t_ve) * (DEF_BLOCKSIZE), hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpy v1_d");

    dim3 dimGrid ( 1 );
    dim3 dimBlock(DEF_BLOCKSIZE);

        /* "warming up", not measured */
    for  ( int i = 0; i < 20; i++ ) {
            summup_kernel_triangle<<<dimGrid,dimBlock>>>( devicemem, outgpu_d, DEF_BLOCKSIZE);
            e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("summup_kernel_triangle");

            summup_kernel_triangle_warpop<<<dimGrid,dimBlock>>>( devicemem, outgpu_d, DEF_BLOCKSIZE);
            e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("summup_kernel_triangle");

    }

    float kernelfor_ms, kerneltriangle_ms, kerneltrianglewarpop_ms;

    {
        START_CUDA_TIMER
        for  ( int i = 0; i < ITERATIONS; i++ ) {
            summup_kernel_for<<<dimGrid,dimBlock>>>( devicemem, outgpu_d, DEF_BLOCKSIZE);
            e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("summup_kernel_for");
        }
        STOP_CUDA_TIMER( &kernelfor_ms )
    }
    e = hipMemcpy( &outgpu_for, outgpu_d, sizeof(t_ve) , hipMemcpyDeviceToHost);

    CUDA_UTIL_ERRORCHECK("&outgpu_for, outgpu_dd");
    //printf("\n\n got from GPU for: %f", outgpu_for);
    printf("\n>>> GPU 'for' runtime: %f ms", kernelfor_ms / ITERATIONS );

    {
        START_CUDA_TIMER
        for  ( int i = 0; i < ITERATIONS; i++ ) {
            summup_kernel_triangle<<<dimGrid,dimBlock>>>( devicemem, outgpu_d, DEF_BLOCKSIZE);
            e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("summup_kernel_triangle");
        }
        STOP_CUDA_TIMER( &kerneltriangle_ms )
    }
    e = hipMemcpy( &outgpu_triangle, outgpu_d, sizeof(t_ve) , hipMemcpyDeviceToHost);

    CUDA_UTIL_ERRORCHECK("&outgpu_for, outgpu_dd");
    printf("\n\n got from GPU triangle: %f", outgpu_triangle );
    {
        START_CUDA_TIMER
        for  ( int i = 0; i < ITERATIONS; i++ ) {
            summup_kernel_triangle_warpop<<<dimGrid,dimBlock>>>( devicemem, outgpu_d, DEF_BLOCKSIZE);
            e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("summup_kernel_triangle");
        }
        STOP_CUDA_TIMER( &kerneltrianglewarpop_ms )
    }

    e = hipMemcpy( &outgpu_triangle, outgpu_d, sizeof(t_ve) , hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK("&outgpu_triangle, outgpu_dd");
    printf("\n\n got from GPU triangleop: %f", outgpu_triangle );
    printf("\n>>> GPU 'triangle' runtime: %f ms", kerneltriangle_ms / ITERATIONS );
    printf("\n>>> GPU 'triangleop' runtime: %f ms", kerneltrianglewarpop_ms  / ITERATIONS );

    printf("\n\n runtime of 'triangle' is %f percent of 'for(...)' runtime \n\n", 100 / kernelfor_ms * kerneltriangle_ms );
    printf("\n\n runtime of 'trianglewarpop' is %f percent of 'for(...)' runtime \n\n", 100 / kernelfor_ms * kerneltrianglewarpop_ms );

    e = hipFree(devicemem);
    CUDA_UTIL_ERRORCHECK("hipFree")

    printf("\n runtime on CPU: %f ms \n", cpu_ms / ITERATIONS  );

   printf("\n\n runtime of 'triangleop' is %f percent of 'cpu' runtime \n\n", 100 / cpu_ms * kerneltrianglewarpop_ms );

/*  --------------------------------------------------  */


}

