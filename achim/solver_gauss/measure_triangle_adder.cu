#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    t_ve* v =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( v == NULL) {
           fprintf(stderr, "sorry, can not allocate memory for you");
           exit( -1 );
    }
    *M = v;
}

/* ----------------------------------------------------------------- */
__global__ void summup_kernel_triangle( t_ve *in, t_ve *out, unsigned int N ) {
    __shared__ t_ve Vs [DEF_BLOCKSIZE];
    Vs[threadIdx.x] = in[threadIdx.x];
    __syncthreads();

    short offset = 1;
    for ( short i = 0; i < BLOCK_EXP ; i++ ) {
        short old = offset;
        offset <<= 1;
        if ( threadIdx.x % offset == 0 ) {
           Vs[threadIdx.x] += Vs[ threadIdx.x + old ];
        }
        __syncthreads();
    }

    if ( threadIdx.x == 0 ) {
        out[0] = Vs[0];
    }
}
/* ----------------------------------------------------------------- */

__global__ void summup_kernel_for( t_ve *in, t_ve *out, unsigned int N ) {
    __shared__ t_ve v [DEF_BLOCKSIZE];
    t_ve blocksum = 0;
    v[threadIdx.x] = in[threadIdx.x];

    __syncthreads();

    if ( threadIdx.x == 0 ) {
        for ( int i = 0; i < N; i++ ) {
           blocksum += v[i];
        }
        out[0] = blocksum;
    }
}
/* ----------------------------------------------------------------- */

__host__ void summup_cpu( t_ve *in, t_ve *out, unsigned int N ) {
    t_ve sum = 0;
    for ( int i = 0; i < N; i++ ) {
       sum += in[i];
    }
    out[0] = sum;
}
/* ----------------------------------------------------------------- */

int main()
{
    hipError_t e;
    printf("\n triangle adder \n");

    t_ve* in;

    t_ve  out;

    t_ve  outgpu_for;
    t_ve  outgpu_triangle;
    t_ve*  outgpu_d;

    short offset =  1;
    for ( short i = 0; i < BLOCK_EXP ; i++ ) {
        short old = offset;
        offset <<= 1;
        printf("\n step: %u. offset %u old %u", i, offset, old );
    }
    malloc_N( DEF_BLOCKSIZE , &in );

    for ( int i = 0; i < DEF_BLOCKSIZE; i++ ) {
       in[i] = 10;
    }

    summup_cpu( in, &out, DEF_BLOCKSIZE );
    printf("\n\n got from CPU calc: %f", out);

/*  --------------------------------------------------  */
    t_ve* devicemem;
    e = hipMalloc ((void **) &devicemem, sizeof(t_ve) * (DEF_BLOCKSIZE + 1) );
    CUDA_UTIL_ERRORCHECK("hipMalloc &devicemem");

    outgpu_d = &devicemem[DEF_BLOCKSIZE];

    e = hipMemcpy(  devicemem, in, sizeof(t_ve) * (DEF_BLOCKSIZE), hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpy v1_d");

    dim3 dimGrid ( 1 );
    dim3 dimBlock(DEF_BLOCKSIZE);

    summup_kernel_for<<<dimGrid,dimBlock>>>( in, outgpu_d, DEF_BLOCKSIZE);
    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("summup_kernel_for");
    e = hipMemcpy( &outgpu_for, outgpu_d, sizeof(t_ve) , hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK("&outgpu_for, outgpu_dd");
    printf("\n\n got from GPU for: %f", outgpu_for);

    summup_kernel_triangle<<<dimGrid,dimBlock>>>( in, outgpu_d, DEF_BLOCKSIZE);
    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("summup_kernel_triangle");
    e = hipMemcpy( &outgpu_triangle, outgpu_d, sizeof(t_ve) , hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK("&outgpu_for, outgpu_dd");
    printf("\n\n got from GPU triangle: %f", outgpu_triangle);

    e = hipFree(devicemem);
    CUDA_UTIL_ERRORCHECK("hipFree")
/*  --------------------------------------------------  */


}

