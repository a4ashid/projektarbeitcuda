#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include <time.h>

#include "dotMul_cuda_gpu.h"
#include "dotMul_cpu.h"
#include "addvector_cpu.h"

#include <time.h>

#define N_PROBLEM 10000
#define ITERSTEPS 2

__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    t_ve* v =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( v == NULL) {
	       fprintf(stderr, "sorry, can not allocate memory for you");
	       exit( -1 );
    }
    *M = v;
}

//    hipMemset ( *pV_d, 0, sizeof(t_ve) * L );

int main()
{
    pt_ve   out, out_d, hostmem;
    clock_t startclocks, endclock, payoffstart, payoffend, startclockscpu, endclockscpu;
    hipError_t e;

    hipEvent_t start_host,stop_host;
    float et;
    float et_gpu;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

     e = hipEventCreate( &start_host );
     CUDA_UTIL_ERRORCHECK("hipEventCreate");
     e = hipEventCreate( &stop_host );
     CUDA_UTIL_ERRORCHECK("hipEventCreate");

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
        printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    }

    int block_size = DEF_BLOCKSIZE ;

    printf("working with blocksize %u \n", block_size );

    dim3 dimBlock(block_size);

    printf("\n measureing operation DOTMUL with N = %u, ITERSTEPS %u", N_PROBLEM, ITERSTEPS );
    //malloc_N( N_PROBLEM * 3 , &v1 );

    for ( unsigned int N = 1; N <= N_PROBLEM; N *= 10 ) {


		malloc_N( N * 3, &hostmem );

        t_ve*       devivemem;
		//t_ve* A   = &hostmem[0];
		//t_ve* b   = &hostmem[ N * N ];
		//t_ve* out = &hostmem[ N * N + N];



        payoffstart = clock();

        e = hipMalloc ((void **) &devivemem, sizeof(t_ve) * N * 3 );
        CUDA_UTIL_ERRORCHECK("hipMalloc &devivemem");

		t_ve* in_1   = &devivemem[0];
		//t_ve* b_d   = &devivemem[ N * N ];
		//t_ve* out_d = &devivemem[ N * N + N];

        e = hipMemcpy(  devivemem, hostmem, sizeof(t_ve) * N * 2 , hipMemcpyHostToDevice);
        CUDA_UTIL_ERRORCHECK("hipMemcpy v1_d");

        //e = hipMemset ( out_d, 0, sizeof(t_ve) * (N + 1) );
        CUDA_UTIL_ERRORCHECK("hipMemset ( out_d, 0)");

        payoffend = clock();
        startclocks = clock( );

        dim3 dimGrid ( N / block_size + 1 );

        //for ( unsigned int i = 0; i < ITERSTEPS; i++ ) {
        {
            hipEvent_t start,stop;
            e = hipEventCreate( &start );
            CUDA_UTIL_ERRORCHECK("hipEventCreate");
            e = hipEventCreate( &stop );
            CUDA_UTIL_ERRORCHECK("hipEventCreate");
            e= hipEventRecord(start,0);
            CUDA_UTIL_ERRORCHECK("hipEventRecord");

            //device_dotMul<<<dimGrid,dimBlock>>>(v1_d, v2_d, out_d, N );
            //matrixMul_kernel<<<dimGrid,dimBlock>>>( out_d,  A_d, b_d, N, N);

        	e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("Kernel matrixMul_kernel")

            e= hipEventRecord(stop,0 );
            CUDA_UTIL_ERRORCHECK("hipEventRecord");
            e = hipEventSynchronize(stop);
            CUDA_UTIL_ERRORCHECK("hipEventSynchronize");
            e = hipEventElapsedTime( &et_gpu, start, stop );
        }
        //}
        endclock = clock( );


        /* ------------------------------------------------------------ */
		startclockscpu = clock( );

            e= hipEventRecord(start_host,0);
            CUDA_UTIL_ERRORCHECK("hipEventRecord");


		    //matrixMul_cpu( out,  A, b, N, N);


            e= hipEventRecord(stop_host,0 );
            CUDA_UTIL_ERRORCHECK("hipEventRecord");
            e = hipEventSynchronize(stop_host);
            CUDA_UTIL_ERRORCHECK("hipEventSynchronize");
            e = hipEventElapsedTime( &et, start_host, stop_host );
		//}
        endclockscpu   = clock( );
        /* ------------------------------------------------------------ */

        printf( "\n ----------------------------------------------------- \n N = %u, ITER = %u", N, ITERSTEPS );
	    printf( "\n GPU: %f seconds,  clocks: %u : CLOCKS_PER_SEC %u \n", ( (float) ( endclock - startclocks)) / CLOCKS_PER_SEC / ITERSTEPS, endclock - startclocks, CLOCKS_PER_SEC );
	    printf( "\n CPU: %f seconds,  clocks: %u : CLOCKS_PER_SEC %u \n", ( (float) ( endclockscpu - startclockscpu)) / CLOCKS_PER_SEC / ITERSTEPS, endclockscpu - startclockscpu, CLOCKS_PER_SEC );
        printf( "\n CPU, measured by CUDA event: %f ms", et );
        printf( "\n GPU, measured by CUDA event: %f ms", et_gpu );
	    printf( "\n cudamemcopy payoff %f secsonds (%u clocks)", (float) (payoffend - payoffstart) / CLOCKS_PER_SEC, payoffend - payoffstart );

        e = hipFree(devivemem);
        CUDA_UTIL_ERRORCHECK("hipFree")
        free( hostmem );
    }
}
