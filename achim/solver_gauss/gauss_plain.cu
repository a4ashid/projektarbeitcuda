#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "gausskernel.h"
#include "util.h"
#include "problemsamples.h"

#define NMAX 22




t_matrix M1;



// -----------------------------------------------------------------------



// -----------------------------------------------------------------------
void pull_problem_from_device( t_pmatrix matrix ) {
   hipError_t e;
    e = hipMemcpy(  matrix->elements, matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyDeviceToHost);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMemcpy(  matrix->x, matrix->device_x, sizeof(t_ve) * matrix->n, hipMemcpyDeviceToHost);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
}
// -----------------------------------------------------------------------
void push_problem_to_device( t_pmatrix matrix ) {


    hipError_t e;
    e = hipMalloc ((void **) &matrix->device_x, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMalloc ((void **) &matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipMemcpy( matrix->device_x, matrix->x , sizeof(t_ve)*matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipMemcpy( matrix->device_elements, matrix->elements , sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}




// -----------------------------------------------------------------------
// -----------------------------------------------------------------------

void eleminate ( t_ve* Ab, unsigned int N ) {
    unsigned int i;   // columns
    unsigned int j;   // rows, equitations
    unsigned int k, max;
    t_ve t;



    for ( i = 1; i <= N ; i++ ) {


       max = i;
       for( j = i + 1; j <= N; j++ ) {
           if ( abs( Ab[ a(j,i) ] ) > abs( Ab[ a(max,i) ] )  ) {
              max = j;
           }
       }

       for ( k = i; k <= N + 1; k++ ) {
          t              = Ab[ a(i,k) ];
          Ab[ a(i,k)   ] = Ab[ a(max,k) ];
          Ab[ a(max,k) ] = t;
       }

       for ( j = i +1; j <= N ; j++ ) {
          for ( k = N + 1; k >= i ; k-- ) {
             Ab[ a(j,k) ] -= Ab[ a(i,k) ] * Ab[ a(j,i) ] /  Ab[ a(i,i) ];
          }
       }
    }
}
// -----------------------------------------------------------------------



// -----------------------------------------------------------------------
void gen_textinput_01( t_pmatrix matrix ) {

// Example from R.Sedgewick, Page 608

   malloc_matrix( 3, matrix );

   matrix->elements[ 0 ]  = 1;
   matrix->elements[ 1 ]  = 3;
   matrix->elements[ 2 ]  = -4;

   matrix->elements[ 4 ]  = 1;
   matrix->elements[ 5 ]  = 1;
   matrix->elements[ 6 ]  = -2;

   matrix->elements[  8 ]  = -1;
   matrix->elements[  9 ]  = -2;
   matrix->elements[ 10 ]  = 5;

   matrix->elements[  3 ]  = 8;
   matrix->elements[  7 ]  = 2;
   matrix->elements[ 11 ]  = -1;
}
// -----------------------------------------------------------------------
// -----------------------------------------------------------------------
void gen_textinput_02( t_pmatrix matrix ) {

// Example from buyu

malloc_matrix( 3, matrix );

matrix->elements[0]=1;
matrix->elements[1]=2;
matrix->elements[2]=3;
matrix->elements[3]=14;
matrix->elements[4]=1;
matrix->elements[5]=1;
matrix->elements[6]=1;
matrix->elements[7]=6;
matrix->elements[8]=2;
matrix->elements[9]=1;
matrix->elements[10]=1;
matrix->elements[11]=7;

}
// -----------------------------------------------------------------------

int main()
{
    unsigned int problem;
    hipError_t e;

    int block_size = NMAX;
    dim3 dimBlock(block_size, block_size );
    dim3 dimGrid ( 1 );

    for ( problem = 1; problem < 3; problem++ ) {
        gen_problemsample( &M1, 2 );
        printf( "\n \nRunning problem No. %u\n", problem );
        backup_problem( &M1 );
        dump_problem( M1.elements, M1.n );
        push_problem_to_device( &M1 );

        device_gauss_solver<<<dimGrid,dimBlock>>>( M1.device_elements, M1.n, M1.device_x );

        e = hipGetLastError();
        if( e != hipSuccess )
        {
            fprintf(stderr, "CUDA Error on add_arrays_gpu: '%s' \n", hipGetErrorString(e));
            exit(-3);
        }


        pull_problem_from_device( &M1 );
        printf( "\n solution: \n" );

        dump_problem( M1.elements, M1.n );
        dump_x( M1.x, M1.n );
        check_correctness( M1.orgelements, M1.n, M1.x );
        e = hipFree(M1.device_elements);
        if( e != hipSuccess )
        {
            fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
            exit(-3);
        }
        e = hipFree(M1.device_x);
        if( e != hipSuccess )
        {
            fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
            exit(-3);
        }
        free_matrix( &M1 );
    }
}



