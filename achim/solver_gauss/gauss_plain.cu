

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define Ae( j , i, N ) (j -1 ) * ( N + 1 ) + i -1

#define ME( A, j , i ) A->elements[ Ae( j , i, A->n ) ]




typedef float        t_ve   ;
//#typedef unsigned int t_vidx ; // index of vector elements

t_ve *a;

typedef struct {

    unsigned int n;
    t_ve*    elements;
    t_ve*    x;

    t_ve*    device_elements;
    t_ve*    device_x;

} t_matrix;

typedef t_matrix* t_pmatrix;

t_matrix M1;

// -----------------------------------------------------------------------

__global__ void device_substitute( t_ve* x, t_ve* Ab, unsigned int N ) {

   unsigned int j,k;
   t_ve t;

   unsigned int tidx = threadIdx.y * blockDim.x + threadIdx.x;



   if ( tidx == 0 ) {

   for (j = N; j >= 1; j-- ) {
       t = 0.0;
       for ( k = j + 1; k <= N; k++ ) {
           t +=  Ab[ Ae( j , k, N ) ] * x[ k - 1 ];
       }
       x[ j - 1 ] = ( Ab[ Ae( j , N + 1, N ) ] - t ) / Ab[ Ae( j , j, N) ] ;
   }
   }
}

// -----------------------------------------------------------------------

__global__ void device_eleminate( t_ve* Ab, unsigned int N  )
{


    __shared__ unsigned int i;
    __shared__ unsigned int max;
    t_ve t;

    unsigned int tidx = threadIdx.y * blockDim.x + threadIdx.x;
//    unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;

    if ( tidx == 0 ) { i = 1; }

    __syncthreads();

//


//       for ( i = 1; i <= N ; i++ ) {
    while ( i <= N ) {
        if ( tidx == 0 ) {
            unsigned int j;
            max = i;
            for( j = i + 1; j <= N; j++ ) {
                if ( abs( Ab[ Ae( j , i , N ) ] ) > abs( Ab[ Ae( max , i, N ) ] )  ) {
                        max = j;
                }
            }
       }
       __syncthreads();


//       for ( k = i; k <= N; k++ ) {
         if ( threadIdx.y == 0 )
         {
             unsigned int k = threadIdx.x + 1;
             if ( ( k >= i ) && ( k <= N )  ) {
                 t                          = Ab[ Ae(   i , k, N ) ];
                 Ab[ Ae( i   , k ,  N )   ] = Ab[ Ae( max , k, N ) ];
                 Ab[ Ae( max , k, N ) ]     = t;

             }
         }
         __syncthreads();

//       if ( tidx == 1 )
      {
           unsigned int j = threadIdx.x + 1;
//           printf("\n **** hallo hallo i %u ", i );
//       for ( j = i +1; j <= N ; j++ ) {
         if (  ( j >= i +1 ) && ( j <= N ) ) {
          unsigned int  k;
          for ( k = N + 1; k >= i ; k-- ) {
             Ab[ Ae( j , k , N ) ] -= Ab[ Ae( i , k, N ) ] * Ab[ Ae( j , i, N ) ] /  Ab[ Ae( i , i, N ) ];
          }
       }
       }
       __syncthreads();
       if ( tidx == 0 ) { i++; }

    }

    }
//}



// -----------------------------------------------------------------------
void pull_problem_from_device( t_pmatrix matrix ) {
   hipError_t e;
    e = hipMemcpy(  matrix->elements, matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyDeviceToHost);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMemcpy(  matrix->x, matrix->device_x, sizeof(t_ve) * matrix->n, hipMemcpyDeviceToHost);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
}
// -----------------------------------------------------------------------
void push_problem_to_device( t_pmatrix matrix ) {


    hipError_t e;
    e = hipMalloc ((void **) &matrix->device_x, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMalloc ((void **) &matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipMemcpy( matrix->device_x, matrix->x , sizeof(t_ve)*matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipMemcpy( matrix->device_elements, matrix->elements , sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}
// -----------------------------------------------------------------------
void malloc_matrix( unsigned int size_n, t_pmatrix matrix ) {

   matrix->n        = size_n;
   matrix->elements =  (t_ve*) malloc( sizeof(t_ve) * (size_n + 1 ) * size_n ); /* store b in array, too */
   if ( matrix->elements == NULL) {
       fprintf(stderr, "sorry, can not allocate memory for you");
       exit( -1 );
   }
   matrix->x = (t_ve*) malloc( sizeof(t_ve)  * size_n ); /* the output vector */

   if ( matrix->elements == NULL) {
       fprintf(stderr, "sorry, can not allocate memory for you");
       exit( -1 );
   }
}




// -----------------------------------------------------------------------
void substitute( t_ve* x, t_ve* Ab, unsigned int N ) {
   unsigned int j, k;
   t_ve t;

   for (j = N; j >= 1; j-- ) {
       t = 0.0;
       for ( k = j + 1; k <= N; k++ ) {
           t +=  Ab[ Ae( j , k, N ) ] * x[ k - 1 ];
       }
       x[ j - 1 ] = ( Ab[ Ae( j , N + 1, N ) ] - t ) / Ab[ Ae( j , j, N) ] ;
   }
}
// -----------------------------------------------------------------------

void eleminate ( t_ve* Ab, unsigned int N ) {
    unsigned int i, j, k, max;
    t_ve t;


    for ( i = 1; i <= N ; i++ ) {
       max = i;
       for( j = i + 1; j <= N; j++ ) {
           if ( abs( Ab[ Ae( j , i , N ) ] ) > abs( Ab[ Ae( max , i, N ) ] )  ) {
              max = j;
           }
       }
       for ( k = i; k <= N; k++ ) {
          t                   = Ab[ Ae(   i , k, N ) ];
          Ab[ Ae( i , k ,  N )   ] = Ab[ Ae( max , k, N ) ];
          Ab[ Ae( max , k, N ) ] = t;
       }

       for ( j = i +1; j <= N ; j++ ) {
          for ( k = N + 1; k >= i ; k-- ) {
             Ab[ Ae( j , k , N ) ] -= Ab[ Ae( i , k, N ) ] * Ab[ Ae( j , i, N ) ] /  Ab[ Ae( i , i, N ) ];
          }
       }
    }
}
// -----------------------------------------------------------------------
void dump_matrix( t_pmatrix matrix ) {
    int n;
    int m;
    for ( m = 0; m < matrix->n; m++ ) {
        printf( "\n  %u. ", m + 1 );
        for ( n = 0; n < matrix->n; n++ ) {
            printf( " %f", matrix->elements[ m * ( matrix->n + 1 ) + n ] );
        }
        printf( " \t b %f", matrix->elements[ m * ( matrix->n + 1 ) + n ] );
   }
   for ( m = 0; m < matrix->n; m++ ) {
      printf( "\n  x%u  = %f",m + 1, matrix->x[ m ] );
   }
}
// -----------------------------------------------------------------------
void gen_textinput_01( t_pmatrix matrix ) {

// Example from R.Sedgewick, Page 608

   malloc_matrix( 3, matrix );

   matrix->elements[ 0 ]  = 1;
   matrix->elements[ 1 ]  = 3;
   matrix->elements[ 2 ]  = -4;

   matrix->elements[ 4 ]  = 1;
   matrix->elements[ 5 ]  = 1;
   matrix->elements[ 6 ]  = -2;

   matrix->elements[  8 ]  = -1;
   matrix->elements[  9 ]  = -2;
   matrix->elements[ 10 ]  = 5;

   matrix->elements[  3 ]  = 8;
   matrix->elements[  7 ]  = 2;
   matrix->elements[ 11 ]  = -1;
}
// -----------------------------------------------------------------------

int main()
{
//    malloc_matrix( 3, &M1 );

    hipError_t e;

    gen_textinput_01( &M1 );

    printf( "hello world , size ist set to %u\n", M1.n );


    dump_matrix( &M1 );
//    eleminate( M1.elements, M1.n );


    push_problem_to_device( &M1 );

    int block_size = 64;
    dim3 dimBlock(block_size );

    dim3 dimGrid ( 1 );

    device_eleminate<<<dimGrid,dimBlock>>>( M1.device_elements, M1.n );
    e = hipGetLastError();
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on add_arrays_gpu: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    device_substitute<<<dimGrid,dimBlock>>>( M1.device_x, M1.device_elements, M1.n );
    e = hipGetLastError();
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on add_arrays_gpu: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    pull_problem_from_device( &M1 );

    printf( "\n" );
    dump_matrix( &M1 );
}



