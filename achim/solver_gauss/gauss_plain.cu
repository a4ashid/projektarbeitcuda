#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "gausskernel.h"
#include "util.h"
#include "problemsamples.h"
#include "idrs.h"

#include "measurehelp.h"

#define ITERATIONS 5

t_matrix M1;



// -----------------------------------------------------------------------



// -----------------------------------------------------------------------
void pull_problem_from_device( t_pmatrix matrix ) {
   hipError_t e;
    e = hipMemcpy(  matrix->elements, matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK("hipMemcpy");

    e = hipMemcpy(  matrix->x, matrix->device_x, sizeof(t_ve) * matrix->n, hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK("hipMemcpy");

}
// -----------------------------------------------------------------------
void push_problem_to_device( t_pmatrix matrix ) {


    hipError_t e;
    e = hipMalloc ((void **) &matrix->device_x, sizeof(t_ve) * matrix->n );
    CUDA_UTIL_ERRORCHECK("hipMalloc");

    e = hipMalloc ((void **) &matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    CUDA_UTIL_ERRORCHECK("hipMalloc");
/*
    e = hipMemcpy( matrix->device_x, matrix->x , sizeof(t_ve)*matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
*/
    e = hipMemcpy( matrix->device_elements, matrix->elements , sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpy");

}




// -----------------------------------------------------------------------
// -----------------------------------------------------------------------

void eleminate ( t_ve* Ab, t_ve* x, unsigned int N ) {
    unsigned int i;   // columns
    unsigned int j;   // rows, equitations
    unsigned int k, max;
    t_ve t;

    for ( i = 1; i <= N ; i++ ) {


       max = i;
       for( j = i + 1; j <= N; j++ ) {
           if ( abs( Ab[ ab(j,i) ] ) > abs( Ab[ ab(max,i) ] )  ) {
              max = j;
           }
       }

       for ( k = i; k <= N + 1; k++ ) {
          t              = Ab[ ab(i,k) ];
          Ab[ ab(i,k)   ] = Ab[ ab(max,k) ];
          Ab[ ab(max,k) ] = t;
       }

       for ( j = i +1; j <= N ; j++ ) {
          for ( k = N + 1; k >= i ; k-- ) {
             Ab[ ab(j,k) ] -= Ab[ ab(i,k) ] * Ab[ ab(j,i) ] /  Ab[ ab(i,i) ];
          }
       }


      // substitute ...

        for (j = N; j >= 1; j-- ) {
            t_ve t = 0.0;
            for ( k = j + 1; k <= N; k++ ) {
                    t +=  Ab[ ab(j,k) ] * x[ k - 1 ];
            }
            x[ j - 1 ] = ( Ab[ ab(j,N+1) ] - t ) / Ab[ ab(j,j) ] ;
        }

    }
}
// -----------------------------------------------------------------------



// -----------------------------------------------------------------------

int main()
{
    unsigned int problem;
    hipError_t e;

//    int block_size = NMAX;
    dim3 dimBlock(  GAUSS_NMAX + 2 );
    dim3 dimGrid ( 1 );

    for ( problem = 1; problem < 5; problem++ ) {
        gen_problemsample( &M1, problem );
        printf( "\n \nRunning problem No. %u , size %u\n", problem, M1.n );
        backup_problem( &M1 );
        if ( M1.n < 0 ) {
            dump_problem( M1.elements, M1.n );
        }
        push_problem_to_device( &M1 );

        float kernel_ms;

        if ( M1.n  <= GAUSS_NMAX ) {

        {
           for ( int i = 0; i < 10; i++ ) {
            device_gauss_solver<<<dimGrid,dimBlock>>>( M1.device_elements, M1.n, M1.device_x );
            e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("kernel");
           }
        }
        {
            START_CUDA_TIMER
            for ( int i = 0; i < ITERATIONS; i++ ) {
                device_gauss_solver<<<dimGrid,dimBlock>>>( M1.device_elements, M1.n, M1.device_x );
                e = hipGetLastError();
                CUDA_UTIL_ERRORCHECK("kernel");
            }

            STOP_CUDA_TIMER( & kernel_ms )
        }

        printf("\nelapsed time GPU: %f ms\n", kernel_ms / ITERATIONS );
        pull_problem_from_device( &M1 );


        if ( M1.n < 9 ) {
           // printf( "\n solution: \n" );
            dump_problem( M1.elements, M1.n );
            dump_x( M1.x, M1.n );
        }
        check_correctness( M1.orgelements, M1.n, M1.x );
        }
        e = hipFree(M1.device_elements);
        CUDA_UTIL_ERRORCHECK("hipFree");
        e = hipFree(M1.device_x);
        CUDA_UTIL_ERRORCHECK("hipFree");

        float cpu_ms;

        {
            START_CUDA_TIMER

            eleminate ( M1.orgelements, M1.x, M1.n );

            STOP_CUDA_TIMER( &cpu_ms )
        }
        printf("\nelapsed time CPU: %f ms\n \n ------------------------", cpu_ms );
        free_matrix( &M1 );
    }
}



