#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "gausskernel.h"
#include "util.h"
#include "problemsamples.h"
#include "idrs.h"
#define NMAX 22




t_matrix M1;



// -----------------------------------------------------------------------



// -----------------------------------------------------------------------
void pull_problem_from_device( t_pmatrix matrix ) {
   hipError_t e;
    e = hipMemcpy(  matrix->elements, matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyDeviceToHost);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMemcpy(  matrix->x, matrix->device_x, sizeof(t_ve) * matrix->n, hipMemcpyDeviceToHost);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
}
// -----------------------------------------------------------------------
void push_problem_to_device( t_pmatrix matrix ) {


    hipError_t e;
    e = hipMalloc ((void **) &matrix->device_x, sizeof(t_ve) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMalloc ((void **) &matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
/*
    e = hipMemcpy( matrix->device_x, matrix->x , sizeof(t_ve)*matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
*/
    e = hipMemcpy( matrix->device_elements, matrix->elements , sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}




// -----------------------------------------------------------------------
// -----------------------------------------------------------------------

void eleminate ( t_ve* Ab, unsigned int N ) {
    unsigned int i;   // columns
    unsigned int j;   // rows, equitations
    unsigned int k, max;
    t_ve t;



    for ( i = 1; i <= N ; i++ ) {


       max = i;
       for( j = i + 1; j <= N; j++ ) {
           if ( abs( Ab[ a(j,i) ] ) > abs( Ab[ a(max,i) ] )  ) {
              max = j;
           }
       }

       for ( k = i; k <= N + 1; k++ ) {
          t              = Ab[ a(i,k) ];
          Ab[ a(i,k)   ] = Ab[ a(max,k) ];
          Ab[ a(max,k) ] = t;
       }

       for ( j = i +1; j <= N ; j++ ) {
          for ( k = N + 1; k >= i ; k-- ) {
             Ab[ a(j,k) ] -= Ab[ a(i,k) ] * Ab[ a(j,i) ] /  Ab[ a(i,i) ];
          }
       }
    }
}
// -----------------------------------------------------------------------



// -----------------------------------------------------------------------

int main()
{
    unsigned int problem;
    hipError_t e;

    int block_size = NMAX;
    dim3 dimBlock( block_size, block_size );
    dim3 dimGrid ( 1 );

    for ( problem = 1; problem < 5; problem++ ) {
        gen_problemsample( &M1, problem );
        printf( "\n \nRunning problem No. %u , size %u\n", problem, M1.n );
        backup_problem( &M1 );
        if ( M1.n < 9 ) {
            dump_problem( M1.elements, M1.n );
        }
        push_problem_to_device( &M1 );

        if ( M1.n <= block_size ) {
        device_gauss_solver<<<dimGrid,dimBlock>>>( M1.device_elements, M1.n, M1.device_x );

        e = hipGetLastError();
        if( e != hipSuccess )
        {
            fprintf(stderr, "CUDA Error on add_arrays_gpu: '%s' \n", hipGetErrorString(e));
            exit(-3);
        }

        }

        pull_problem_from_device( &M1 );


        if ( M1.n < 9 ) {
            printf( "\n solution: \n" );
            dump_problem( M1.elements, M1.n );
            dump_x( M1.x, M1.n );
        }
        check_correctness( M1.orgelements, M1.n, M1.x );
        e = hipFree(M1.device_elements);
        if( e != hipSuccess )
        {
            fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
            exit(-3);
        }
        e = hipFree(M1.device_x);
        if( e != hipSuccess )
        {
            fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
            exit(-3);
        }
        free_matrix( &M1 );
    }
}



