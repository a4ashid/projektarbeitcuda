

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

typedef float        t_ve   ;
//#typedef unsigned int t_vidx ; // index of vector elements

t_ve *a;

typedef struct {

    unsigned int n;
    t_ve*    elements;
    t_ve*    x;

    t_ve*    device_elements;
    t_ve*    device_x;

} t_matrix;

typedef t_matrix* t_pmatrix;

t_matrix M1;

// -----------------------------------------------------------------------
void push_problem_to_device( t_pmatrix matrix ) {


    hipError_t e;
    e = hipMalloc ((void **) &matrix->device_x, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMalloc ((void **) &matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipMemcpy( matrix->device_x, matrix->x , sizeof(t_ve)*matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipMemcpy( matrix->device_elements, matrix->elements , sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}
// -----------------------------------------------------------------------
void malloc_matrix( unsigned int size_n, t_pmatrix matrix ) {

   matrix->n        = size_n;
   matrix->elements =  (t_ve*) malloc( sizeof(t_ve) * (size_n + 1 ) * size_n ); /* store b in array, too */
   if ( matrix->elements == NULL) {
       fprintf(stderr, "sorry, can not allocate memory for you");
       exit( -1 );
   }
   matrix->x = (t_ve*) malloc( sizeof(t_ve)  * size_n ); /* the output vector */

   if ( matrix->elements == NULL) {
       fprintf(stderr, "sorry, can not allocate memory for you");
       exit( -1 );
   }
}


#define ME( A, j , i ) A->elements[ (j -1 ) * ( A->n + 1 ) + i -1 ]

// -----------------------------------------------------------------------
void substitute( t_pmatrix matrix ) {
   unsigned int j, k, N;
   t_ve t;

   N = matrix->n;

   for (j = N; j >= 1; j-- ) {
       t = 0.0;
       for ( k = j + 1; k <= N; k++ ) {
           t +=  ME( matrix, j , k ) * matrix->x[ k - 1 ];
       }
       matrix->x[ j - 1 ] = ( ME( matrix, j , N + 1 ) - t ) / ME( matrix, j , j );
   }
}
// -----------------------------------------------------------------------

void eleminate ( t_pmatrix matrix ) {
    unsigned int i, j, k, max, N;
    t_ve t;

    N = matrix->n;

    for ( i = 1; i <= N ; i++ ) {
       max = i;
       for( j = i + 1; j <= N; j++ ) {
           if ( abs( ME( matrix, j , i ) ) > abs( ME( matrix, max , i ) ) ) {
              max = j;
           }
       }
       for ( k = i; k <= N; k++ ) {
          t                     = ME( matrix, i , k );
          ME( matrix, i , k )   = ME( matrix, max , k );
          ME( matrix, max , k ) = t;
       }

       for ( j = i +1; j <= N ; j++ ) {
          for ( k = N + 1; k >= i ; k-- ) {
             ME( matrix, j , k ) -= ME( matrix, i , k ) * ME( matrix, j , i ) /  ME( matrix, i , i );
          }
       }
    }
}
// -----------------------------------------------------------------------
void dump_matrix( t_pmatrix matrix ) {
    int n;
    int m;
    for ( m = 0; m < matrix->n; m++ ) {
        printf( "\n  %u. ", m + 1 );
        for ( n = 0; n < matrix->n; n++ ) {
            printf( " %f", matrix->elements[ m * ( matrix->n + 1 ) + n ] );
        }
        printf( " \t b %f", matrix->elements[ m * ( matrix->n + 1 ) + n ] );
   }
   for ( m = 0; m < matrix->n; m++ ) {
      printf( "\n  x%u  = %f",m + 1, matrix->x[ m ] );
   }
}
// -----------------------------------------------------------------------
void gen_textinput_01( t_pmatrix matrix ) {

// Example from R.Sedgewick, Page 608

   malloc_matrix( 3, matrix );

   matrix->elements[ 0 ]  = 1;
   matrix->elements[ 1 ]  = 3;
   matrix->elements[ 2 ]  = -4;

   matrix->elements[ 4 ]  = 1;
   matrix->elements[ 5 ]  = 1;
   matrix->elements[ 6 ]  = -2;

   matrix->elements[  8 ]  = -1;
   matrix->elements[  9 ]  = -2;
   matrix->elements[ 10 ]  = 5;

   matrix->elements[  3 ]  = 8;
   matrix->elements[  7 ]  = 2;
   matrix->elements[ 11 ]  = -1;
}
// -----------------------------------------------------------------------

int main()
{
//    malloc_matrix( 3, &M1 );

    gen_textinput_01( &M1 );

    printf( "hello world , size ist set to %u\n", M1.n );


    dump_matrix( &M1 );
//    eleminate( &M1 );
//    substitute( &M1 );

    push_problem_to_device( &M1 );

    printf( "\n" );
    dump_matrix( &M1 );
}



