

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define a( r, s ) (r -1 ) * ( N + 1 ) + s -1

#define NMAX 22

typedef float        t_ve   ;

typedef struct {

    unsigned int n;
    t_ve*    elements;
    t_ve*    x;

    t_ve*    device_elements;
    t_ve*    device_x;

} t_matrix;

typedef t_matrix* t_pmatrix;

t_matrix M1;

// -----------------------------------------------------------------------

__global__ void device_substitute( t_ve* x, t_ve* Ab, unsigned int N ) {



    if ( 0 == threadIdx.y * blockDim.x + threadIdx.x ) {
        unsigned int j,k;
        for (j = N; j >= 1; j-- ) {
            t_ve t = 0.0;
            for ( k = j + 1; k <= N; k++ ) {
                    t +=  Ab[ a(j,k) ] * x[ k - 1 ];
            }
            x[ j - 1 ] = ( Ab[ a(j,N+1) ] - t ) / Ab[ a(j,j) ] ;
        }
   }
}

// -----------------------------------------------------------------------

__global__ void device_gauss_solve( t_ve* p_Ab, unsigned int N, t_ve* p_x )
{


    __shared__ unsigned int i;
    __shared__ unsigned int max;

    __shared__ t_ve Ab[ NMAX * ( NMAX + 1) ];
     t_ve x[ NMAX ];

    unsigned int tidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int n;
    if ( tidx == 0 ) {
        i = 1;
        for  ( n = 0; n <  N * (N+1); n++ ) {
            Ab[n] = p_Ab[n];
        }
    }
    __syncthreads();

    while ( i <= N ) {                  /* for ( i = 1; i <= N ; i++ ) */
        if ( tidx == 0 ) {
            unsigned int j;
            max = i;
            for( j = i + 1; j <= N; j++ ) {
                if ( abs( Ab[ a(j,i) ] ) > abs( Ab[ a(max,i) ] )  ) {
                    max = j;
                }
            }
       }
       __syncthreads();


       if ( threadIdx.y == 0 ) {
           unsigned int k = threadIdx.x + 1;
           if ( ( k >= i ) && ( k <= N + 1 ) ) {
               t_ve t         = Ab[ a(i  ,k) ];
               Ab[ a(i,k)   ] = Ab[ a(max,k) ];
               Ab[ a(max,k) ] = t;
           }
       }
       __syncthreads();

      {
          unsigned int j = threadIdx.x + 1;
          if (  ( j >= i +1 ) && ( j <= N ) && threadIdx.y == 0 ) {       /*   for ( j = i +1; j <= N ; j++ ) */
              unsigned int  k ;
              for ( k = N + 1; k >= i ; k-- ) {
                 Ab[ a(j,k) ] -= Ab[ a(i,k) ] * Ab[ a(j,i) ] /  Ab[ a(i, i) ];
              }
           }
       }
       __syncthreads();
       if ( tidx == 0 ) { i++; }
    }
    __syncthreads();

    if ( tidx == 0 ) {

        /* the substitute part */
        unsigned int j,k;
        for (j = N; j >= 1; j-- ) {
            t_ve t = 0.0;
            for ( k = j + 1; k <= N; k++ ) {
                    t +=  Ab[ a(j,k) ] * x[ k - 1 ];
            }
            x[ j - 1 ] = ( Ab[ a(j,N+1) ] - t ) / Ab[ a(j,j) ] ;
        }
        /* copy result back to global memory */

        for  ( n = 0; n <  N * (N+1); n++ ) {
            p_Ab[n] = Ab[n];
        }
        for  ( n = 0; n < N; n++ ) {
            p_x[n] = x[n];
        }
    }
   __syncthreads();
}



// -----------------------------------------------------------------------
void pull_problem_from_device( t_pmatrix matrix ) {
   hipError_t e;
    e = hipMemcpy(  matrix->elements, matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyDeviceToHost);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMemcpy(  matrix->x, matrix->device_x, sizeof(t_ve) * matrix->n, hipMemcpyDeviceToHost);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
}
// -----------------------------------------------------------------------
void push_problem_to_device( t_pmatrix matrix ) {


    hipError_t e;
    e = hipMalloc ((void **) &matrix->device_x, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipMalloc ((void **) &matrix->device_elements, sizeof(t_ve) * (matrix->n + 1 ) * matrix->n );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipMemcpy( matrix->device_x, matrix->x , sizeof(t_ve)*matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipMemcpy( matrix->device_elements, matrix->elements , sizeof(t_ve) * (matrix->n + 1 ) * matrix->n, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}
// -----------------------------------------------------------------------
void malloc_matrix( unsigned int size_n, t_pmatrix matrix ) {

   matrix->n        = size_n;
   matrix->elements =  (t_ve*) malloc( sizeof(t_ve) * (size_n + 1 ) * size_n ); /* store b in array, too */
   if ( matrix->elements == NULL) {
       fprintf(stderr, "sorry, can not allocate memory for you");
       exit( -1 );
   }
   matrix->x = (t_ve*) malloc( sizeof(t_ve)  * size_n ); /* the output vector */

   if ( matrix->elements == NULL) {
       fprintf(stderr, "sorry, can not allocate memory for you");
       exit( -1 );
   }
}




// -----------------------------------------------------------------------
void substitute( t_ve* x, t_ve* Ab, unsigned int N ) {
   unsigned int j, k;
   t_ve t;

   for (j = N; j >= 1; j-- ) {
       t = 0.0;
       for ( k = j + 1; k <= N; k++ ) {
           t +=  Ab[ a( j , k ) ] * x[ k - 1 ];
       }
       x[ j - 1 ] = ( Ab[ a(j,N+1) ] - t ) / Ab[ a(j,j) ] ;
   }
}
// -----------------------------------------------------------------------

void eleminate ( t_ve* Ab, unsigned int N ) {
    unsigned int i;   // columns
    unsigned int j;   // rows, equitations
    unsigned int k, max;
    t_ve t;



    for ( i = 1; i <= N ; i++ ) {


       max = i;
       for( j = i + 1; j <= N; j++ ) {
           if ( abs( Ab[ a(j,i) ] ) > abs( Ab[ a(max,i) ] )  ) {
              max = j;
           }
       }

       for ( k = i; k <= N + 1; k++ ) {
          t              = Ab[ a(i,k) ];
          Ab[ a(i,k)   ] = Ab[ a(max,k) ];
          Ab[ a(max,k) ] = t;
       }

       for ( j = i +1; j <= N ; j++ ) {
          for ( k = N + 1; k >= i ; k-- ) {
             Ab[ a(j,k) ] -= Ab[ a(i,k) ] * Ab[ a(j,i) ] /  Ab[ a(i,i) ];
          }
       }
    }
}
// -----------------------------------------------------------------------
void dump_matrix( t_pmatrix matrix ) {
    unsigned int j;
    unsigned int i;
    unsigned int N;

    N = matrix->n;

    for ( j = 1; j <= N; j++ ) {
        printf( "\n  %u. ", j );
        for ( i = 1; i <= N; i++ ) {
            printf( " %f", matrix->elements[ a(j,i) ] );
        }
        printf( " \t b %f", matrix->elements[ a(j,i) ] );
   }
   {
       unsigned int m;
       for ( m = 0; m < matrix->n; m++ ) {
           printf( "\n  x%u  = %f",m + 1, matrix->x[ m ] );
       }
   }

}
// -----------------------------------------------------------------------
void gen_textinput_01( t_pmatrix matrix ) {

// Example from R.Sedgewick, Page 608

   malloc_matrix( 3, matrix );

   matrix->elements[ 0 ]  = 1;
   matrix->elements[ 1 ]  = 3;
   matrix->elements[ 2 ]  = -4;

   matrix->elements[ 4 ]  = 1;
   matrix->elements[ 5 ]  = 1;
   matrix->elements[ 6 ]  = -2;

   matrix->elements[  8 ]  = -1;
   matrix->elements[  9 ]  = -2;
   matrix->elements[ 10 ]  = 5;

   matrix->elements[  3 ]  = 8;
   matrix->elements[  7 ]  = 2;
   matrix->elements[ 11 ]  = -1;
}
// -----------------------------------------------------------------------
// -----------------------------------------------------------------------
void gen_textinput_02( t_pmatrix matrix ) {

// Example from buyu

malloc_matrix( 3, matrix );

matrix->elements[0]=1;
matrix->elements[1]=2;
matrix->elements[2]=3;
matrix->elements[3]=14;
matrix->elements[4]=1;
matrix->elements[5]=1;
matrix->elements[6]=1;
matrix->elements[7]=6;
matrix->elements[8]=2;
matrix->elements[9]=1;
matrix->elements[10]=1;
matrix->elements[11]=7;

}
// -----------------------------------------------------------------------

int main()
{
//    malloc_matrix( 3, &M1 );

    hipError_t e;

    gen_textinput_01( &M1 );

    printf( "hello world , size ist set to %u\n", M1.n );


    dump_matrix( &M1 );

//    eleminate( M1.elements, M1.n );
//    substitute( M1.x, M1.elements, M1.n );


    push_problem_to_device( &M1 );

    int block_size = NMAX;
    dim3 dimBlock(block_size, block_size );

    dim3 dimGrid ( 1 );

    device_gauss_solve<<<dimGrid,dimBlock>>>( M1.device_elements, M1.n, M1.device_x );
    e = hipGetLastError();
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on add_arrays_gpu: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
//    device_substitute<<<dimGrid,dimBlock>>>( M1.device_x, M1.device_elements, M1.n );

    pull_problem_from_device( &M1 );

    printf( "\n solution: \n" );
    dump_matrix( &M1 );
    e = hipFree(M1.device_elements);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipFree(M1.device_x);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
}



