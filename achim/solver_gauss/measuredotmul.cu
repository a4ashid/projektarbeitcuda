#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include <time.h>

#include "dotMul_cuda_gpu.h"
#include "dotMul_cpu.h"
#include <time.h>

#define N_PROBLEM 50000
#define ITERSTEPS 2

__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    t_ve* v =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( v == NULL) {
	       fprintf(stderr, "sorry, can not allocate memory for you");
	       exit( -1 );
    }
    *M = v;

}



__host__ void malloc_vector_on_device(  t_ve** pV_d, unsigned int L ) {
	hipError_t e;

    e = hipMalloc ((void **) pV_d, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    hipMemset ( *pV_d, 0, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on memset 0: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}


int main()
{
    printf("\n measureing operation DOTMUL with N = %u, ITERSTEPS %u", N_PROBLEM, ITERSTEPS );

    t_ve* v1;
    t_ve* v2;

    t_ve* v1_d;
    t_ve* v2_d;

    t_ve* out ;  /* output vector */
    t_ve* out_d;

    int block_size = 512;


    clock_t startclocks;
    clock_t endclock;

    clock_t payoffstart;
    clock_t payoffend;


    clock_t startclockscpu;
    clock_t endclockscpu;

    hipError_t e;

    malloc_N( N_PROBLEM * 3 , &v1 );
    //malloc_N( N_PROBLEM  , &v2 );
    //malloc_N( N_PROBLEM  , &out );






    for ( unsigned long N = 1; N < N_PROBLEM; N *= 5 ) {

		/* ------------------------------------------------------------ */



    payoffstart = clock();

        malloc_vector_on_device( &v1_d , N * 3 );

        v2_d  = &v1_d[N    ];
        out_d = &v1_d[N * 2];

        e = hipMemcpy(  v1_d, v1, sizeof(t_ve) * N * 2 , hipMemcpyHostToDevice);
        if( e != hipSuccess )
        {
            fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
            exit(-3);
        }

        payoffend = clock();


        startclocks = clock( );
        for ( int i = 0; i < ITERSTEPS; i++ ) {
            dim3 dimBlock(block_size);
            dim3 dimGrid ( N / block_size + 1 );
            device_dotMul<<<dimGrid,dimBlock>>>(v1_d, v2_d, out_d, N );
        	e = hipGetLastError();
			if( e != hipSuccess ) {
			       fprintf(stderr, "CUDA Error on add_arrays_gpu: '%s' \n", hipGetErrorString(e));
			       exit(-3);
            }
        }
        endclock = clock( );

        e = hipFree(v1_d);
        if( e != hipSuccess )
        {
            fprintf(stderr, "CUDA Error on hipMemcpy: '%s' \n", hipGetErrorString(e));
            exit(-3);
        }
        /* ------------------------------------------------------------ */
		startclockscpu = clock( );
		for ( int i = 0; i < ITERSTEPS; i++ ) {
		    v1[1]++; /* ensure opearation is not answerde from cache */
		    dotMul_cpu(v1, v2, out, N );
		}
        endclockscpu   = clock( );
        /* ------------------------------------------------------------ */

        printf( "\n ----------------------------------------------------- \n N = %u", N );
	    printf( "\n GPU: %f seconds,  clocks: %u : CLOCKS_PER_SEC %u \n", ( (float) ( endclock - startclocks)) / CLOCKS_PER_SEC / ITERSTEPS, endclock - startclocks, CLOCKS_PER_SEC );
	    printf( "\n CPU: %f seconds,  clocks: %u : CLOCKS_PER_SEC %u \n", ( (float) ( endclockscpu - startclockscpu)) / CLOCKS_PER_SEC / ITERSTEPS, endclockscpu - startclockscpu, CLOCKS_PER_SEC );
	    printf( "\n cudamemcopy payoff %f secsonds (%u clocks)", (float) (payoffend - payoffstart) / CLOCKS_PER_SEC, payoffend - payoffstart );
    }
}