#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include <time.h>

#include "dotMul_cuda_gpu.h"
#include "dotMul_cpu.h"
#include <time.h>

#define N_PROBLEM 100000
#define ITERSTEPS 2

__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    t_ve* v =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( v == NULL) {
	       fprintf(stderr, "sorry, can not allocate memory for you");
	       exit( -1 );
    }
    *M = v;
}

//    hipMemset ( *pV_d, 0, sizeof(t_ve) * L );

int main()
{
    pt_ve v1, v2, v1_d, v2_d, out, out_d;
    clock_t startclocks, endclock, payoffstart, payoffend, startclockscpu, endclockscpu;
    hipError_t e;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
        printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    }
    int block_size = 512;
    dim3 dimBlock(block_size);

    printf("\n measureing operation DOTMUL with N = %u, ITERSTEPS %u", N_PROBLEM, ITERSTEPS );
    malloc_N( N_PROBLEM * 3 , &v1 );

    for ( unsigned int N = 1; N <= N_PROBLEM; N *= 10 ) {
        payoffstart = clock();

        e = hipMalloc ((void **) &v1_d, sizeof(t_ve) * N * 3 );
        CUDA_UTIL_ERRORCHECK("hipMalloc &v1_d");

        v2_d  = &v1_d[N    ];
        out_d = &v1_d[N * 2];

        e = hipMemcpy(  v1_d, v1, sizeof(t_ve) * N * 2 , hipMemcpyHostToDevice);
        CUDA_UTIL_ERRORCHECK("hipMemcpy v1_d");

        payoffend = clock();
        startclocks = clock( );

        dim3 dimGrid ( N / block_size + 1 );

        for ( unsigned int i = 0; i < ITERSTEPS; i++ ) {

            device_dotMul<<<dimGrid,dimBlock>>>(v1_d, v2_d, out_d, N );
        	e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("Kernel device_dotMul")
        }
        endclock = clock( );

        e = hipFree(v1_d);
        CUDA_UTIL_ERRORCHECK("hipFree")

        /* ------------------------------------------------------------ */
		startclockscpu = clock( );

        v2  = &v1[N];
        out = &v1[N*2];

		for ( unsigned int i = 0; i < ITERSTEPS; i++ ) {
		    v1[1]++; /* ensure opearation is not answered from cache */
		    dotMul_cpu(v1, v2, out, N );
		}
        endclockscpu   = clock( );
        /* ------------------------------------------------------------ */

        printf( "\n ----------------------------------------------------- \n N = %u, ITER = %u", N, ITERSTEPS );
	    printf( "\n GPU: %f seconds,  clocks: %u : CLOCKS_PER_SEC %u \n", ( (float) ( endclock - startclocks)) / CLOCKS_PER_SEC / ITERSTEPS, endclock - startclocks, CLOCKS_PER_SEC );
	    printf( "\n CPU: %f seconds,  clocks: %u : CLOCKS_PER_SEC %u \n", ( (float) ( endclockscpu - startclockscpu)) / CLOCKS_PER_SEC / ITERSTEPS, endclockscpu - startclockscpu, CLOCKS_PER_SEC );
	    printf( "\n cudamemcopy payoff %f secsonds (%u clocks)", (float) (payoffend - payoffstart) / CLOCKS_PER_SEC, payoffend - payoffstart );
    }
}
