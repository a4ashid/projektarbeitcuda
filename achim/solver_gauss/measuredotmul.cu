#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include <time.h>

#include "dotMul_cuda_gpu.h"
#include "dotMul_cpu.h"
#include <time.h>

#define N_PROBLEM 50000
#define ITERSTEPS 2

__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    t_ve* v =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( v == NULL) {
	       fprintf(stderr, "sorry, can not allocate memory for you");
	       exit( -1 );
    }
    *M = v;

}



__host__ void malloc_vector_on_device(  t_ve** pV_d, unsigned int L ) {
	hipError_t e;

    e = hipMalloc ((void **) pV_d, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    hipMemset ( *pV_d, 0, sizeof(t_ve) * L );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on memset 0: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

}


int main()
{
    printf("\n measureing operation DOTMUL with N = %u, ITERSTEPS %u", N_PROBLEM, ITERSTEPS );

    t_ve* v1;
    t_ve* v2;

    t_ve* v1_d;
    t_ve* v2_d;

    t_ve* out ;  /* output vector */
    t_ve* out_d;

    int block_size = 512;


    clock_t startclocks;
    clock_t endclock;

    clock_t payoffstart;
    clock_t payoffend;


    clock_t startclockscpu;
    clock_t endclockscpu;

    hipError_t e;

    malloc_N( N_PROBLEM * 3 , &v1 );

    dim3 dimBlock(block_size);

    for ( unsigned int N = 1; N < N_PROBLEM; N *= 5 ) {
        payoffstart = clock();

        e = hipMalloc ((void **) &v1_d, sizeof(t_ve) * N * 3 );
        CUDA_UTIL_ERRORCHECK("hipMalloc &v1_d");

        v2_d  = &v1_d[N    ];
        out_d = &v1_d[N * 2];

        e = hipMemcpy(  v1_d, v1, sizeof(t_ve) * N * 2 , hipMemcpyHostToDevice);
        CUDA_UTIL_ERRORCHECK("hipMemcpy v1_d");

        payoffend = clock();
        startclocks = clock( );

        dim3 dimGrid ( N / block_size + 1 );

        for ( unsigned int i = 0; i < ITERSTEPS; i++ ) {

            device_dotMul<<<dimGrid,dimBlock>>>(v1_d, v2_d, out_d, N );
        	e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("Kernel device_dotMul")
        }
        endclock = clock( );

        e = hipFree(v1_d);
        CUDA_UTIL_ERRORCHECK("hipFree")

        /* ------------------------------------------------------------ */
		startclockscpu = clock( );

        v2  = &v1[N];
        out = &v1[N*2];

		for ( unsigned int i = 0; i < ITERSTEPS; i++ ) {
		    v1[1]++; /* ensure opearation is not answerde from cache */
		    dotMul_cpu(v1, v2, out, N );
		}
        endclockscpu   = clock( );
        /* ------------------------------------------------------------ */

        printf( "\n ----------------------------------------------------- \n N = %u", N );
	    printf( "\n GPU: %f seconds,  clocks: %u : CLOCKS_PER_SEC %u \n", ( (float) ( endclock - startclocks)) / CLOCKS_PER_SEC / ITERSTEPS, endclock - startclocks, CLOCKS_PER_SEC );
	    printf( "\n CPU: %f seconds,  clocks: %u : CLOCKS_PER_SEC %u \n", ( (float) ( endclockscpu - startclockscpu)) / CLOCKS_PER_SEC / ITERSTEPS, endclockscpu - startclockscpu, CLOCKS_PER_SEC );
	    printf( "\n cudamemcopy payoff %f secsonds (%u clocks)", (float) (payoffend - payoffstart) / CLOCKS_PER_SEC, payoffend - payoffstart );
    }
}
