
#include "projektcuda.h"

__host__ void matrixMul_cpu( t_ve* C, t_ve* A, t_ve* B, int mA, int nB)
{
	int i, j;
	for(i = 0; i < mA; i++){
        C[i]=0;
		for(j = 0; j < nB; j++){
			C[i] += A[i*nB+j] * B[j];
		}
	}
}


