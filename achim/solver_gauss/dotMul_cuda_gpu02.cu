#include "hip/hip_runtime.h"

#include "projektcuda.h"

#include <stdio.h>
#include <hip/device_functions.h>

__device__ unsigned int count = 0;
__shared__ bool isLastBlockDone;
__global__ void device_dotMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
	__shared__ float Cs[512];
	int idx = blockIdx.x*blockDim.x+threadIdx.x;

	Cs[threadIdx.x] = 0;

	if ( idx < N ) {
	    Cs[threadIdx.x] = in1[ idx ] * in2[ idx ];
	}

	t_ve blocksum = 0;

	__syncthreads();

	if(threadIdx.x==0){
	    for ( int i = 0; i < blockDim.x; i++ ) {
		     blocksum += Cs[i];
		}
		out[blockIdx.x]=blocksum ;
    }
     __threadfence();
    if(threadIdx.x==0){
        unsigned int value = atomicInc( &count, gridDim.x );
        isLastBlockDone = ( value == ( gridDim.x -1 ) );
    }
    __syncthreads();
    if(threadIdx.x==0){
         if ( isLastBlockDone ) {
             for ( int i = 1; i < gridDim.x; i++ ) {
                 out[0] += out[i];
             }
        }
    }
}

