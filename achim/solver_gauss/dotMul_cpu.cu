

#include "projektcuda.h"

__host__ void dotMul_cpu(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
	unsigned int i;

	out[0] = 0;

	for( i = 0; i < N; i++){
		out[0] += in1[i]*in2[i];
	}

}
