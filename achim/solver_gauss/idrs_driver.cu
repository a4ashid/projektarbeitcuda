#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "idrs.h"
#include <time.h>

#define N_PROBLEM 10000


__host__ void malloc_NN( unsigned int size_n, t_ve** M ) {

    t_ve*  v;
    v =  (t_ve*) malloc( sizeof(t_ve) * size_n  * size_n );
    if ( v == NULL) {
	       fprintf(stderr, "sorry, can not allocate memory for you");
	       exit( -1 );
    }
    *M = v;

}

__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    t_ve* v =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( v == NULL) {
	       fprintf(stderr, "sorry, can not allocate memory for you");
	       exit( -1 );
    }
    *M = v;

}

int main()
{
    clock_t startime;
    clock_t endtime;
    startime = clock( );

	   t_ve* A ; /* the problem */
	   t_ve* b ; /* the problems right side */



       t_ve* x0 ;

       t_ve* x ;  /* output vector */
       t_ve* resvec ;
       unsigned int iter;

       malloc_NN( N_PROBLEM , &A );
       malloc_N( N_PROBLEM  , &b );
       malloc_N( N_PROBLEM  , &x0 );
       malloc_N( N_PROBLEM  , &x );



       unsigned int N  =  N_PROBLEM;

       unsigned int j, i;

           for ( j = 1; j <= N_PROBLEM ; j++ ) {
               for ( i = 1; i <= N_PROBLEM; i++ ) {
                   A[ a(j,i) ] = 0;
                   if ( i == j ) {
                      A[ a(j,i) ] = 1;
                   }
                   if ( i == j + 1 ) {
                       A[ a(j,i) ] = 2;
                   }
               }
               b[ j -1 ] = j;
           }

   idrs(
                      A ,
                      b ,
                     20 ,  /* s iterations */
                     0.1,  /* tol          */
                     50,   /* masit        */
                     x0,

                     N_PROBLEM,

                      x,  /* output vector */
                      resvec,
                     &iter
                  );

       endtime = clock();

	   printf("idrs solver - testdriver");

	   printf( "\n %f seconds,  clocks: %u : CLOCKS_PER_SEC %u \n", ( (float) (endtime - startime)) / CLOCKS_PER_SEC, endtime - startime, CLOCKS_PER_SEC );
	}

