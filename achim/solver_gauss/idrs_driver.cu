
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "idrs.h"

#define N_PROBLEM 20000


__host__ void malloc_NN( unsigned int size_n, t_ve** M ) {

    t_ve*  v;
    v =  (t_ve*) malloc( sizeof(t_ve) * size_n  * size_n );
    if ( v == NULL) {
	       fprintf(stderr, "sorry, can not allocate memory for you");
	       exit( -1 );
    }
    *M = v;

}

__host__ void malloc_N( unsigned int size_n, t_ve** M ) {

    *M =  (t_ve*) malloc( sizeof(t_ve) * size_n  );
    if ( *M == NULL) {
	       fprintf(stderr, "sorry, can not allocate memory for you");
	       exit( -1 );
    }

}

int main()
{

	   t_ve* A = NULL; /* the problem */
	   t_ve* b = NULL; /* the problems right side */
       t_ve* s = NULL;


       t_ve* x0 = NULL;

       t_ve* x = NULL;  /* output vector */
       t_ve* resvec = NULL;
       unsigned int iter;

       malloc_NN( N_PROBLEM , &A );
       malloc_N( N_PROBLEM  , &b );

       unsigned int N  =  N_PROBLEM;

       unsigned int j, i;

           for ( j = 1; j <= N_PROBLEM ; j++ ) {
               for ( i = 1; i <= N_PROBLEM; i++ ) {
                   A[ a(j,i) ] = 0;
                   if ( i == j ) {
                      A[ a(j,i) ] = 1;
                   }
                   if ( i == j + 1 ) {
                       A[ a(j,i) ] = 2;
                   }
               }
               b[ j -1 ] = j;
           }

   idrs(
                      A ,
                      b ,
                      s ,
                     0.1, /* tol */
                     50,  /* masit  */
                     x0,

                     N_PROBLEM,

                      x,  /* output vector */
                      resvec,
                     &iter
                  );


	   printf("idrs solver - testdriver");
	}

