
#include <hip/hip_runtime.h>
#include "stdio.h"

/*  First Hello World, Achim Grolms, 2009-10-20  */
//  nvcc -deviceemu -o add_vector_emu add_vector.cu


typedef float        t_ve   ; // vector element, change this to double if required
typedef unsigned int t_vidx ; // index of vector elements

#define N 4000

__global__ void add_arrays_gpu( t_ve *in1, t_ve *in2, t_ve *out, t_vidx Ntot)
{
	t_vidx i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < Ntot )
		out[i] = in1[i] + in2[i];
}

int main()
{
	/* pointers to host memory */
	t_ve *a, *b, *c;
	/* pointers to device memory */
	t_ve *a_d, *b_d, *c_d;
//	t_vidx N=18;
	t_vidx i;

	/* Allocate arrays a, b and c on host*/
	a = (t_ve*) malloc(N*sizeof(t_ve));
	b = (t_ve*) malloc(N*sizeof(t_ve));
	c = (t_ve*) malloc(N*sizeof(t_ve));

	/* Allocate arrays a_d, b_d and c_d on device*/
	hipMalloc ((void **) &a_d, sizeof(t_ve)*N);
	hipMalloc ((void **) &b_d, sizeof(t_ve)*N);
	hipMalloc ((void **) &c_d, sizeof(t_ve)*N);

	/* Initialize arrays a and b */
	for (i=0; i<N; i++)
	{
		a[i] = (float) i;
		b[i] = (float) i;
	}


	/* Copy data from host memory to device memory */
	hipMemcpy(a_d, a, sizeof(t_ve)*N, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, sizeof(t_ve)*N, hipMemcpyHostToDevice);

	/* Compute the execution configuration */

	int block_size = 200;             // threads per block
    int grid_x     = 100;
    int grid_y     = 1;

	dim3 dimBlock(block_size);
	dim3 dimGrid ( grid_x, grid_y );        // threads = blocksize * gridx * grid y

	/* Add arrays a and b, store result in c */
	add_arrays_gpu<<<dimGrid,dimBlock>>>(a_d, b_d, c_d, N);

	/* Copy data from deveice memory to host memory */
	hipMemcpy(c, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

	/* Print c */
	for (i=0; i<N; i++)
		printf(" c[%d]=%f\n",i,c[i]);

	/* Free the memory */

    hipFree(a_d); hipFree(b_d);hipFree(c_d);
	free(a); free(b); free(c);


}

