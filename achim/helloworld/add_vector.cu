
#include <hip/hip_runtime.h>
#include "stdio.h"

/*  First Hello World, Achim Grolms, 2009-10-20  */
//  nvcc -deviceemu -o add_vector_emu add_vector.cu


typedef float        t_ve   ; // vector element, change this to double if required
typedef unsigned int t_vidx ; // index of vector elements

#define N 400000

__global__ void add_arrays_gpu( t_ve *in1, t_ve *in2, t_ve *out, t_vidx Ntot)
{
	t_vidx i = threadIdx.y * blockDim.x + threadIdx.x;
	if ( i < Ntot )
		out[i] = in1[i] + in2[i];
}

int main()
{

    hipError_t e;

    hipEvent_t start, stop;
    float time;

	/* pointers to host memory */
	t_ve *a, *b, *c;
	/* pointers to device memory */
	t_ve *a_d, *b_d, *c_d;
//	t_vidx N=18;
	t_vidx i;

    e = hipEventCreate(&start);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaEventCreate: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipEventCreate(&stop);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaEventCreate: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

	/* Allocate arrays a, b and c on host*/
	a = (t_ve*) malloc(N*sizeof(t_ve));
	b = (t_ve*) malloc(N*sizeof(t_ve));
	c = (t_ve*) malloc(N*sizeof(t_ve));

	/* Allocate arrays a_d, b_d and c_d on device*/
	e = hipMalloc ((void **) &a_d, sizeof(t_ve)*N);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
	e = hipMalloc ((void **) &b_d, sizeof(t_ve)*N);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
	e = hipMalloc ((void **) &c_d, sizeof(t_ve)*N);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMalloc: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

	/* Initialize arrays a and b */
	for (i=0; i<N; i++)
	{
		a[i] = (float) i;
		b[i] = (float) i;
        c[i] = (float) -1111;
	}


	/* Copy data from host memory to device memory */
	e = hipMemcpy(a_d, a, sizeof(t_ve)*N, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
	e = hipMemcpy(b_d, b, sizeof(t_ve)*N, hipMemcpyHostToDevice);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

	/* Compute the execution configuration */

	int block_size = 512;             // threads per block
    int grid_x     =  N / block_size + 1 ;
    int grid_y     =  1;

	dim3 dimBlock(block_size);
	dim3 dimGrid ( grid_x, grid_y );        // threads = blocksize * gridx * grid y

	/* Add arrays a and b, store result in c */

    e = hipEventRecord( start, 0 );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaEventRecord: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

	add_arrays_gpu<<<dimGrid,dimBlock>>>(a_d, b_d, c_d, N);
    e = hipGetLastError();
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on add_arrays_gpu: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipEventRecord( stop, 0 );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaEventRecord: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipEventSynchronize( stop );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaEventRecord: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipEventElapsedTime( &time, start, stop );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaEventElapsedTime: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

	/* Copy data from deveice memory to host memory */
	e = hipMemcpy(c, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

	/* Print c */
//	for (i=0; i<N; i++)
//		printf(" c[%d]=%f\n",i,c[i]);

	/* Free the memory */

    e = hipFree(a_d);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipFree(b_d);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipFree(c_d);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on cudaMemcpy: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
	free(a); free(b); free(c);

    printf( "kernel runtime (size %d): %f milliseconds\n", N , time );
}

