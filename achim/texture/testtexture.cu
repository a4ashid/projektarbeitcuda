#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"


texture<float,1, hipReadModeElementType> texRef;


int MSIZE = 1000000;

__global__ void kernel_testtex() {


	float bla = tex1Dfetch( texRef , 0 );
}



int main()
{


    printf("\n test texture \n");

    hipError_t e;

    hipChannelFormatDesc channelDesc
        = hipCreateChannelDesc( 32, 0, 0, 0, hipChannelFormatKindFloat );

    /* hipChannelFormatKindNone */

    float* dev_mem;


    e = hipMalloc ((void **) &dev_mem, sizeof(double) * MSIZE );
    CUDA_UTIL_ERRORCHECK("hipMalloc")



     e = hipBindTexture (
		               NULL ,      /*size_t  offset,*/
                       texRef,     /*const struct textureReference  texref */
                       dev_mem,    /* const void devPtr, */
                       channelDesc,
                       sizeof(double) * MSIZE
                     );

     CUDA_UTIL_ERRORCHECK("hipBindTexture")


     dim3 dimGrid ( 2 );
     dim3 dimBlock(32);

     kernel_testtex<<<dimGrid,dimBlock>>>();

     e = hipGetLastError();
     CUDA_UTIL_ERRORCHECK("kernel_testtex");

}