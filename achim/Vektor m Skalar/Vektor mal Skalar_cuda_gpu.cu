#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */
__global__ void device_skalaMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
 
__shared__ float vOut[16];
int idx = blockIdx.x*blockDim.x+threadIdx.x;

if ( idx < N)vOut[idx] = in1[idx]*in2[idx];

__syncthreads();

if(idx == 0) {
    out[0] = 0;
	int i;
	for ( i = 0; i < N; i++ ) {
	   out[0] += vOut[i];
	}
}

__syncthreads();

}
void host_skalaMul(double* pIn1, double* pIn2,double *pOut, int sizeIn, int sizeOut)
{

int i, j;
double *pdata_in1_d, *pdata_in2_d, *pdata_out_d;
t_ve *pdata_in1_f, data_in2_f, *pdata_out_f;
t_ve *pdata_in1_f_gpu, *pdata_out_f_gpu;
int sizeBlock;
sizeBlock = 16;

/* get Input data pointer */
pdata_in1_d = pIn1;
pdata_in2_d = pIn2;
/* get Ouput data pointer */
pdata_out_d = pOut;


/* Find the dimensions of the data */

/* Create an mxArray for the output data */

/* Create an input and output data array on the GPU*/
hipMalloc( (void **) &data_in1_f_gpu,sizeof(t_ve)*sizeIn);

hipMalloc( (void **) &data_out_f_gpu,sizeof(t_ve)*sizeOut);
/* Retrieve the input data */

/* Check if the input array is single or double precision */

/* The input array is in double precision, it needs to be converted t
floats before being sent to the card */
data_in1_f = (t_ve *) malloc(sizeof(t_ve)*sizeIn);
data_in2_f = (t_ve *) malloc(sizeof(t_ve)*1);

data_out_f = (t_ve *) malloc(sizeof(t_ve)*sizeOut);

for (j = 0; j < sizeIn; j++)
{
data_in1_f[j] = (t_ve) data_in1_d[j];

}
data_in2_f = (t_ve) data_in2_d[0];


hipMemcpy( data_in1_f_gpu, data_in1_f, sizeof(t_ve)*sizeIn, hipMemcpyHostToDevice);

/* Compute execution configuration using 128 threads per block */
dim3 dimBlock(sizeBlock);
dim3 dimGrid((sizeIn)/dimBlock.x);
if ( (sizeIn) % sizeBlock !=0 ) dimGrid.x+=1;
    
/* Call function on GPU */
device_skalarMul<<<dimGrid,dimBlock>>>(data_in1_f_gpu, data_in2_f_gpu, data_out_f_gpu, sizeIn);

hipError_t e;
e = hipGetLastError();
if ( e != hipSuccess)
{
    fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
    exit(-1);
}

/* Copy result back to host */
hipMemcpy( data_out_f, data_out_f_gpu, sizeof(float)*sizeOut, hipMemcpyDeviceToHost);
    for (i = 0; i < sizeOut; i++)
    {
        printf("data_out_f[%d] = %f, ", i, data_out_f[i]);
    }
        printf("\n");


/* Create a pointer to the output data */

/* Convert from single to double before returning */
for (j = 0; j < sizeOut; j++)
{
data_out_d[j] = (double) data_out_f[j];
}
/* Clean-up memory on device and host */
free(data_in1_f);
free(data_in2_f);
free(data_out_f);
hipFree(data_in1_f_gpu);
hipFree(data_in2_f_gpu);
hipFree(data_out_f_gpu);
}

int main()
{

    double *pIn1, *pIn2,*pOut;
    int sizeIn, sizeOut;
    int i;
    sizeIn = 3;
    sizeOut = sizeIn;
    pIn1 = (double*)malloc(sizeof(double)*sizeIn);
	pIn2 = (double*)malloc(sizeof(double)*1);
    pOut = (double*)malloc(sizeof(double)*sizeOut);
    pIn1[0] = 1;
    pIn1[1] = 2;
    pIn1[2] = 3;
	pIn2[0] = 1;
    host_dotMul(pIn1, pIn2, pOut, sizeIn, sizeOut);
	
	printf("output square result");
    for (i = 0; i < sizeOut; i++)
    {	
        printf(" pOut[%d] = %lf, ", i, pOut[i]);
    }
        printf("\n");
	printf("output norm result");
    for (i = 0; i < sizeOut; i++)
    {
		pOut[i] = sqrt(pOut[i]);
        printf("squre of pOut[%d] = %lf, ", i, pOut[i]);
    }
        printf("\n");
    
   
    free(pIn1);
	free(pIn2);
    free(pOut);
    return 0;
}
