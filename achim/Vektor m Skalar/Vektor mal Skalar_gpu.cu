#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */
// 

__global__ void device_skalarMul(t_ve* pin1, t_ve in2,t_ve* out, unsigned int N)
{
	
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	out[idx] = 0;
	__syncthreads();
	
	if ( idx < N)out[idx] = pin1[idx]*in2;
	__syncthreads();
	
}

