#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

typedef int        t_mindex;

typedef struct SparseMatrix{

    t_mindex  cnt_elements;
    t_mindex  cnt_colums;

    t_mindex* ir;
    t_ve*     pr;
    t_mindex* jc;

} t_SparseMatrix;

// ---------------------------------------------------------------------
__global__ void kernel_sparse( t_SparseMatrix m ) {


    t_ve sum = 0;
    if ( threadIdx.x ==  0) {
       for ( int i = 0; i < m.cnt_elements; i++ ) {
          sum += m.pr[i];
       }
       printf("got sum: %f",  sum);
    }
}
// ---------------------------------------------------------------------
__host__ void set_sparse_data( t_SparseMatrix* m, void* mv ) {

   m->ir = (t_mindex *) mv;
   m->pr = (t_ve *) (&m->ir[m->cnt_elements] ) ;
   m->jc = (t_mindex *) (&m->pr[m->cnt_elements]);

}
// ---------------------------------------------------------------------
__host__ void dump_sparse_matrix( t_SparseMatrix m ) {

    printf( "\n cols: %u elements: %u \n",  m.cnt_colums, m.cnt_elements );

//    for ( t_mindex i = 0; i <= m.cnt_colums; i++ ) {
//        printf("\n %u\t%u", i, m.jc[i] );
//    }
    for ( t_mindex i = 0; i < m.cnt_elements; i++ ) {
        printf("\n%u\t%f", m.ir[i], m.pr[i] );
        if ( i <= m.cnt_colums ) {
            printf("\t %u", m.jc[i] );
        }
    }
    printf("\n\n");
}
// ---------------------------------------------------------------------
__host__ int smat_size( int cnt_elements, int cnt_cols ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * cnt_elements
           + sizeof(t_mindex)  * (cnt_cols + 1);
}
// ---------------------------------------------------------------------
int main()

{

    t_SparseMatrix host_m, device_m;

    host_m.cnt_elements = 6;
    host_m.cnt_colums   = 3;

    device_m.cnt_elements = host_m.cnt_elements;
    device_m.cnt_colums   = host_m.cnt_colums;

    printf("\n Testting sparse basics \n");

    int msize = smat_size( host_m.cnt_elements, host_m.cnt_colums );

    printf(" got result %u \n", msize);

    void *hostmem =   malloc( msize );
    if ( hostmem == NULL) {
           fprintf(stderr, "sorry, can not allocate memory for you");
           exit( -1 );
    }

   /* ---------------------------------- */



//   host_m.ir = (t_mindex *) hostmem;
//   host_m.pr = (t_ve *) (&host_m.ir[6]);
//   host_m.jc  = (t_mindex *) (&host_m.pr[6]);

   set_sparse_data( &host_m, hostmem);

   host_m.jc[0] = 0;
   host_m.jc[1] = 2;
   host_m.jc[2] = 3;
   host_m.jc[3] = 6;

   host_m.ir[0] = 1;
   host_m.ir[1] = 4;
   host_m.ir[2] = 2;
   host_m.ir[3] = 1;
   host_m.ir[4] = 4;
   host_m.ir[5] = 5;

   host_m.pr[0] = 1;
   host_m.pr[1] = 1;
   host_m.pr[2] = 1;
   host_m.pr[3] = 2;
   host_m.pr[4] = 1;
   host_m.pr[5] = 1;


   dump_sparse_matrix( host_m );


   void *devicemem;
   hipError_t e;

   e = hipMalloc ( &devicemem, msize );
   CUDA_UTIL_ERRORCHECK("hipMalloc")

   e = hipMemcpy(  devicemem, hostmem, msize , hipMemcpyHostToDevice);
   CUDA_UTIL_ERRORCHECK("hipMemcpy")

   set_sparse_data( &device_m, devicemem);

   dim3 dimGrid ( 1 );
   dim3 dimBlock(32);


   kernel_sparse<<<dimGrid,dimBlock>>>( device_m );
   e = hipGetLastError();
   CUDA_UTIL_ERRORCHECK("summup_kernel_kernel_dotmul");


}

