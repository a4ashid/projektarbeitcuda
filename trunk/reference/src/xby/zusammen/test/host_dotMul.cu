#include "test_comm.h"

void host_dotMul(double* pIn1, double* pIn2,double *pOut, int sizeIn, int sizeOut)
{

int i, j;
double *data_in1_d, *data_in2_d, *data_out_d;
float *data_in1_f, *data_in2_f, *data_out_f;
float *data_in1_f_gpu, *data_in2_f_gpu , *data_out_f_gpu;
int sizeBlock;
sizeBlock = VECTOR_BLOCK_SIZE;

// get Input data pointer
data_in1_d = pIn1;
data_in2_d = pIn2;
// get Ouput data pointer
data_out_d = pOut;


// Find the dimensions of the data 

// Create an mxArray for the output data 

// Create an input and output data array on the GPU
hipMalloc( (void **) &data_in1_f_gpu,sizeof(t_ve)*sizeIn);
hipMalloc( (void **) &data_in2_f_gpu,sizeof(t_ve)*sizeIn);
hipMalloc( (void **) &data_out_f_gpu,sizeof(t_ve)*sizeOut);
// Retrieve the input data 

// Check if the input array is single or double precision 

// The input array is in double precision, it needs to be converted t floats before being sent to the card 
data_in1_f = (t_ve *) malloc(sizeof(t_ve)*sizeIn);
data_in2_f = (t_ve *) malloc(sizeof(t_ve)*sizeIn);

data_out_f = (t_ve *) malloc(sizeof(t_ve)*sizeOut);

for (j = 0; j < sizeIn; j++)
{
data_in1_f[j] = (t_ve) data_in1_d[j];
data_in2_f[j] = (t_ve) data_in2_d[j];
}
    for (i = 0; i < sizeIn; i++)
    {
     //    printf("data_in1_f[%d] = %f, ", i, data_in1_f[i]);
    }
        printf("\n");

hipMemcpy( data_in1_f_gpu, data_in1_f, sizeof(t_ve)*sizeIn, hipMemcpyHostToDevice);
hipMemcpy( data_in2_f_gpu, data_in2_f, sizeof(t_ve)*sizeIn, hipMemcpyHostToDevice);


//hipMemcpy( data2f_gpu, data2f, sizeof(float)*sizeOut, hipMemcpyHostToDevice);

// Compute execution configuration using 128 threads per block 
dim3 dimBlock(sizeBlock);
dim3 dimGrid((sizeIn)/dimBlock.x);

if ( (sizeIn) % sizeBlock !=0 ) dimGrid.x+=1;
    
//Call function on GPU 
device_dotMul<<<dimGrid,dimBlock>>>(data_in1_f_gpu, data_in2_f_gpu, data_out_f_gpu, sizeIn);
hipError_t e;
e = hipGetLastError();
if ( e != hipSuccess)
{
    fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
    exit(-1);
}

// Copy result back to host 
hipMemcpy( data_out_f, data_out_f_gpu, sizeof(float)*sizeOut, hipMemcpyDeviceToHost);
    for (i = 0; i < sizeOut; i++)
    {
     //   printf("data_out_f[%d] = %f, ", i, data_out_f[i]);
    }
        printf("\n");


// Create a pointer to the output data 

// Convert from single to double before returning 
for (j = 0; j < sizeOut; j++)
{
data_out_d[j] = (double) data_out_f[j];
}
// Clean-up memory on device and host 
free(data_in1_f);
free(data_in2_f);
free(data_out_f);
hipFree(data_in1_f_gpu);
hipFree(data_in2_f_gpu);
hipFree(data_out_f_gpu);
}     
     
int test_dotMul()
{
    double *pIn1, *pIn2,*pOut;
    int sizeIn, sizeOut;
    int i;
	
	double expect;
	int loop;
	for (loop = 9000; loop < 10000; loop++) {
	int expect_error = 0;
    sizeIn = loop;
    //sizeOut =3;
	sizeOut =sizeIn/VECTOR_BLOCK_SIZE + 1;
    pIn1 = (double*)malloc(sizeof(double)*sizeIn);
    pIn2 = (double*)malloc(sizeof(double)*sizeIn);
    pOut = (double*)malloc(sizeof(double)*sizeOut);
    for (i = 0; i < sizeIn; i++){
		pIn1[i] = 1;
		pIn2[i] = 1;
	}
	/*
	pIn1[0] = 1;
    pIn1[1] = 2;
    pIn1[2] = 3;
    pIn2[0] = 1;
    pIn2[1] = 2;
    pIn2[2] = 3;
	*/
    host_dotMul(pIn1, pIn2, pOut, sizeIn, sizeOut);
	expect=sizeIn;
	printf("output square result");
	
	if(pOut[0] != expect){
		
		for (i = 0; i < sizeOut; i++)
		{	
        printf(" pOut[%d] = %lf, ", i, pOut[i]);
		}
		
		expect_error = loop;
		printf(" pOut[0] = %lf,  ", pOut[0]);
        printf("\n");
		printf("expect error = %d,\n",expect_error);
		
	}
	/*
		expect_error = loop;
		printf(" pOut[0] = %lf,  ", pOut[0]);
        printf("\n");
		printf("expect error = %d,\n",expect_error);
*/

    free(pIn1);
    free(pIn2);
    free(pOut);
	}
    return 0;

}

