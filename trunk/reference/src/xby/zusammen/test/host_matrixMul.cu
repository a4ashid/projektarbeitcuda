#include "test_comm.h"

void host_matrixMul(double* pC, double* pA,double *pB, int mA, int nB)
{

int i, j;
double *data_in1_d, *data_in2_d, *data_out_d;
float *data_in1_f, *data_in2_f, *data_out_f;
float *data_in1_f_gpu, *data_in2_f_gpu , *data_out_f_gpu;
int sizeBlock;
sizeBlock = VECTOR_BLOCK_SIZE;
int sizeA = mA*nB;
int sizeB = nB;
int sizeC = mA;

// get Input data pointer
data_in1_d = pA;
data_in2_d = pB;
// get Ouput data pointer
data_out_d = pC;


// Find the dimensions of the data 

// Create an mxArray for the output data 

// Create an input and output data array on the GPU
hipMalloc( (void **) &data_in1_f_gpu,sizeof(t_ve)*sizeA);
hipMalloc( (void **) &data_in2_f_gpu,sizeof(t_ve)*sizeB);
hipMalloc( (void **) &data_out_f_gpu,sizeof(t_ve)*sizeC);
// Retrieve the input data 

// Check if the input array is single or double precision 

// The input array is in double precision, it needs to be converted t floats before being sent to the card 
data_in1_f = (t_ve *) malloc(sizeof(t_ve)*sizeA);
data_in2_f = (t_ve *) malloc(sizeof(t_ve)*sizeB);

data_out_f = (t_ve *) malloc(sizeof(t_ve)*sizeC);

	for (j = 0; j < sizeA; j++)
	{
		data_in1_f[j] = (t_ve) data_in1_d[j];
	}
	for (j = 0; j < sizeB; j++)
	{
		data_in2_f[j] = (t_ve) data_in2_d[j];
	}
    for (i = 0; i < sizeA; i++)
    {
       // printf("data_in1_f[%d] = %f, ", i, data_in1_f[i]);
    }
        printf("\n");

	hipMemcpy( data_in1_f_gpu, data_in1_f, sizeof(t_ve)*sizeA, hipMemcpyHostToDevice);
	hipMemcpy( data_in2_f_gpu, data_in2_f, sizeof(t_ve)*sizeB, hipMemcpyHostToDevice);




// Compute execution configuration using 128 threads per block 
dim3 dimBlock(sizeBlock);
//dim3 dimGrid((sizeIn)/dimBlock.x);
dim3 dimGrid(mA);
//if ( (sizeA) % sizeBlock !=0 ) dimGrid.x+=1;
    
//Call function on GPU 
matrixMul<<<dimGrid,dimBlock>>>(data_out_f_gpu,data_in1_f_gpu, data_in2_f_gpu, mA,nB);
hipError_t e;
e = hipGetLastError();
if ( e != hipSuccess)
{
    fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
    exit(-1);
}

// Copy result back to host 
hipMemcpy( data_out_f, data_out_f_gpu, sizeof(float)*sizeC, hipMemcpyDeviceToHost);
    for (i = 0; i < sizeC; i++)
    {
    //    printf("data_out_f[%d] = %f, ", i, data_out_f[i]);
    }
     //   printf("\n");


// Create a pointer to the output data 

// Convert from single to double before returning 
for (j = 0; j < sizeC; j++)
{
data_out_d[j] = (double) data_out_f[j];
}
// Clean-up memory on device and host 
free(data_in1_f);
free(data_in2_f);
free(data_out_f);
hipFree(data_in1_f_gpu);
hipFree(data_in2_f_gpu);
hipFree(data_out_f_gpu);
}     
     
int test_matrixMul()
{
    double *pA, *pB,*pC;
    int mA, nB;
    int i;
	double expect;
	int loop;
	for (loop = 256; loop < 260; loop++) {
	int expect_error = 0;
    mA = loop;
    nB = loop;
    pA = (double*)malloc(sizeof(double)*mA*nB);
    pB = (double*)malloc(sizeof(double)*nB);
    pC = (double*)malloc(sizeof(double)*mA);
    for (i = 0; i < mA*nB; i++){
		pA[i] = 1;
	}
	for (i = 0; i < nB; i++){
		pB[i] = 1;
	}

    host_matrixMul(pC,pA, pB, mA, nB);
	
	expect = (double) nB;
	printf("output square result");
    for (i = 0; i < nB; i++)
    {	if(pC[i] != expect)
        printf(" pC[%d] = %lf, ", i, pC[i]);
		expect_error = loop;
    }
        printf("\n");
		printf("expect error = %d,\n",expect_error);
	
		

    free(pA);
    free(pB);
    free(pC);
	}

    return 0;
 
}

