#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

//#include "mex.h"
/* Kernel to square elements of the array on the GPU */

/*
  create in 28.11.2009
 It should be runging only in one block.
 separate vectors in parts in length of VECTOR_BLOCK_SIZE

*/

/*

N size of Vector  
*/

__global__ void device_dotMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if(idx > N) return;
	if(idx == 0)out[idx] = 0;
	__syncthreads();
	
   //block index
   int blockRow = blockIdx.x;
   // thread index
   int row = threadIdx.x;
   //int aBegin = blockRow*VECTOR_BLOCK_SIZE;   
   //int aEnd = aBegin + VECTOR_BLOCK_SIZE - 1;
   //working only in one block
   int aBegin = 0;
   int aEnd = N;
   
   int aStep = VECTOR_BLOCK_SIZE;
   //
   
   
   // comupted by the thread
   t_ve outValue = 0;
  
   //for (int a = aBegin;(a <= aEnd)&&(a <= N);a += aStep){
   for (int a = aBegin;a <= aEnd;a += aStep){
         // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[VECTOR_BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[VECTOR_BLOCK_SIZE];
		
		__shared__ float Cs[VECTOR_BLOCK_SIZE];
		Cs[row] = 0;

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        AS(row) = in1[a + row];
        BS(row) = in2[a + row];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();    
		
		Cs[row] = AS(row) * BS(row);
        
		/*
        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
		for (int k = 0; (k < VECTOR_BLOCK_SIZE)&&(k < N); ++k)
        //for (int k = 0; (k < VECTOR_BLOCK_SIZE); ++k)
            outValue += AS(k) * BS(k);
		*/
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();  
		
		// computing summe in one thread for one Loop 
		if (row == 0) {		
            //for (int k = 0; (k < VECTOR_BLOCK_SIZE)&&(idx < N); k++)
			//out[blockIdx.x] += Cs[k];
			
			for (int k = 1; (k < VECTOR_BLOCK_SIZE)&&(idx < N); k++){
                 Cs[0]+= Cs[k];
            }
            outValue += Cs[0]
			//outValue += 1;
		}
		__syncthreads();
   }
   //__syncthreads();
   
   //if(idx==0){
	//	for(int k = 1; k <= gridDim.x; k++)out[0] += out[k];
   //}
   
   out[0] = outValue;

}

