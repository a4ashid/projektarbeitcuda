#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */

/*
typedef struct{
        int width;
        int height;
        int stride;
        float * elements;
} Matrix;
///
__device__ float GetVectorElement(const Matrix A, int row, int offset){
          return A.elements[row * VECTOR_BLOCK_SIZE + offset];           
}

///??????????????????
__device__ void setVectorElement(Matrix A, int row, int offset, float value){
           A.elements[row * VECTOR_BLOCK_SIZE + offset] = value;           
}

__device__ Matrix GetSubVector(Matrix A, int row){
           Matrix Asub;
           Asub.width = 1;     
           Asub.height = VECTOR_BLOCK_SIZE;
           Asub.stride = 1;
           Asub.elements = & A.elements[row * VECTOR_BLOCK_SIZE]      
}
*/
/*
N size of Vector  
*/

__global__ void device_dotMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
   //block index
   int blockRow = blockIdx.x;
   // thread index
   int row = threadIdx.x;
   int aBegin = blockRow*VECTOR_BLOCK_SIZE;
   int aEnd = aBegin + VECTOR_BLOCK_SIZE - 1;
   int aStep = VECTOR_BLOCK_SIZE;
   //
   // comupted by the thread
   t_ve outValue = 0;
   
   for (int a = aBegin;a <= aEnd;a += aStep){
         // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[VECTOR_BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[VECTOR_BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        AS(row) = in1[a + row];
        BS(row) = in2[a + row];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();    
        
        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < VECTOR_BLOCK_SIZE; ++k)
            outValue += AS(k) * BS(k);

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();  
   }
   
   out[0] = outValue;


//	__syncthreads();

}

