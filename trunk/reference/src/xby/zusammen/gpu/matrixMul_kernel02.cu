#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

/* Kernel to square elements of the array on the GPU */
/*
	Matrix A is mA x nB  , Vector B is nB
	Vector C output vector in size of mA
	C=A*B
description:
	each row of A occuppy one block. if gridDim is smaller than the row number of A  
*/

__global__ void matrixMul( t_ve* C, t_ve* A, t_ve* B, int mA, int nB)
{
	

	//define a Result Vector for each block
	__shared__ float Cs[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 512
	
	//define gridIndex, if gridDim < mA, gridIndex > 0; 
	int gridIndex = 0;
	// get a thread indentifier
	int idx = blockIdx.x*blockDim.x+threadIdx.x;//int idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
	int aBegin = 0;
	int bBegin = 0;
	int aStep = gridDim.x;
	int bStep = VECTOR_BLOCK_SIZE; // blockDim.x
	int aEnd = mA;

		//initialize output vector for each block
		if(threadIdx.x==0){
			C[gridIndex*gridDim.x+blockIdx.x]=0;
		}
		__syncthreads();
	
	// if nB > gridDim???????
	for(int a = aBegin; (a < aEnd)&&(idx < mA*nB); a += aStep, gridIndex++){
		
		//following is operations within one block 
		// initialize the dot product for each row in A and vector B
		t_ve blocksum = 0;
		//if nB> blockDim, split repeat the
		for(int b = bBegin; (b < nB)&&((threadIdx.x+b) < nB); b += bStep ) {
			//initialise Cs 
			Cs[threadIdx.x] = 0;
			__syncthreads();
			// compute scalar product
			if (( idx < mA*nB )&&(threadIdx.x < nB)) {
				//Cs[threadIdx.x] = A[a + blockIdx.x ][b + threadIdx.x] * B[b + threadIdx.x ];
				Cs[threadIdx.x] = A[(a + blockIdx.x)* nB+b + threadIdx.x] * B[b + threadIdx.x ];
			}
			__syncthreads();
				
			if(threadIdx.x==0){
				for (int k = 1; k < VECTOR_BLOCK_SIZE; k++) Cs[0] += Cs[k];
				blocksum += Cs[0];
			}
			__syncthreads();
			
			Cs[threadIdx.x] = 0;
			__syncthreads();
			
		}
		__syncthreads();

		if(threadIdx.x == 0) C[gridIndex*gridDim.x+blockIdx.x] = blocksum;
		__syncthreads();
		// summe all block
	
	}


}