#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

/* Kernel to square elements of the array on the GPU */
/*
	Matrix A is mA x nB  , Vector B is nB
	Vector C output vector in size of mA
	C=A*B
description:
	each row of A occuppy one block. if gridDim is smaller than the row number of A  
*/

__global__ void matrixMul( t_ve* C, t_ve* A, t_ve* B, int mA, int nB)
{
	
	//define a Result Vector for each block
	__shared__ float Cs[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 512
	
	//define gridIndex, if gridDim < mA, gridIndex > 0; 
	int gridIndex = 0;
	// get a thread indentifier
	//int idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
	int aBegin = 0;
	int bBegin = 0;
	int aStep = gridDim.x;
	int bStep = VECTOR_BLOCK_SIZE; // blockDim.x
	int aEnd = mA;
	int bEnd = nB;

		//initialise Cs 
		Cs[threadIdx.x] = 0;
		__syncthreads();
		//initialize output vector for each block
	if(threadIdx.x==0){
		C[gridIndex*gridDim.x+blockIdx.x]=0;
	}
		__syncthreads();
	// if nB > gridDim???????
	//idx < (gridIndex*gridDim.x+mA%VECTOR_BLOCK_SIZE)*()
	for(int a = aBegin; (a < aEnd)&&((gridIndex*gridDim.x+blockIdx.x)<aEnd); a += aStep, gridIndex++){
		//initialize output vector for each block
		if(threadIdx.x==0){
			C[gridIndex*gridDim.x+blockIdx.x]=0;
		}
		__syncthreads();
		
		//following is operations within one block 
		// initialize the dot product for each row in A and vector B
		t_ve blocksum = 0;
		//if nB> blockDim, split repeat the
		for(int b = bBegin; (b < bEnd)&&((threadIdx.x+b) < bEnd); b += bStep ) {
			//initialise Cs 
			Cs[threadIdx.x] = 0;
			__syncthreads();
			// compute scalar product
			if (( (gridIndex*gridDim.x+blockIdx.x)<aEnd)&&((b+threadIdx.x) < bEnd)) {
				//Cs[threadIdx.x] = A[a + blockIdx.x ][b + threadIdx.x] * B[b + threadIdx.x ];
				Cs[threadIdx.x] = A[(a + blockIdx.x)* nB+b + threadIdx.x] * B[b + threadIdx.x ];
			}
			__syncthreads();
				
			if(threadIdx.x == 0){
				//30.Nov.2009 fixeded for Cs summe
				int kEnd = bEnd-b;
				if(kEnd > VECTOR_BLOCK_SIZE)kEnd = VECTOR_BLOCK_SIZE;
				//Because I add Cs[0...k], if blockSize and Matrix does not fit, Parts of Cs[k] are not initialized as 0.  		
				for (int k = 0; k < kEnd; k++) blocksum += Cs[k];
			
			}
			__syncthreads();
			
			//Cs[threadIdx.x] = 0;
			//__syncthreads();	
		}//for b
		__syncthreads();

		if(threadIdx.x == 0) C[gridIndex*gridDim.x+blockIdx.x] = blocksum;
		__syncthreads();
		// summe all block, need test for mA bigger than one Grid
		//idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
	
	}

}