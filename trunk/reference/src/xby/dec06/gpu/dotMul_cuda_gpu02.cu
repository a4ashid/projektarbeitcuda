#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */

__global__ void device_dotMul(t_ve* in1, t_ve* in2,t_ve* out, unsigned int N)
{
	//define a Result Vector for each block
	__shared__ float Cs[VECTOR_BLOCK_SIZE];
	// get a thread indentifier
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	//initialise Cs
	Cs[threadIdx.x] = 0;
	// compute scalar product
	if ( idx < N ) {
	    Cs[threadIdx.x] = in1[ idx ] * in2[ idx ];
	}

	t_ve blocksum = 0;
	
	//initialize output vector for each block
	if(threadIdx.x==0){
		out[blockIdx.x]=0;
	}
	__syncthreads();
	
	//compute summe of all thread's results for each block 
	if(threadIdx.x==0){
	    for ( int i = 0; i < blockDim.x; i++ ) {
		     blocksum += Cs[i];
		}
		out[blockIdx.x]=blocksum;
	}
	__syncthreads();
	
	//compute the sume of all block's result for the grid
	
	if ( idx == 0 ) {
	     for ( int i = 1; i < gridDim.x; i++ ) {
		     out[0] += out[i];
		 }
	}

}