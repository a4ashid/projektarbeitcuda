#include "hip/hip_runtime.h"
/*
mexUnitTestMain.cu
*/
#include <stdio.h>
#include "mex.h"
#include "host_dotMul.cu"
#include "host_norm.cu"
#include "host_matrixMul.cu"

void callTestFunction(double** ppIn,int *pmIn,int *pnIn, int callFuncType,double** ppOut){
//call gpu
//testdotMul
	//printf("in callTestFunction \n");

	switch(callFuncType){
		case 0://dotMul
			if((pmIn[0]==pmIn[1])&&(pnIn[0]==pnIn[1])){
				if(1==pnIn[0])mexTest_dotMul(ppIn[0],ppIn[1],ppOut[0],pmIn[0]);
			}
			break;
		case 1://norm
			//printf("call norm \n");	
				if(1==pnIn[0])mexTest_norm(ppIn[0],ppOut[0],pmIn[0]);
		break;
		case 2: //matrixMul
			//test matrixMul
			//ppIn[0]:matrix A, ppIn[1]: vector B
			//pmIn[0]=mA pmIn[1]=nB, pnIn[0]=nB,pnIn[1]=1
			if((pmIn[1]==pnIn[0])&&(1==pnIn[1])){
			//mexTest_matrixMul(double* pC, double *pA,double *pB, int mA, int nB);
				mexTest_matrixMul(ppOut[0],ppIn[0],ppIn[1],pmIn[0], pmIn[1]);
			}
		default:  
			if((pmIn[0]==pmIn[1])&&(pnIn[0]==pnIn[1])){
				if(1==pnIn[0])mexTest_dotMul(ppIn[0],ppIn[1],ppOut[0],pmIn[0]);
			}
	}

	//printf(DOTMUL_CPU);
	//printf("\n");
	//printf("pnIn[0]=%d,pmIn[0]=%d, \n",pnIn[0],pmIn[0]);
}
/* Gateway function */
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{
     //int inputArgNum = nrhs;
     //int outputArgNum = nlhs;
     int i,j,k,m, n, retNum, outNum;
     double *pMatrix;
     double * pIn, * pOut;  
     double ** ppIn, **ppOut;
     int *pmIn;
     int *pnIn;
	 outNum=0;
     pnIn = (int*)mxMalloc(sizeof(int)*nrhs);
     pmIn = (int*)mxMalloc(sizeof(int)*nrhs);
     ppIn = (double**)mxMalloc(sizeof(double*)*nrhs);
	 ppOut = (double**)mxMalloc(sizeof(double*)*1);
     //printf("nrhs = %d \n",nrhs);
     for (i = 0; i < nrhs; i++){
        /* Find the dimensions of the data */
        m = mxGetM(prhs[i]);
        n = mxGetN(prhs[i]);
        //printf("m = %d , n= %d \n",m,n);
        pmIn[i] = (int)m;
        pnIn[i] = (int)n;
        pMatrix = mxGetPr(prhs[i]); 
        pIn = (double*)mxMalloc(sizeof(double)*m*n);       
        ppIn[i]=pIn;
        
        for( k = 0; k < m; k++)
            for( j = 0; j < n; j++){
                pIn[k*n+j] = (double)pMatrix[j*m+k];
			//	printf("%f \n",pIn[k*n+j]);
            }
       
     }// for i
	 
	 	 if((nrhs > 0)){
		pMatrix = mxGetPr(prhs[nrhs-1]);
		retNum = (int)pMatrix[0];
		//printf("retNum = %d \n",retNum);
		switch (retNum){
			case 0://doutMul
				outNum = 1;
			break;
			case 1://norm
				outNum = 1;
			break;
			case 2://matrixMul
				outNum = pmIn[0];
			break;
			default://0
				outNum = 1;
		}
		if(outNum < 1) {
			printf("please define function Type! \ n");
			printf("0:dotMul,1:norm, 2: matrixMul \ n");
			return;
		}
		
		//printf("before mxCreateDoubleMatrix \n");
		plhs[0] = mxCreateDoubleMatrix(outNum,1,mxREAL);
		//printf("after mxCreateDoubleMatrix \n");
		pOut = mxGetPr(plhs[0]);
		ppOut[0] = pOut;

	 }//if nrhs>0
	 
	 callTestFunction(ppIn,pmIn,pnIn, retNum, ppOut);
	 
     for (i = 0; i < nrhs; i++){
        pIn=ppIn[i];
        mxFree(pIn);
     }
     mxFree(pnIn);
     mxFree(pmIn);
     mxFree(ppIn);
	 mxFree(ppOut);
}
/*
void callTestFunction(double** ppIn,int *pmIn,int *pnIn, int ArgNum){
//call gpu
//testdotMul
printf("pnIn[0]=%d,pmIn[0]=%d, \n",pnIn[0],pmIn[0]);
if(ArgNum ==2){
	if((pmIn[0]==pmIn[1])&&(pnIn[0]==pnIn[1])){
		if(1==pnIn[0])mexTest_dotMul(ppIn[0],ppIn[1],pmIn[0]);
	}
}
//test norm
if(ArgNum ==1){
	if((pmIn[0]==pmIn[1])&&(pnIn[0]==pnIn[1])){
		if(1==pnIn[0])mexTest_norm(ppIn[0],pmIn[0]);
	}
}
//test matrixMul
//ppIn[0]:matrix A, ppIn[1]: vector B
if(ArgNum ==2){
	//pmIn[0]=mA pmIn[1]=nB, pnIn[0]=nB,pnIn[1]=1
	if((pmIn[1]==pnIn[0])&&(1==pnIn[1])){
		//mexTest_matrixMul(double *pA,double *pB,int mA, int nB);
		mexTest_matrixMul(ppIn[0],ppIn[1],pmIn[0], pmIn[1]);
	}
}
//call cpu

}
*/

/* Gateway function */
/*
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{
     int inputArgNum = nrhs;
     int outputArgNum = nlhs;
     int i,j,k,m, n;
     double *pMatrix;
     double * pIn;  
     double ** ppIn;
     int *pmIn;
     int *pnIn;
     pnIn = (int*)mxMalloc(sizeof(int)*nrhs);
     pmIn = (int*)mxMalloc(sizeof(int)*nrhs);
     ppIn = (double**)mxMalloc(sizeof(double*)*nrhs);
     //printf("nrhs = %d \n",nrhs);
     for (i = 0; i < nrhs; i++){
        // Find the dimensions of the data 
        m = mxGetM(prhs[i]);
        n = mxGetN(prhs[i]);
        printf("m = %d , n= %d \n",m,n);
        pmIn[i] = (int)m;
        pnIn[i] = (int)n;
        pMatrix = mxGetPr(prhs[i]); 
        pIn = (double*)mxMalloc(sizeof(double)*m*n);       
        ppIn[i]=pIn;
        
        for( k = 0; k < m; k++)
            for( j = 0; j < n; j++){
                pIn[k*n+j] = (double)pMatrix[j*m+k];
			//	printf("%f \n",pIn[k*n+j]);
            }
       
     }// for i
	 
	 //
	 callTestFunction(ppIn,pmIn,pnIn, inputArgNum);
	 
     for (i = 0; i < nrhs; i++){
        pIn=ppIn[i];
        //printf("m = %d , n= %d \n",pmIn[i],pnIn[i]);
        //for(j = 0; j < pnIn[i]*pmIn[i]; j++)printf("%f ,",pIn[j]);
        //printf("\n");
        mxFree(pIn);
     }
     mxFree(pnIn);
     mxFree(pmIn);
     mxFree(ppIn);
}
*/