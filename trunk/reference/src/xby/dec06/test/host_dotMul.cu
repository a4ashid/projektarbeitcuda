#include "hip/hip_runtime.h"
#include "test_comm.h"
#include "time.h"

void host_dotMul(double* pIn1, double* pIn2,double *pOut, int sizeIn, int sizeOut)
{

	int i, j;

	double *data_in1_d, *data_in2_d, *data_out_d;
	float *data_in1_f, *data_in2_f, *data_out_f;
	float *data_in1_f_gpu, *data_in2_f_gpu , *data_out_f_gpu;
	int sizeBlock;
	//VECTOR_BLOCK_SIZE defined in project_comm.h
	sizeBlock = VECTOR_BLOCK_SIZE;
	// variable for time measure
	int it;
	float t_avg;
	t_avg = 0;
	//ITERATE defined in project_comm.h
	it = ITERATE;
	// get Input data pointer
	data_in1_d = pIn1;
	data_in2_d = pIn2;
	// get Ouput data pointer
	data_out_d = pOut;
	//CUDA event
    hipError_t e; 
    //hipEvent_t start, stop;
    //float time;
	/*
	e = hipEventCreate(&start);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipEventCreate: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
    e = hipEventCreate(&stop);
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipEventCreate: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
	*/

	//create Timer
	/*
	unsigned int timer ;
	timer = 0;
	cutilCheckError(cutCreateTimer(&timer));
	cutilCheckError(cutStartTimer(timer));
	*/
	

	// Create an mxArray for the output data 
	
	// Create an input and output data array on the GPU
	hipMalloc( (void **) &data_in1_f_gpu,sizeof(t_ve)*sizeIn);
	hipMalloc( (void **) &data_in2_f_gpu,sizeof(t_ve)*sizeIn);
	hipMalloc( (void **) &data_out_f_gpu,sizeof(t_ve)*sizeOut);

	// The input array is in double precision, it needs to be converted t floats before being sent to the card 
	data_in1_f = (t_ve *) malloc(sizeof(t_ve)*sizeIn);
	data_in2_f = (t_ve *) malloc(sizeof(t_ve)*sizeIn);
	data_out_f = (t_ve *) malloc(sizeof(t_ve)*sizeOut);

	//startTime=clock();
	// Retrieve the input data 
	for (j = 0; j < sizeIn; j++)
	{
		data_in1_f[j] = (t_ve) data_in1_d[j];
		data_in2_f[j] = (t_ve) data_in2_d[j];
	}

	//startTime=clock();
 	////zeit
  /*
   e = hipEventRecord( start, 0 );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipEventRecord: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
   */
		// copy data from host to device
	hipMemcpy( data_in1_f_gpu, data_in1_f, sizeof(t_ve)*sizeIn, hipMemcpyHostToDevice);
	hipMemcpy( data_in2_f_gpu, data_in2_f, sizeof(t_ve)*sizeIn, hipMemcpyHostToDevice);
	// Compute execution configuration using sizeBlock threads per block 
	dim3 dimBlock(sizeBlock);
	//define enough grid Size
	dim3 dimGrid((sizeIn)/dimBlock.x);
	if ( (sizeIn) % sizeBlock !=0 ) dimGrid.x+=1;  
	

	for (i = 0; i < it ; i++){
		clock_t startTime;
		clock_t endTime;
		startTime=clock();		

		//Call function on GPU 
		device_dotMul<<<dimGrid,dimBlock>>>(data_in1_f_gpu, data_in2_f_gpu, data_out_f_gpu, sizeIn);
		//hipError_t e;
		
		e = hipGetLastError();
		if ( e != hipSuccess)
		{
			fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
			exit(-1);
		}
		
		endTime=clock();
		t_avg += endTime-startTime;
	}//for it
	printf("laufTime  in CPU = %lf (ms)\n", ((double) t_avg)*1000 /(it* CLOCKS_PER_SEC));
	
	// Copy result back to host 
	hipMemcpy( data_out_f, data_out_f_gpu, sizeof(float)*sizeOut, hipMemcpyDeviceToHost);
	
	/*
	e = hipEventRecord( stop, 0 );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipEventRecord: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }

    e = hipEventSynchronize( stop );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipEventRecord: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
	*/
/*
    e = hipEventElapsedTime( &time, start, stop );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipEventElapsedTime: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
	printf( "kernel runtime (size %d): %f milliseconds\n", sizeIn , time );
	*/
	//stop and destroy timer
	/*
	cutilCheckError(cutStopTimer(timer));
	printf("Processing time: %f (ms)\n", cutGetTimerValue(timer));
	*/

    //for (i = 0; i < sizeOut; i++)
    //{
     //   printf("data_out_f[%d] = %f, ", i, data_out_f[i]);
    //}
     //   printf("\n");


// Create a pointer to the output data 

	// Convert from single to double before returning 
	for (j = 0; j < sizeOut; j++)
	{
		data_out_d[j] = (double) data_out_f[j];
	}
	// Clean-up memory on device and host 
	free(data_in1_f);
	free(data_in2_f);
	free(data_out_f);
	hipFree(data_in1_f_gpu);
	hipFree(data_in2_f_gpu);
	hipFree(data_out_f_gpu);
}     
     
int test_dotMul()
{
    double *pIn1, *pIn2,*pOut;
    int sizeIn, sizeOut;
    int i;
	
	double expect;
	int loop;
	for (loop = 5000; loop < 5001; loop++) {
	int expect_error = 0;
    sizeIn = loop;
    //sizeOut =3;
	sizeOut =sizeIn/VECTOR_BLOCK_SIZE + 1;
    pIn1 = (double*)malloc(sizeof(double)*sizeIn);
    pIn2 = (double*)malloc(sizeof(double)*sizeIn);
    pOut = (double*)malloc(sizeof(double)*sizeOut);
    for (i = 0; i < sizeIn; i++){
		pIn1[i] = 1;
		pIn2[i] = 1;
	}
	/*
	pIn1[0] = 1;
    pIn1[1] = 2;
    pIn1[2] = 3;
    pIn2[0] = 1;
    pIn2[1] = 2;
    pIn2[2] = 3;
	*/
    host_dotMul(pIn1, pIn2, pOut, sizeIn, sizeOut);
	expect=sizeIn;
	printf("output square result");
	
	if(pOut[0] != expect){
		
		for (i = 0; i < sizeOut; i++)
		{	
        printf(" pOut[%d] = %lf, ", i, pOut[i]);
		}
		
		expect_error = loop;
		printf(" pOut[0] = %lf,  ", pOut[0]);
        printf("\n");
		printf("expect error = %d,\n",expect_error);
		
	}
	/*
		expect_error = loop;
		printf(" pOut[0] = %lf,  ", pOut[0]);
        printf("\n");
		printf("expect error = %d,\n",expect_error);
*/

    free(pIn1);
    free(pIn2);
    free(pOut);
	}
    return 0;

}
//mexInterface 
//int mexTest_dotMul(double *pIn1,double *pIn2,int sizeIn)
int mexTest_dotMul(double *pIn1,double *pIn2,double* pOut,int sizeIn)
{
    //double *pOut;
    int sizeOut;
    //int i;
    sizeOut =1;
	//sizeOut =sizeIn/VECTOR_BLOCK_SIZE + 1;
    //pIn1 = (double*)malloc(sizeof(double)*sizeIn);
    //pIn2 = (double*)malloc(sizeof(double)*sizeIn);
    //pOut = (double*)malloc(sizeof(double)*sizeOut);
	
    host_dotMul(pIn1, pIn2, pOut, sizeIn, sizeOut);
	//double expect=sizeIn;
	//printf("output square result");
	
	//if(pOut[0] != expect){
		
		//for (i = 0; i < sizeOut; i++)
		//{	
			//printf(" pOut[%d] = %lf, ", i, pOut[i]);
		//}
	//}
    //free(pOut);
    return 0;
}
