#include "hip/hip_runtime.h"
/*
UnitTestMain
*/

#include <stdio.h>
#include "host_dotMul.cu"
#include "host_norm.cu"
#include "host_matrixMul.cu"

#include "..\cpu\dotMul_cpu.c"
#include "..\cpu\norm_cpu.c"
#include "..\cpu\matrixMul_cpu.c"
/*
int main()
{
	 //test_matrixMul();
     test_dotMul();
	 //test_norm();
	 return 0;
}
*/
void callTestFunction(double** ppIn,int *pmIn,int *pnIn, int callFuncType,double** ppOut)
{
	//call cpu
	//printf("in callTestFunction \n");
	switch(callFuncType){
		case 0://dotMul
			if((pmIn[0]==pmIn[1])&&(pnIn[0]==pnIn[1])){
				//if(1==pnIn[0])test_dotMul_cpu(ppIn[0],ppIn[1],ppOut[0],pmIn[0]);
				if(1==pnIn[0]){
					host_dotMul(ppIn[0],ppIn[1],ppOut[0],pmIn[0],pnIn[0]);
					test_dotMul_cpu(ppIn[0],ppIn[1],ppOut[0],pmIn[0]);
				}
			}
			break;
		case 1://norm	
				//if(1==pnIn[0])test_norm_cpu(ppIn[0],ppOut[0],pmIn[0]);
				if(1==pnIn[0]){
					host_norm(ppIn[0],ppOut[0],pmIn[0], pnIn[0]);
					test_norm_cpu(ppIn[0],ppOut[0],pmIn[0]);
				}
		break;
		case 2: //matrixMul
			//test matrixMul A*B = C
			//ppIn[0]:matrix A, ppIn[1]: vector B, ppOut[0]: result verctor C
			//pmIn[0]=mA pmIn[1]=nB, pnIn[0]=nB,pnIn[1]=1
			if((pmIn[1]==pnIn[0])&&(1==pnIn[1])){
				//host_matrixMul(ppOut[0],ppIn[0],ppIn[1],pmIn[0], pmIn[1]);
				test_matrixMul_cpu(ppOut[0],ppIn[0],ppIn[1],pmIn[0], pmIn[1]);
			}
			
		default:  
			if((pmIn[0]==pmIn[1])&&(pnIn[0]==pnIn[1])){
				if(1==pnIn[0])host_dotMul(ppIn[0],ppIn[1],ppOut[0],pmIn[0],pnIn[0]);
			}
	}
}
void createMatrix(double **ppMatrix, unsigned int m, unsigned n,double val)
{
	double *pMatrix;
	int i;
	pMatrix = (double *)malloc(sizeof(double)*m*n);
	ppMatrix[0] = pMatrix;
	for (i = 0; i < m*n; i++){
		pMatrix[i] = val;
	}
}

int main()
{
	double** ppIn,**ppOut;
	double *pIn,*pOut;
	int *pmIn, *pnIn;
	int i, j, argInNum, argOutNum;
	int funcType,vectorSize;
	//0 dotMul
	//1 norm
	//2 matrixMul
	funcType = 2;
	vectorSize = 5000;
	switch(funcType){
		case 0://dotMul
			argInNum = 2;
			argOutNum = 1;
			
			ppOut = (double **)malloc(sizeof(double *)*argOutNum);
			ppIn = (double **)malloc(sizeof(double *)*argInNum);			
			pmIn = (int *) malloc(sizeof(int)*argInNum);			
			pnIn = (int *) malloc(sizeof(int)*argInNum);
			for(i = 0; i < argInNum ; i++){
				pmIn[i] =  vectorSize;
				pnIn[i] = 1;
				pIn = (double *)malloc(sizeof(double)*pmIn[i]*pnIn[i]);
				for(j = 0; j < pmIn[i]*pnIn[i]; j++)pIn[j] = 1;
				ppIn[i] = pIn;
			}		
			for(i = 0; i < argOutNum ; i++){
				pOut = (double *)malloc(sizeof(double)*1);
				pOut[i] = 0;
				ppOut[i] = pOut;
			}
		break;
		case 1:// norm
			argInNum = 1;
			argOutNum = 1;
			
			ppOut = (double **)malloc(sizeof(double *)*argOutNum);
			ppIn = (double **)malloc(sizeof(double *)*argInNum);			
			pmIn = (int *) malloc(sizeof(int)*argInNum);			
			pnIn = (int *) malloc(sizeof(int)*argInNum);
			for(i = 0; i < argInNum; i++){
				pmIn[i] =  vectorSize;
				pnIn[i] = 1;
				pIn = (double *)malloc(sizeof(double)*pmIn[i]*pnIn[i]);
				for(j = 0; j < pmIn[i]*pnIn[i]; j++)pIn[j] = 1;
				ppIn[i] = pIn;
			}		
			for(i = 0; i < argOutNum; i++){
				pOut = (double *)malloc(sizeof(double)*1);
				pOut[i] = 0;
				ppOut[i] = pOut;
			}
		break;
		case 2: //matrixMul
			argInNum = 2;
			argOutNum = 1;
			
			ppOut = (double **)malloc(sizeof(double *)*argOutNum);
			ppIn = (double **)malloc(sizeof(double *)*argInNum);			
			pmIn = (int *) malloc(sizeof(int)*argInNum);			
			pnIn = (int *) malloc(sizeof(int)*argInNum);
			
			pmIn[0] = vectorSize;
			pnIn[0] = vectorSize;
			pmIn[1] = vectorSize;
			pnIn[1] = 1;
			createMatrix(&(ppIn[0]),pmIn[0],pnIn[0],1);
			createMatrix(&(ppIn[1]),pmIn[1],pnIn[1],1);
			createMatrix(&(ppOut[0]),pmIn[0],pnIn[1],0);

		break;
		default:
		return 1;
		
	}

	callTestFunction(ppIn,pmIn,pnIn,funcType,ppOut);

	for ( i = 0; i < argInNum; i++){
		pIn = ppIn[i];
		//for(j = 0; j < pmIn[i]*pnIn[i]; j ++)printf("pIn[j] = %lf ,", pIn[j]);
		//printf("\n");
		free(pIn);
	}
	for ( i = 0; i < argOutNum; i++){
		pOut = ppOut[i];
		
        for(j = 0; j<pnIn[0];j++)printf("pOut[%d] = %lf \n", j,pOut[j]);
        
		free(pOut);
	}
	free(ppIn);
	free(ppOut);
	free(pmIn);
	free(pnIn);
	return 0;
}