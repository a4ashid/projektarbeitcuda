#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */
__global__ void square_elements(float* in, float* out, int N)
{
 
__shared__ float vOut[16];
int idx = blockIdx.x*blockDim.x+threadIdx.x;

if ( idx < N)vOut[idx] = in[idx]*in[idx];

__syncthreads();

if(idx == 0) {
    out[0] = 0;
	int i;
	for ( i = 0; i < N; i++ ) {
	   out[0] += vOut[i];
	}
}

__syncthreads();
if(idx == 0) {
	   out[0] = sqrt(out[0]);
}

}
void square_host(double* pIn, double *pOut, int sizeIn, int sizeOut)
{

int i, j;
double *data1, *data2;
float *data1f, *data2f;
float *data1f_gpu, *data2f_gpu;
int sizeBlock;
sizeBlock = 16;
data1 = pIn;
data2 = pOut;

/* Find the dimensions of the data */

/* Create an mxArray for the output data */

/* Create an input and output data array on the GPU*/
hipMalloc( (void **) &data1f_gpu,sizeof(float)*sizeIn);
hipMalloc( (void **) &data2f_gpu,sizeof(float)*sizeOut);
/* Retrieve the input data */

/* Check if the input array is single or double precision */

/* The input array is in double precision, it needs to be converted t
floats before being sent to the card */
data1f = (float *) malloc(sizeof(float)*sizeIn);
for (j = 0; j < sizeIn; j++)
{
data1f[j] = (float) data1[j];
}
    for (i = 0; i < sizeIn; i++)
    {
        printf("data1f[%d] = %f, ", i, data1f[i]);
    }
        printf("\n");

hipMemcpy( data1f_gpu, data1f, sizeof(float)*sizeIn, hipMemcpyHostToDevice);

data2f = (float *) malloc(sizeof(float)*sizeOut);
//cudaMemcpy( data2f_gpu, data2f, sizeof(float)*sizeOut, cudaMemcpyHostToDevice);

/* Compute execution configuration using 128 threads per block */
dim3 dimBlock(sizeBlock);
dim3 dimGrid((sizeIn)/dimBlock.x);
if ( (sizeIn) % sizeBlock !=0 ) dimGrid.x+=1;
    
/* Call function on GPU */
square_elements<<<dimGrid,dimBlock>>>(data1f_gpu, data2f_gpu, sizeIn);
hipError_t e;
e = hipGetLastError();
if ( e != hipSuccess)
{
    fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
    exit(-1);
}

/* Copy result back to host */
hipMemcpy( data2f, data2f_gpu, sizeof(float)*sizeOut, hipMemcpyDeviceToHost);
    for (i = 0; i < sizeOut; i++)
    {
        printf("data2f[%d] = %f, ", i, data2f[i]);
    }
        printf("\n");


/* Create a pointer to the output data */

/* Convert from single to double before returning */
for (j = 0; j < sizeOut; j++)
{
data2[j] = (double) data2f[j];
}
/* Clean-up memory on device and host */
free(data1f);
free(data2f);
hipFree(data1f_gpu);
hipFree(data2f_gpu);
}

int main()
{

    double *pIn, *pOut;
    int sizeIn, sizeOut;
    int i;
    sizeIn = 2;
    sizeOut = 1;
    pIn = (double*)malloc(sizeof(double)*sizeIn);
    pOut = (double*)malloc(sizeof(double)*sizeOut);
    pIn[0] = 3;
    pIn[1] = 4;
    //pIn[2] = 3;
    square_host(pIn, pOut, sizeIn, sizeOut);
	
	printf("output square result");
    for (i = 0; i < sizeOut; i++)
    {	
        printf(" pOut[%d] = %lf, ", i, pOut[i]);
    }
        printf("\n");
	printf("output norm result");
    for (i = 0; i < sizeOut; i++)
    {
		//pOut[i] = sqrt(pOut[i]);
        printf("squre of pOut[%d] = %lf, ", i, pOut[i]);
    }
        printf("\n");
    
   
    free(pIn);
    free(pOut);
    return 0;
}




/* Gateway function */
/*
void mexFunction(int nlhs, mxArray *plhs[],
int nrhs, const mxArray *prhs[])
{
int i, j, m, n;
double *data1, *data2;
float *data1f, *data2f;
float *data1f_gpu, *data2f_gpu;
mxClassID category;
if (nrhs != nlhs)
mexErrMsgTxt("The number of input and output arguments must be the same.");

//Create an mxArray for the output data by buyu 
//plhs[0] = mxCreateDoubleMatrix(1, 1, mxREAL);
//create a float host output variable for float
//data2f = (float *) mxMalloc(sizeof(float));

for (i = 0; i < nrhs; i++)
{
// Find the dimensions of the data 
m = mxGetM(prhs[i]);
n = mxGetN(prhs[i]);

//Create an mxArray for the output data 
//plhs[i] = mxCreateDoubleMatrix(m, n, mxREAL);
  plhs[i] = mxCreateDoubleMatrix(1, 1, mxREAL);


// Create an input and output data array on the GPU
cudaMalloc( (void **) &data1f_gpu,sizeof(float)*m*n);
//cudaMalloc( (void **) &data2f_gpu,sizeof(float)*m*n);
cudaMalloc( (void **) &data2f_gpu,sizeof(float));

// Retrieve the input data 
data1 = mxGetPr(prhs[i]);
// Check if the input array is single or double precision 
category = mxGetClassID(prhs[i]);
if( category == mxSINGLE_CLASS)
{
// The input array is single precision, it can be sent directly to the card
cudaMemcpy( data1f_gpu, data1, sizeof(float)*m*n,
cudaMemcpyHostToDevice);
}
if( category == mxDOUBLE_CLASS)
{
// The input array is in double precision, it needs to be converted t floats before being sent to the card 
data1f = (float *) mxMalloc(sizeof(float)*m*n);
for (j = 0; j < m*n; j++)
{
data1f[j] = (float) data1[j];
}
printf("before copyHost to device \n");
cudaMemcpy( data1f_gpu, data1f, sizeof(float)*n*m, cudaMemcpyHostToDevice);
}

//orginal output
//data2f = (float *) mxMalloc(sizeof(float)*m*n);
data2f = (float *) mxMalloc(sizeof(float));

// Compute execution configuration using 128 threads per block 
dim3 dimBlock(128);
dim3 dimGrid((m*n)/dimBlock.x);
if ( (n*m) % 128 !=0 ) dimGrid.x+=1;
    
printf("before calling GPU \n");
// Call function on GPU 
square_elements<<<dimGrid,dimBlock>>>(data1f_gpu, data2f_gpu, n*m);

printf("before copy result back \n");
// Copy result back to host 
//cudaMemcpy( data2f, data2f_gpu, sizeof(float)*n*m, cudaMemcpyDeviceToHost);
cudaMemcpy( data2f, data2f_gpu, sizeof(float), cudaMemcpyDeviceToHost);
// Create a pointer to the output data 
data2 = mxGetPr(plhs[i]);
// Convert from single to double before returning 

//for (j = 0; j < m*n; j++)
//{
//data2[j] = (double) data2f[j];
//}

printf("before return result to matlab \n");
data2[0] = 0;
data2[0] = (double) data2f[0];

// Clean-up memory on device and host 
mxFree(data1f);
mxFree(data2f);
cudaFree(data1f_gpu);
cudaFree(data2f_gpu);
}// for i

}
*/