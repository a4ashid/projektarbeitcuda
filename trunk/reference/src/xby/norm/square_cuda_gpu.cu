#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
//#include "mex.h"
/* Kernel to square elements of the array on the GPU */
__global__ void square_elements(float* in, float* out, int N)
{
int idx = blockIdx.x*blockDim.x+threadIdx.x;
//int idx = threadIdx.x;
if ( idx < N) out[idx]=in[idx]*in[idx];
}

void square_host(double* pIn, double *pOut, int sizeIn, int sizeOut)
{

int i, j;
double *data1, *data2;
float *data1f, *data2f;
float *data1f_gpu, *data2f_gpu;
int sizeBlock;
sizeBlock = 1;
data1 = pIn;


/* Find the dimensions of the data */

/* Create an mxArray for the output data */

/* Create an input and output data array on the GPU*/
hipMalloc( (void **) &data1f_gpu,sizeof(float)*sizeIn);
hipMalloc( (void **) &data2f_gpu,sizeof(float)*sizeOut);
/* Retrieve the input data */

/* Check if the input array is single or double precision */

/* The input array is in double precision, it needs to be converted t
floats before being sent to the card */
data1f = (float *) malloc(sizeof(float)*sizeIn);
for (j = 0; j < sizeIn; j++)
{
data1f[j] = (float) data1[j];
}
    for (i = 0; i < sizeOut; i++)
    {
        printf("data1f[%d] = %f, ", i, data1f[i]);
    }
        printf("\n");

hipMemcpy( data1f_gpu, data1f, sizeof(float)*sizeIn, hipMemcpyHostToDevice);

data2f = (float *) malloc(sizeof(float)*sizeOut);
/* Compute execution configuration using 128 threads per block */
dim3 dimBlock(sizeBlock);
dim3 dimGrid((sizeIn)/dimBlock.x);
if ( (sizeIn) % sizeBlock !=0 ) dimGrid.x+=1;
    
/* Call function on GPU */
hipError_t e;
square_elements<<<dimGrid,dimBlock>>>(data1f_gpu, data2f_gpu, sizeIn);
e = hipGetLastError();
if ( e != hipSuccess)
{
    fprintf(stderr, "CUDA Error on square_elements: '%s' \n", cudaGestErrorString(e));
    exit(-1);
}



/* Copy result back to host */
hipMemcpy( data2f, data2f_gpu, sizeof(float)*sizeOut, hipMemcpyDeviceToHost);
    for (i = 0; i < sizeOut; i++)
    {
        printf("data2f[%d] = %f, ", i, data2f[i]);
    }
        printf("\n");


/* Create a pointer to the output data */
data2 = pOut;
/* Convert from single to double before returning */
for (j = 0; j < sizeOut; j++)
{
data2[j] = (double) data2f[j];
}
/* Clean-up memory on device and host */
free(data1f);
free(data2f);
hipFree(data1f_gpu);
hipFree(data2f_gpu);


}

int main()
{

    double *pIn, *pOut;
    int sizeIn, sizeOut;
    int i;
    sizeIn = 1;
    sizeOut = 1;
    pIn = (double*)malloc(sizeof(double)*sizeIn);
    pOut = (double*)malloc(sizeof(double)*sizeOut);
    pIn[0] = 1;
  //  pIn[1] = 2;
   // pIn[2] = 3;
    square_host(pIn, pOut, sizeIn, sizeOut);
    for (i = 0; i < sizeOut; i++)
    {
        printf("pOut[%d] = %lf, ", i, pOut[i]);
    }
        printf("\n");
    

    free(pIn);
    free(pOut);

    return 0;
}

/* Gateway function */
/*
void mexFunction(int nlhs, mxArray *plhs[],
int nrhs, const mxArray *prhs[])
{
int i, j, m, n;
double *data1, *data2;
float *data1f, *data2f;
float *data1f_gpu, *data2f_gpu;
mxClassID category;
if (nrhs != nlhs)
mexErrMsgTxt("The number of input and output arguments must be the same.");
for (i = 0; i < nrhs; i++)
{
// Find the dimensions of the data 
m = mxGetM(prhs[i]);
n = mxGetN(prhs[i]);
// Create an mxArray for the output data 
plhs[i] = mxCreateDoubleMatrix(m, n, mxREAL);
// Create an input and output data array on the GPU
hipMalloc( (void **) &data1f_gpu,sizeof(float)*m*n);
hipMalloc( (void **) &data2f_gpu,sizeof(float)*m*n);
//Retrieve the input data 
data1 = mxGetPr(prhs[i]);
// Check if the input array is single or double precision 
category = mxGetClassID(prhs[i]);
if( category == mxSINGLE_CLASS)
{
// The input array is single precision, it can be sent directly to the card 
hipMemcpy( data1f_gpu, data1, sizeof(float)*m*n,
hipMemcpyHostToDevice);
}
if( category == mxDOUBLE_CLASS)
{
// The input array is in double precision, it needs to be converted t floats before being sent to the card 
data1f = (float *) mxMalloc(sizeof(float)*m*n);
for (j = 0; j < m*n; j++)
{
data1f[j] = (float) data1[j];
}


hipMemcpy( data1f_gpu, data1f, sizeof(float)*n*m, hipMemcpyHostToDevice);
}
data2f = (float *) mxMalloc(sizeof(float)*m*n);
// Compute execution configuration using 128 threads per block 
dim3 dimBlock(128);
dim3 dimGrid((m*n)/dimBlock.x);
if ( (n*m) % 128 !=0 ) dimGrid.x+=1;
    
//Call function on GPU 
square_elements<<<dimGrid,dimBlock>>>(data1f_gpu, data2f_gpu, n*m);
// Copy result back to host 
hipMemcpy( data2f, data2f_gpu, sizeof(float)*n*m, hipMemcpyDeviceToHost);
// Create a pointer to the output data 
data2 = mxGetPr(plhs[i]);
// Convert from single to double before returning 
for (j = 0; j < m*n; j++)
{
data2[j] = (double) data2f[j];
}
// Clean-up memory on device and host 
mxFree(data1f);
mxFree(data2f);
hipFree(data1f_gpu);
hipFree(data2f_gpu);
}
}
*/