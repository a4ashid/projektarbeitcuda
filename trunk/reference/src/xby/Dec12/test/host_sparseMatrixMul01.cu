//host_SparseMV
#include "test_comm.h"
void host_sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector){

	t_SparseMatrix data_in1,*data_in1_host,*data_in1_gpu;//input sparse Matrix
	t_FullMatrix data_in2,*data_in2_host,*data_in2_gpu;//input vector
	t_FullMatrix data_out,*data_out_host,*data_out_gpu;//output vector
	size_t size_NZElement,size_Row,size_Col;
	int sizeBlock,i;
	data_in1_host=&data_in1;
	data_in2_host=&data_in2;
	data_out_host=&data_out;
	sizeBlock = VECTOR_BLOCK_SIZE;
	//=====debug==================
	printf("=======in host========== \n");
	printf("pSparseMatrix->m=%d \n",pSparseMatrix->m);
	printf("pSparseMatrix->n=%d \n",pSparseMatrix->n);
	//============================
	
	size_NZElement = sizeof(t_ve)*pSparseMatrix->nzmax;
	size_Row =sizeof(int)*(pSparseMatrix->m+1); //sizeof(int)*pSparseMatrix->m;
	size_Col = sizeof(int)*pSparseMatrix->nzmax;//sizeof(int)*(pSparseMatrix->n+1);
	// Create an input and output data array on the GPU
	//malloc memory for Input Sparse-Matrix
	
	printf("malloc sparse Matrix \n");
	hipMalloc( (void **) &(data_in1_host->pNZElement),size_NZElement);
	hipMalloc( (void **) &(data_in1_host->pRow),size_Row);
	hipMalloc( (void **) &(data_in1_host->pCol),size_Col);
	data_in1_host->m = pSparseMatrix->m;
	data_in1_host->n = pSparseMatrix->n;
	data_in1_host->nzmax = pSparseMatrix->nzmax;
	hipMemcpy(data_in1_host->pNZElement,pSparseMatrix->pNZElement,size_NZElement,hipMemcpyHostToDevice);
	hipMemcpy(data_in1_host->pRow,pSparseMatrix->pRow,size_Row,hipMemcpyHostToDevice);
	hipMemcpy(data_in1_host->pCol,pSparseMatrix->pCol,size_Col,hipMemcpyHostToDevice);
	//hipMalloc( (void **) &data_in1_gpu,sizeof(t_SparseMatrix)*1);
	//hipMemcpy(data_in1_gpu,data_in1_host,sizeof(t_SparseMatrix)*1,hipMemcpyHostToDevice);
	
	//malloc device memory for Input vector
	printf("malloc vector \n");
	size_t size_VElement, size_RElement;
	size_VElement = sizeof(t_ve)*pVector->m*pVector->n;
	size_RElement = sizeof(t_ve)*pSparseMatrix->m;
	hipMalloc( (void **) &(data_in2_host->pElement),size_VElement);
	data_in2_host->m = pVector->m;
	data_in2_host->n = pVector->n;
	hipMemcpy(data_in2_host->pElement,pVector->pElement,size_Col,hipMemcpyHostToDevice);
	//hipMalloc( (void **) &data_in2_gpu,sizeof(t_FullMatrix)*1);
	//hipMemcpy(data_in2_gpu,data_in2_host,sizeof(t_FullMatrix)*1,hipMemcpyHostToDevice);

	
	
	printf("malloc output \n");
	//malloc output Vector
	data_out_host->m = pSparseMatrix->m;
	data_out_host->n = 1;
	hipMalloc( (void **) &(data_out_host->pElement),size_RElement);
	//hipMalloc( (void **) &data_out_gpu,sizeof(t_FullMatrix)*1);
	//hipMemcpy(data_out_gpu,data_out_host,sizeof(t_FullMatrix)*1,hipMemcpyHostToDevice);

	// Compute execution configuration using 128 threads per block 
	dim3 dimBlock(sizeBlock);
	//dim3 dimGrid((sizeIn)/dimBlock.x);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp,0);
	printf("number of multiProcessors: %d \n",deviceProp.multiProcessorCount);
	int sizeGrid = 65535;
	if (sizeGrid > pSparseMatrix->m)sizeGrid = pSparseMatrix->m;
	printf("grid size = %d\n",sizeGrid);
	dim3 dimGrid(sizeGrid);
	//if ( (sizeA) % sizeBlock !=0 ) dimGrid.x+=1;
	//sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector)
	//sparseMatrixMul<<<dimGrid,dimBlock>>>(data_out_gpu,data_in1_gpu,data_in2_gpu);
	printf("calling kernel \n");
	sparseMatrixMul<<<dimGrid,dimBlock>>>(data_out,data_in1,data_in2);
	hipError_t e;	
	e = hipGetLastError();
	if ( e != hipSuccess)
	{
			fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
			exit(-1);
	}
	
	printf("get Result \n");
	//hipMemcpy( data_out_host->pElement,pResultVector->pElement,  size_RElement, hipMemcpyDeviceToHost);
	hipMemcpy( pResultVector->pElement,data_out_host->pElement,  size_RElement, hipMemcpyDeviceToHost);
	
	pResultVector->m = pSparseMatrix->m;
	pResultVector->n = 1;
	//=========debug==============
		printf("==================Result in host============\n");
		for( i = 0; i < pResultVector->m; i++) printf("pResultVector->pElement[%d]=%f \n",i,pResultVector->pElement[i]);
	//=======================
	
	printf("free host \n");
	hipFree(data_in1_host->pNZElement);
	hipFree(data_in1_host->pRow);
	hipFree(data_in1_host->pCol);
	hipFree(data_in2_host->pElement);
	hipFree(data_out_host->pElement);

}


