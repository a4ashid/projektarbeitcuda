#include "hip/hip_runtime.h"
#include "test_comm.h"
#include "time.h"

void host_scalarMul(double* pIn1, double* pIn2,double *pOut, int sizeIn, int sizeOut)
{

	int i, j;

	double *data_in1_d, *data_in2_d, *data_out_d;
	float *data_in1_f, *data_in2_f, *data_out_f;
	float *data_in1_f_gpu, *data_in2_f_gpu , *data_out_f_gpu;
	int sizeBlock;
	//VECTOR_BLOCK_SIZE defined in project_comm.h
	sizeBlock = VECTOR_BLOCK_SIZE;
	// variable for time measure
	int it;
	float t_avg;
	t_avg = 0;
	//ITERATE defined in project_comm.h
	it = ITERATE;
	// get Input data pointer
	data_in1_d = pIn1;
	data_in2_d = pIn2;
	// get Ouput data pointer
	data_out_d = pOut;
	//CUDA event
    hipError_t e; 
    //hipEvent_t start, stop;
    //float time;

	// Create an mxArray for the output data 
	//change sizeOut for cuda
	//sizeOut = (sizeIn)/sizeBlock;
	// Create an input and output data array on the GPU
	hipMalloc( (void **) &data_in1_f_gpu,sizeof(t_ve)*sizeIn);
	hipMalloc( (void **) &data_in2_f_gpu,sizeof(t_ve)*1);
	hipMalloc( (void **) &data_out_f_gpu,sizeof(t_ve)*sizeOut);

	// The input array is in double precision, it needs to be converted t floats before being sent to the card 
	data_in1_f = (t_ve *) malloc(sizeof(t_ve)*sizeIn);
	data_in2_f = (t_ve *) malloc(sizeof(t_ve)*1);
	data_out_f = (t_ve *) malloc(sizeof(t_ve)*sizeOut);

	//startTime=clock();
	// Retrieve the input data 
	for (j = 0; j < sizeIn; j++)
	{
		data_in1_f[j] = (t_ve) data_in1_d[j];
	
	}
		data_in2_f[0] = (t_ve) data_in2_d[0];

	//startTime=clock();
 	////zeit
  /*
   e = hipEventRecord( start, 0 );
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error on hipEventRecord: '%s' \n", hipGetErrorString(e));
        exit(-3);
    }
   */
	
		// copy data from host to device
	hipMemcpy( data_in1_f_gpu, data_in1_f, sizeof(t_ve)*sizeIn, hipMemcpyHostToDevice);
	hipMemcpy( data_in2_f_gpu, data_in2_f, sizeof(t_ve)*1, hipMemcpyHostToDevice);
	

		
	
	// Compute execution configuration using sizeBlock threads per block 
	dim3 dimBlock(sizeBlock);
	//define enough grid Size
	
	
	dim3 dimGrid((sizeIn)/dimBlock.x);
	if ( (sizeIn) % sizeBlock !=0 ) dimGrid.x+=1;  
	
	//it =1;
	for (i = 0; i < it ; i++){


		//Call function on GPU 
		device_scalarMul<<<dimGrid,dimBlock>>>(data_in1_f_gpu, data_in2_f_gpu, data_out_f_gpu, sizeIn);
		//hipError_t e;
		
		e = hipGetLastError();
		if ( e != hipSuccess)
		{
			fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
			exit(-1);
		}
		// Copy result back to host 
		hipMemcpy( data_out_f, data_out_f_gpu, sizeof(float)*sizeOut, hipMemcpyDeviceToHost);
		

	}//for it

	//cuda Timer end

    for (i = 0; i < sizeOut; i++)
    {
        printf("data_out_f[%d] = %f, ", i, data_out_f[i]);
    }
        printf("\n");


// Create a pointer to the output data 

	// Convert from single to double before returning 
	for (j = 0; j < sizeOut; j++)
	{
		data_out_d[j] = (double) data_out_f[j];
	}
	// Clean-up memory on device and host 
	free(data_in1_f);
	free(data_in2_f);
	free(data_out_f);
	hipFree(data_in1_f_gpu);
	hipFree(data_in2_f_gpu);
	hipFree(data_out_f_gpu);
}     
     
int test_scalarMul()
{
    double *pIn1, *pIn2,*pOut;
    int sizeIn, sizeOut;
    int i;
	
	double expect;
	int loop;
	for (loop = 5000; loop < 5001; loop++) {
	int expect_error = 0;
    sizeIn = loop;
    //sizeOut =3;
	sizeOut =sizeIn;
    pIn1 = (double*)malloc(sizeof(double)*sizeIn);
    pIn2 = (double*)malloc(sizeof(double)*1);
    pOut = (double*)malloc(sizeof(double)*sizeOut);
    for (i = 0; i < sizeIn; i++){
		pIn1[i] = 1;
		
	}
	pIn2[0] = 1;
	/*
	pIn1[0] = 1;
    pIn1[1] = 2;
    pIn1[2] = 3;
    pIn2[0] = 1;
    pIn2[1] = 2;
    pIn2[2] = 3;
	*/
    host_scalarMul(pIn1, pIn2, pOut, sizeIn, sizeOut);
	expect=sizeIn;
	printf("output square result");
	/*
	if(pOut[0] != expect){
		
		for (i = 0; i < sizeOut; i++)
		{	
        printf(" pOut[%d] = %lf, ", i, pOut[i]);
		}
		
		expect_error = loop;
		printf(" pOut[0] = %lf,  ", pOut[0]);
        printf("\n");
		printf("expect error = %d,\n",expect_error);
		
	}*/
	/*
		expect_error = loop;
		printf(" pOut[0] = %lf,  ", pOut[0]);
        printf("\n");
		printf("expect error = %d,\n",expect_error);
*/

    free(pIn1);
    free(pIn2);
    free(pOut);
	}
    return 0;

}
//mexInterface 
//int mexTest_dotMul(double *pIn1,double *pIn2,int sizeIn)
int mexTest_scalarMul(double *pIn1,double *pIn2,double* pOut,int sizeIn)
{
    //double *pOut;
    int sizeOut;
    //int i;
    sizeOut =sizeIn;

    host_scalarMul(pIn1, pIn2, pOut, sizeIn, sizeOut);

    return 0;
}
