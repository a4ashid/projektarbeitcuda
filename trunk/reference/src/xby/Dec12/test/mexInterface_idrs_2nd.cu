/* matlab interface for 

	idrs2nd(
			t_FullMatrix P,
			t_ve tol,
			unsigned int s,
			unsigned int maxit,
			t_idrshandle  ih_in,
			
			t_ve* x,  // output vector of size N
            t_ve* resvec, // output vector of size ??????
            unsigned int* piter //output int point 
	)

);
*/

#include <math.h> 
#include "mex.h"
#include "..\\gpu\\projektcuda.h"



/* Gateway function */
//[x,resvec,iter]=mexInterface_idrs_2nd(P, tol, s, maxit, ih_in);
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{

	int inputIdx, outputIdx;
	t_ve tol;
	t_ve *x0, *x, *resvec;
	t_mindex  maxit, N, *piter;
    t_mindex m,n;
	t_SparseMatrix A;
	//struct FullMatrix P_in;
	t_FullMatrix P_in;
	int i, s,ih_in;
    double *pr,*pi;
    int size_resvec;
	int msize ;
	//void *devicemem;
	if( nrhs < 5 ) {
		printf("not enough input argument!\n");
		printf("[x,resvec,iter]=mexInterface_idrs_2nd(P, tol, s, maxit, ih_in);\n");
		return;
	}
    //=======read input===============================================================
	//read Matrix P_in
	printf("read  Matrix P!\n");
	inputIdx = 0;
	//printf("inputIdx=%d\n",inputIdx);
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);	
	// inupt matrix rotate
	P_in.m = m;
	P_in.n = n;
	msize = P_in.m*P_in.n;
	N = P_in.m; ///N is the size of Vector or m of P
	//printf("N=%d\n",N);
	P_in.pElement = (t_ve*)mxMalloc(sizeof(t_ve)*msize);
	for(i = 0; i < msize; i++){
        P_in.pElement[i] =(t_ve)pr[i];
    }
	
	//read tol
	inputIdx = 1;
	//printf("inputIdx=%d\n",inputIdx);
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	tol = (unsigned int)pr[0];
	 
	//read s
	inputIdx = 2;
	//printf("inputIdx=%d\n",inputIdx);
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	s = (unsigned int)pr[0];
	//read maxit
	inputIdx = 3;
	//printf("inputIdx=%d\n",inputIdx);
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	maxit = (unsigned int)pr[0];
	
	//read ih_in
	inputIdx = 4;
	//printf("inputIdx=%d\n",inputIdx);
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	ih_in = pr[0];	
	
	
	printf("create output!\n");
	//create output vector x of size N
	x = (t_ve*)mxMalloc(sizeof(t_ve)*N);
	//create output vector resvec, size ??????
	resvec = (t_ve*)mxMalloc(sizeof(t_ve)*N); 
	//output piter
	piter = (unsigned int *)mxMalloc(sizeof(unsigned int)*1);
	//=======================================================================
	//call idrs interface 
	
	//idrs2(P_in, tol, s, maxit,ih_in, t_ve* x, resvec, piter );
	
	//=======================================================================
	//output x, resvec,piter in matlab
	outputIdx = 0;
	//printf("outputIdx=%d\n",outputIdx);
    plhs[outputIdx] = mxCreateDoubleMatrix(N,1,mxREAL);
	pr = mxGetPr(plhs[outputIdx]);
	for(i = 0; i < N; i++){
		//x[i]=1;
		pr[i] = (double)x[i];
	}
	//output resvec
	outputIdx = 1;
	//printf("outputIdx=%d\n",outputIdx);
    size_resvec = N;
    plhs[outputIdx] = mxCreateDoubleMatrix(size_resvec,1,mxREAL);
	pr = mxGetPr(plhs[outputIdx]);
	for(i = 0; i < size_resvec; i++){
		//resvec[i]=2;
		pr[i] = (double)resvec[i];
	}
	//output iter of scalar value
	
	outputIdx =2;
	//printf("outputIdx=%d\n",outputIdx);
	//plhs[outputIdx] = mxCreateNumericMatrix(1,1,mxUINT32_CLASS,mxREAL);
	plhs[outputIdx] = mxCreateDoubleMatrix(1,1,mxREAL);
	pr = mxGetPr(plhs[outputIdx]);
	for(i = 0; i < 1; i++){
		//piter[i] = 3;
		pr[i] = (unsigned int) piter[i];
	}
	//=======================================================================
	mxFree(P_in.pElement);
	mxFree(x);
	mxFree(resvec);
	mxFree(piter);
}