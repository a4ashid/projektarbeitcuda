/* matlab interface for 
__host__ void idrs(
                     SparseMatrix A, // size NxN
                     t_ve* b, // size N
                     unsigned int s,
                     t_ve  tol, // t_ve scalar 
                     unsigned int maxit, //int scalar
                     t_ve* x0, //size N

                     unsigned int N, //vector and matrix size

                     t_ve* x,  // output vector of size N
                     t_ve* resvec, // output vector of size ??????
                     unsigned int* piter //output int point 
                  );
	idrs2nd(
			t_fullMatrix P,
			t_ve tol,
			unsigned int s,
			unsigned int maxit,
			t_idrshandle  ih_in,
			
			t_ve* x,  // output vector of size N
            t_ve* resvec, // output vector of size ??????
            unsigned int* piter //output int point 
	)
				  */


#include <math.h> 
#include "mex.h"
#include "..\\gpu\\projektcuda.h"

int smat_size( int count_nzmax, int cunt_rows ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * count_nzmax
           + sizeof(t_mindex)  * (cunt_rows + 1);
}

// ---------------------------------------------------------------------
 void set_sparse_data( t_SparseMatrix* m, void* mv ) {

   m->pCol = (t_mindex *) mv;
   m->pNZElement = (t_ve *) (&m->pCol[m->nzmax] ) ;
   m->pRow = (t_mindex *) (&m->pNZElement[m->nzmax]);

}
/* Gateway function */
//[x,resvec,iter]=mexInterface_idrs(A,b,s,tol,maxit,x0);
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{

	int inputIdx;
	t_fullMatrix P;
	t_ve tol, *x0, *x, *resvec;
	unsigned int  maxit, N, *piter;
    unsigned int m,n;
    //unsigned int nzmax;
    //int *ir,*jc,i;
	int i;
    //unsigned int *irs,*jcs;
    //int cmplx,isfull;
    double *pr,*pi;//,*si,*sr;
    int size_resvec;
	int msize ;
	//void *devicemem;
	if( nrhs < 5 ) {
		printf("not enough input argument!\n");
		printf("[x,resvec,iter]=mexInterface_idrs_2nd(P, tol, s, maxit, ih_in);\n");
		return;
	}
    //=======read input===============================================================
	//read Matrix P
	printf("read  Matrix P!\n");
	inputIdx = 0;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);	
	// inupt matrix rotate
	P.m = m;
	P.n = n;
	msize = P.m*P.n;
	N = P.m; ///N is the size of Vector or m of P
	for(i = 0; i < msize; i++){
        P.pElement[i] =(t_ve)pr[i];
    }
	
	//read tol
	inputIdx = 1;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	tol = (unsigned int)pr[0];
	 
	//read s
	inputIdx = 2;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	s = (unsigned int)pr[0];
	//read maxit
	inputIdx = 3;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	maxit = (unsigned int)pr[0];
	
	//read ih_in
	inputIdx = 4;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	ih_in = pr[0];	
	
	
	printf("create output!\n");
	//create output vector x of size N
	x = (t_ve*)mxMalloc(sizeof(t_ve)*N);
	//create output vector resvec, size ??????
	resvec = (t_ve*)mxMalloc(sizeof(t_ve)*m*n); 
	//output piter
	piter = (unsigned int *)mxMalloc(sizeof(unsigned int)*1);
	//=======================================================================
	//call idrs interface 
	
	//idrs2(P, tol, s, maxit,ih_in, t_ve* x, resvec, piter );
	
	//=======================================================================
	//output x, resvec,piter in matlab
	outputIdx = 0;
    plhs[outputIdx] = mxCreateDoubleMatrix(N,1,mxREAL);
	pr = mxGetPr(plhs[outputIdx]);
	for(i = 0; i < N; i++){
		pr[i] = (double)x[i];
	}
	//output resvec
	outputIdx = 1;
    size_resvec =10;
    plhs[outputIdx] = mxCreateDoubleMatrix(size_resvec,1,mxREAL);
	pr = mxGetPr(plhs[outputIdx]);
	for(i = 0; i < size_resvec; i++){
		pr[i] = (double)resvec[i];
	}
	//output iter of scalar value
	outputIdx =2;
	plhs[outputIdx] = mxCreateNumericMatrix(1,1,mxUINT32_CLASS,mxREAL);
	pr = mxGetPr(plhs[outputIdx]);
	for(i = 0; i < 1; i++){
		pr[i] = piter[i];
	}
	//=======================================================================

	mxFree(x);
	mxFree(resvec);
	mxFree(piter);

	
	
}