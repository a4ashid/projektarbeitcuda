//host_SparseMV
#include "test_comm.h"
void host_sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector){

	t_SparseMatrix *data_in1_gpu;//input sparse Matrix
	t_FullMatrix *data_in2_gpu;//input vector
	t_FullMatrix *data_out_gpu;//output vector
	// Create an input and output data array on the GPU
	hipMalloc( (void **) &data_in1_gpu,sizeof(t_SparseMatrix)*1);
	//hipMalloc( (void **) &(data_in1_gpu->pNZElement),sizeof(t_ve)*pSparseMatrix->nzmax+sizeof(int)*pSparseMatrix->m+sizeof(int)*(pSparseMatrix->n+1));
	hipMalloc( (void **) &(data_in1_gpu->pNZElement),sizeof(t_ve)*pSparseMatrix->nzmax);
	hipMalloc( (void **) &(data_in1_gpu->pRow),sizeof(int)*pSparseMatrix->m);
	hipMalloc( (void **) &(data_in1_gpu->pCol),sizeof(int)*(pSparseMatrix->n+1));
	data_in1_gpu->m = pSparseMatrix->m;
	data_in1_gpu->n = pSparseMatrix->n;
	data_in1_gpu->nzmax = pSparseMatrix->nzmax;
	cudaMamcpy(data_in1_gpu->pNZElement)
	
	hipMalloc( (void **) &data_in2_gpu,sizeof(t_FullMatrix)*1);
	hipMalloc( (void **) &(data_in2_gpu->pElement),sizeof(t_ve)*pVector->m*pVector->n);
	
	hipMalloc( (void **) &data_out_gpu,sizeof(t_FullMatrix)*1);
	hipMalloc( (void **) &(data_out_gpu->pElement),sizeof(t_ve)*pSparseMatrix->m);

//sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector)

}


