//host_SparseMV
#include "test_comm.h"
void host_sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector){

	t_SparseMatrix *data_in1_gpu;//input sparse Matrix
	t_FullMatrix *data_in2_gpu;//input vector
	t_FullMatrix *data_out_gpu;//output vector
	
	size_t size_NZElement,size_Row,size_Col;
	
	int sizeBlock;
	sizeBlock = VECTOR_BLOCK_SIZE;
	
	size_NZElement = sizeof(t_ve)*pSparseMatrix->nzmax;
	size_Row + sizeof(int)*pSparseMatrix->m;
	size_Col = sizeof(int)*(pSparseMatrix->n+1);
	// Create an input and output data array on the GPU
	//malloc memory for Input Sparse-Matrix
	hipMalloc( (void **) &data_in1_gpu,sizeof(t_SparseMatrix)*1);
	//hipMalloc( (void **) &(data_in1_gpu->pNZElement),sizeof(t_ve)*pSparseMatrix->nzmax+sizeof(int)*pSparseMatrix->m+sizeof(int)*(pSparseMatrix->n+1));
	
	hipMalloc( (void **) &(data_in1_gpu->pNZElement),size_NZElement);
	hipMalloc( (void **) &(data_in1_gpu->pRow),size_Row);
	hipMalloc( (void **) &(data_in1_gpu->pCol),size_Col);
	data_in1_gpu->m = pSparseMatrix->m;
	data_in1_gpu->n = pSparseMatrix->n;
	data_in1_gpu->nzmax = pSparseMatrix->nzmax;
	cudaMamcpy(data_in1_gpu->pNZElement,pSparseMatrix->pNZElement,size_NZElement,hipMemcpyHostToDevice);
	cudaMamcpy(data_in1_gpu->pRow,pSparseMatrix->pRow,size_Row,hipMemcpyHostToDevice);
	cudaMamcpy(data_in1_gpu->pCol,pSparseMatrix->pCol,size_Col,hipMemcpyHostToDevice);
	
	//malloc device memory for Input vector
	size_t size_VElement, size_RElement;
	size_VElement = sizeof(t_ve)*pVector->m*pVector->n;
	size_RElement = sizeof(t_ve)*pSparseMatrix->m;
	hipMalloc( (void **) &data_in2_gpu,sizeof(t_FullMatrix)*1);
	hipMalloc( (void **) &(data_in2_gpu->pElement),size_VElement);
	data_in2_gpu->m = pVector->m;
	data_in2_gpu->n = pVector->n;
	cudaMamcpy(data_in2_gpu->pElement,pVector->pElement,size_Col,hipMemcpyHostToDevice);
	
	//malloc output Vector
	hipMalloc( (void **) &data_out_gpu,sizeof(t_FullMatrix)*1);
	hipMalloc( (void **) &(data_out_gpu->pElement),size_RElement);

	// Compute execution configuration using 128 threads per block 
	dim3 dimBlock(sizeBlock);
	//dim3 dimGrid((sizeIn)/dimBlock.x);
	dim3 dimGrid(pSparseMatrix->m);
	//if ( (sizeA) % sizeBlock !=0 ) dimGrid.x+=1;
//sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector)
	sparseMatrixMul<<<dimGrid,dimBlock>>>(data_out_f_gpu,data_in1_f_gpu, data_in2_f_gpu);
	e = hipGetLastError();
	if ( e != hipSuccess)
	{
			fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
			exit(-1);
	}
	hipMemcpy( pResultVector->pElement, data_out_f_gpu->pElement, size_RElement, hipMemcpyDeviceToHost);
	pResultVector->m = pSparseMatrix->m;
	pResultVector->m = 1;
	
	hipFree(data_in1_f_gpu);
	hipFree(data_in2_f_gpu);
	hipFree(data_out_f_gpu);
}


