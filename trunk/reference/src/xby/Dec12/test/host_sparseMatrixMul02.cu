//host_SparseMV
#include "test_comm.h"

// ---------------------------------------------------------------------
__host__ int smat_size( int count_nzmax, int cunt_rows ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * count_nzmax
           + sizeof(t_mindex)  * (cunt_rows + 1);
}

// ---------------------------------------------------------------------
__host__ void set_sparse_data( t_SparseMatrix* m, void* mv ) {

   m->pCol = (t_mindex *) mv;
   m->pNZElement = (t_ve *) (&m->pCol[m->nzmax] ) ;
   m->pRow = (t_mindex *) (&m->pNZElement[m->nzmax]);

}
void host_sparseMatrixMul(t_FullMatrix * pResultVector,t_SparseMatrix *pSparseMatrix, t_FullMatrix * pVector){

	t_SparseMatrix host_SparseMatrix,dev_SparseMatrix;
	t_FullMatrix host_Vector,dev_Vector,host_ResultVector,dev_ResultVector;
	size_t size_NZElement,size_Row,size_Col;
	hipError_t e;	
	int sizeBlock,i;
	
	sizeBlock = VECTOR_BLOCK_SIZE;
	//=====debug==================
	printf("=======in host========== \n");
	printf("pSparseMatrix->m=%d \n",pSparseMatrix->m);
	printf("pSparseMatrix->n=%d \n",pSparseMatrix->n);
	//============================

	// Create an input and output data array on the GPU
	//malloc memory for Input Sparse-Matrix
	printf("malloc sparse Matrix \n");
	dev_SparseMatrix.m = pSparseMatrix->m;
	dev_SparseMatrix.n = pSparseMatrix->n;
	dev_SparseMatrix.nzmax = pSparseMatrix->nzmax;
	int msize = smat_size( dev_SparseMatrix.nzmax, dev_SparseMatrix.n );
    printf(" got result %u \n", msize);
	void *devicemem;
    
	e = hipMalloc ( &devicemem, msize );
	CUDA_UTIL_ERRORCHECK("hipMalloc")
	//pSparseMatrix->pCol is the begin of memery block
	e = hipMemcpy(  devicemem, pSparseMatrix->pCol, msize , hipMemcpyHostToDevice);
   CUDA_UTIL_ERRORCHECK("hipMemcpy")
   set_sparse_data( &dev_SparseMatrix, devicemem);
	//malloc device memory for Input vector
	printf("malloc vector \n");
	size_t size_VElement, size_RElement;
	size_VElement = sizeof(t_ve)*pVector->m*pVector->n;
	size_RElement = sizeof(t_ve)*pSparseMatrix->m;
	hipMalloc( (void **) &(dev_Vector.pElement),size_VElement);
	dev_Vector.m = pVector->m;//host_Vector.m;
	dev_Vector.n = pVector->n;//host_Vector.n;
	hipMemcpy(dev_Vector.pElement,pVector->pElement,size_VElement,hipMemcpyHostToDevice);
	
	printf("malloc output \n");
	//malloc output Vector
	dev_ResultVector.m = pSparseMatrix->m;
	dev_ResultVector.n = 1;
	hipMalloc( (void **) &(dev_ResultVector.pElement),size_RElement);

	// Compute execution configuration using 128 threads per block 
	//for sparseMatrixMul_kernel04
	dim3 dimBlock(sizeBlock);
	//dim3 dimGrid((sizeIn)/dimBlock.x);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp,0);
	printf("number of multiProcessors: %d \n",deviceProp.multiProcessorCount);
	int sizeGrid = deviceProp.multiProcessorCount;
	if (sizeGrid > pSparseMatrix->m)sizeGrid = pSparseMatrix->m;
	
	//for sparseMatrixMul_kernel05
	/*
	int blockX = 32;
	int blockY = 16;
	dim3 dimBlock(blockX,blockY);
	
	if (sizeGrid*blockY > pSparseMatrix->m)sizeGrid = pSparseMatrix->m/blockY;
	if ( (pSparseMatrix->m) % blockY !=0 ) sizeGrid+=1;
	*/
	//================================
	
	printf("grid size = %d\n",sizeGrid);
	dim3 dimGrid(sizeGrid);
	//if ( (sizeA) % sizeBlock !=0 ) dimGrid.x+=1;

	printf("calling kernel \n");
	sparseMatrixMul<<<dimGrid,dimBlock>>>(dev_ResultVector,dev_SparseMatrix,dev_Vector);
	
	e = hipGetLastError();
	if ( e != hipSuccess)
	{
			fprintf(stderr, "CUDA Error on square_elements: '%s' \n", hipGetErrorString(e));
			exit(-1);
	}
	
	printf("get Result \n");
	//hipMemcpy( data_out_host->pElement,pResultVector->pElement,  size_RElement, hipMemcpyDeviceToHost);
	hipMemcpy( pResultVector->pElement,dev_ResultVector.pElement,  size_RElement, hipMemcpyDeviceToHost);
	
	pResultVector->m = pSparseMatrix->m;
	pResultVector->n = 1;
	//=========debug==============
		printf("==================Result in host============\n");
		for( i = 0; i < pResultVector->m; i++) printf("pResultVector->pElement[%d]=%f \n",i,pResultVector->pElement[i]);
	//=======================
	
	printf("free host \n");
	hipFree(devicemem);
	hipFree(dev_Vector.pElement);
	hipFree(dev_ResultVector.pElement);

}


