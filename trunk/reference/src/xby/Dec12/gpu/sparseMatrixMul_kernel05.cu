#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

/* Kernel to square elements of the array on the GPU */
/*	sparseMatrixMul_kernel05.cu
	input Matrix pSparseMatrix  ,input Vector pVector 
	Vector pResultVector output vector 
	C=A*B
description:
	
	each row of A occuppy one block. if gridDim is smaller than the row number of A  
	
	enchance of sparseMatrixMul_kernel03.cu for supprting Vector size biger than Maximu block size
	
*/

__device__ unsigned int getSparseRowIdx(unsigned int gridIdx,unsigned int gridStep,unsigned int blockIdx,unsigned int blockStepY,unsigned int threadIdxY)
{
	return gridIdx*gridStep+blockIdx*blockStepY+threadIdxY;
}
__global__ void sparseMatrixMul(t_FullMatrix pResultVector,t_SparseMatrix pSparseMatrix, t_FullMatrix pVector)
{
	//__shared__ float As[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	__shared__ float Bs[blockDim.x];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	__shared__ float Cs[blockDim.y][blockDim.x];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	//define gridIndex, if gridDim < mA, gridIndex > 0; 
	int gridIndex = 0;
	//int idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
    t_ve *pMatrixElements, *pVectorElements, *pResultElements;
    unsigned int m, n;//, i, j;
    unsigned int *pRow, *pCol;
    //unsigned int colbegin, colend;
    pMatrixElements = pSparseMatrix.pNZElement;
    pVectorElements = pVector.pElement;
    pResultElements = pResultVector.pElement;
    m = pSparseMatrix.m;
    n = pSparseMatrix.n;
	//aBegin,aEnd,aStep are defined for 
	int aBegin = 0;
	int aEnd = pSparseMatrix.m;
	//int aStep = gridDim.x;
	int aStep = gridDim.x*blockDim.y;//gridDim.x
	
	int bBegin = 0;

	int bStep = blockDim.x;
	//int aEnd = mA;
	int bEnd;
    //==check size of Arguments========================================================
    if(m != pResultVector.m*(pResultVector.n)){
        //printf("Result Vector does not match the Matrix\n");
        return;
    }   
    if(n != pVector.m*(pVector.n)){
        //printf("input Vector does not match the Matrix\n");
        return;
    }
	//pRow and pCol may should in share memory or texture
    pRow = pSparseMatrix.pRow;
    pCol = pSparseMatrix.pCol;
    //cal
	//for(int a = aBegin; (a < aEnd)&&((gridIndex*gridDim.x+blockIdx.x)<aEnd); a += aStep, gridIndex++){
	for(int a = aBegin; (a < aEnd)&&((gridIndex*aStep+blockIdx.x)<aEnd); a += aStep, gridIndex++){
		rowIdx = getSparseRowIdx(gridIndex,aStep,blockIdx.x,blockDim.y,threadIdx.y);
		if(threadIdx.x==0){
			//pResultElements[gridIndex*gridDim.x+blockIdx.x]=0;
			pResultElements[rowIdx]=0;
		}
		__syncthreads();
		
		//following is operations within one block 
		// initialize the dot product for each row in A and vector B
		//t_ve blocksum = 0;
		t_ve blocksum[] 
		blocksum[threadIdx.y]= 0;
		//if nB> blockDim, split repeat the
		bBegin = pRow[gridIndex*aStep+blockIdx.x];
		bEnd = pRow[gridIndex*aStep+blockIdx.x + 1];
		for(int b = bBegin; (b < bEnd)&&((threadIdx.x+b) < bEnd); b += bStep ) {
			//initialise Cs 
			//As[threadIdx.x] = 0;
			//Bs[threadIdx.x] = 0;// consider text memory
			Cs[threadIdx.x] = 0;
			__syncthreads();
			// compute scalar product
	
			if (( (gridIndex*gridDim.x+blockIdx.x)<aEnd)&&((b+threadIdx.x) < bEnd)) {
				
				Cs[threadIdx.x] = pMatrixElements[b + threadIdx.x] * pVectorElements[pCol[b + threadIdx.x ]];
			}
			__syncthreads();
				
			if(threadIdx.x == 0){
				int kEnd = bEnd-b;
				if(kEnd > blockDim.x)kEnd = blockDim.x;
				//Because I add Cs[0...k], if blockSize and Matrix does not fit, Parts of Cs[k] are not initialized as 0.  		
				//for (int k = 0; k < kEnd; k++) blocksum += Cs[k];
				for (int k = 0; k < kEnd; k++) blocksum[threadIdx.y] += Cs[threadIdx.y][k];
				
			
			}
			__syncthreads();
			
			//Cs[threadIdx.x] = 0;
			//__syncthreads();	
		}//for b
		__syncthreads();

		if(threadIdx.x == 0) pResultElements[rowIdx] = blocksum[threadIdx.y];//?????????????
		__syncthreads();	
    
	}//for {int a = aBegin;....
}