#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

/* Kernel to square elements of the array on the GPU */
/*	sparseMatrixMul_kernel05.cu
	input Matrix pSparseMatrix  ,input Vector pVector 
	Vector pResultVector output vector 
	C=A*B
description:
	
	each row of A occuppy one block. if gridDim is smaller than the row number of A  
	
	enchance of sparseMatrixMul_kernel03.cu for supprting Vector size biger than Maximu block size
	
*/

__device__ unsigned int getSparseRowIdx(unsigned int gridIdx,unsigned int gridStep,unsigned int blockIdx,unsigned int blockStepY,unsigned int threadIdxY)
{
	return gridIdx*gridStep+blockIdx*blockStepY+threadIdxY;
}
__global__ void sparseMatrixMul(t_FullMatrix pResultVector,t_SparseMatrix pSparseMatrix, t_FullMatrix pVector)
{
	//__shared__ float As[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	__shared__ float Bs[blockDim.x];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	__shared__ float Cs[blockDim.y][blockDim.x];//VECTOR_BLOCK_SIZE shuld equal blockDim 
	//define gridIndex, if gridDim < mA, gridIndex > 0; 
	int gridIndex = 0;
	int tx,ty,bx;
	//int idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
    t_ve *pMatrixElements, *pVectorElements, *pResultElements;
    unsigned int m, n;//, i, j;
    unsigned int *pRow, *pCol;
    tx = threadIdx.x;
	ty = threadIdx.y;
	bx = blockIdx.x;
	//unsigned int colbegin, colend;
    pMatrixElements = pSparseMatrix.pNZElement;
    pVectorElements = pVector.pElement;
    pResultElements = pResultVector.pElement;
    m = pSparseMatrix.m;
    n = pSparseMatrix.n;
	//aBegin,aEnd,aStep are defined for 
	int aBegin = 0;
	int aEnd = pSparseMatrix.m;
	//int aStep = gridDim.x;
	int aStep = gridDim.x*blockDim.y;//gridDim.x
	
	int bBegin = 0;

	int bStep = blockDim.x;
	//int aEnd = mA;
	int bEnd;
    //==check size of Arguments========================================================
    if(m != pResultVector.m*(pResultVector.n)){
        //printf("Result Vector does not match the Matrix\n");
        return;
    }   
    if(n != pVector.m*(pVector.n)){
        //printf("input Vector does not match the Matrix\n");
        return;
    }
	//pRow and pCol may should in share memory or texture
    pRow = pSparseMatrix.pRow;
    pCol = pSparseMatrix.pCol;
    //cal
	//for(int a = aBegin; (a < aEnd)&&((gridIndex*gridDim.x+blockIdx.x)<aEnd); a += aStep, gridIndex++){
	for(int a = aBegin; (a < aEnd)&&((gridIndex*aStep+bx*gridDim.x+ty)<aEnd); a += aStep, gridIndex++){
		rowIdx = getSparseRowIdx(gridIndex,aStep,bx,blockDim.y,ty);
		if(tx==0){
			//pResultElements[gridIndex*gridDim.x+blockIdx.x]=0;
			pResultElements[rowIdx]=0;
		}
		__syncthreads();
		
		//following is operations within one block 
		// initialize the dot product for each row in A and vector B
		//t_ve blocksum = 0;
		t_ve blocksum[blockDim.y]; 
		blocksum[ty]= 0;
		//if nB> blockDim, split repeat the
		bBegin = pRow[rowIdx];
		bEnd = pRow[rowIdx + 1];
		
		for(int b = bBegin; (b < bEnd)&&((tx+b) < bEnd); b += bStep ) {
			//initialise Cs 
			//As[threadIdx.x] = 0;		
			//if(ty==0)
			Bs[tx] = pVectorElements[pCol[b + tx ]];
			Cs[ty][tx] = 0;
			__syncthreads();
			// compute scalar product
	
			if (( (gridIndex*gridDim.x+blockIdx.x)<aEnd)&&((b+threadIdx.x) < bEnd)) {
				
				Cs[ty][tx] = pMatrixElements[b + tx] * pVectorElements[pCol[b + tx ]];//
			}
			__syncthreads();
				
			if(tx == 0){
				int kEnd = bEnd-b;
				if(kEnd > blockDim.x)kEnd = blockDim.x;
				//Because I add Cs[0...k], if blockSize and Matrix does not fit, Parts of Cs[k] are not initialized as 0.  		
				//for (int k = 0; k < kEnd; k++) blocksum += Cs[k];
				for (int k = 0; k < kEnd; k++) blocksum[ty] += Cs[ty][k];
				
			
			}
			
			/*
			int offset; 
			offset = blockDim.x/2;
			while (offset > 0) {
				if(tx < offset) {
					Cs[ty][tx] += Cs[ty][tx + offset];
				}
				offset >>= 1;
				__syncthreads();
			}
			blocksum[ty] += Cs[ty][0];
			__syncthreads();
			*/
		}//for b
		__syncthreads();

		if(tx == 0) pResultElements[rowIdx] = blocksum[ty];//?????????????
		__syncthreads();	
    
	}//for {int a = aBegin;....
}