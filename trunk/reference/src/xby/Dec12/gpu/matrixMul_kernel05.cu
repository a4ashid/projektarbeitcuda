#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"
#include "project_comm.h"

/* Kernel to square elements of the array on the GPU */
/*
	Matrix A is mA x nB  , Vector B is nB
	Vector C output vector in size of mA
	C=A*B
description:
	Release: matrixMul_kernel05.cu
	create 28.Jan.2010
	using share memory to stor B vector
	each row of A occuppy one block. if gridDim is smaller than the row number of A  
*/
__device__ unsigned int getRowBaseIdx(unsigned int gridIdx,unsigned int gridStep,unsigned int blockIdx,unsigned int stepY)
{
	return gridIdx*gridStep+blockIdx*stepY;
}
__global__ void matrixMul( t_ve* C, t_ve* A, t_ve* B, int mA, int nB)
{
	
	//define a Result Vector for each block
	//__shared__ float Cs[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 512
	__shared__ float Cs[VECTOR_BLOCK_Y][VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 512
	__shared__ float Bs[VECTOR_BLOCK_SIZE];
	//define gridIndex, if gridDim < mA, gridIndex > 0; 
	int gridIndex = 0;
	// get a thread indentifier
	//int idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
	int aBegin = 0;
	int bBegin = 0;
	int aStep = gridDim.x*VECTOR_BLOCK_Y;//gridDim.x
	int bStep = VECTOR_BLOCK_SIZE; // blockDim.x
	int aEnd = mA;
	int bEnd = nB;
	int tx,bx,y;
	tx = threadIdx.x;
	bx = blockIdx.x;

    //initialise Cs 
	//for(y = 0; y < VECTOR_BLOCK_Y; y++) Cs[y][tx] = 0;
	//__syncthreads();
	
	//initialize output vector 
	//if(tx==0){
	//	for(y = 0; y < VECTOR_BLOCK_Y; y++)
	//			C[gridIndex*aStep+blockIdx.x*VECTOR_BLOCK_Y+y] = 0;
	//}
	//__syncthreads();
	// if nB > gridDim???????
	rowIdx = getRowBaseIdx(gridIndex,aStep,bx,VECTOR_BLOCK_Y);
	//idx < (gridIndex*gridDim.x+mA%VECTOR_BLOCK_SIZE)*()
	for(int a = aBegin; (a < aEnd)&&((gridIndex*gridDim.x+blockIdx.x)<aEnd); a += aStep, gridIndex++){
		//initialize output vector 
		if(tx==0){
			for(y = 0; y < VECTOR_BLOCK_Y; y++)
				C[rowIdx+y] = 0;
		}
		__syncthreads();
		
		//following is operations within one block 
		// initialize the dot product for each row in A and vector B
		t_ve blocksum = 0;
		//if nB> blockDim, split repeat the
		//for(int b = bBegin; (b < bEnd)&&((threadIdx.x+b) < bEnd); b += bStep ) {
		for(int b = bBegin; b < bEnd; b += bStep ) {
				
		//initialise Cs 
			for(y = 0; y < VECTOR_BLOCK_Y; y++) Cs[y][tx] = 0;
			__syncthreads();
			// compute scalar product
			Bs[tx] = B[b+tx];
			for(y = 0; y < VECTOR_BLOCK_Y; y++)
				if (( (rowIdx +y)<aEnd)&&((b+tx) < bEnd)) {
				//Cs[threadIdx.x] = A[a + blockIdx.x ][b + threadIdx.x] * B[b + threadIdx.x ];
					Cs[threadIdx.x] = A[(rowIdx + y)* nB+ b + tx] * Bs[tx];
				}
			__syncthreads();
			
			if(tx == 0){
				//30.Nov.2009 fixeded for Cs summe
				int kEnd = bEnd-b;
				if(kEnd > VECTOR_BLOCK_SIZE)kEnd = VECTOR_BLOCK_SIZE;
				//Because I add Cs[0...k], if blockSize and Matrix does not fit, Parts of Cs[k] are not initialized as 0.  
				for(y = 0; y < VECTOR_BLOCK_Y; y++){
					for (int k = 1; k < kEnd; k++) Cs[y][0] += Cs[y][k];
					blocksum[y] += Cs[y][0];
				}
			}
			__syncthreads();
			/*
			int offset; 
			offset = VECTOR_BLOCK_SIZE/2;
			for(y = 0; y < VECTOR_BLOCK_Y; y++)
			while (offset > 0) {
				if(tx < offset) {
					Cs[y][tx] += Cs[y][tx + offset];
				}
				offset >>= 1;
				__syncthreads();
			}
			__syncthreads();
			if(threadIdx.x == 0)
			blocksum[y] += Cs[y][0]; //??? blocksum = Cs[0];
			
		}//for b
		__syncthreads();
		*/
		if(tx == 0) 
			for(y = 0; y < VECTOR_BLOCK_Y; y++)C[rowIdx+y] = blocksum[y];
		__syncthreads();	
	}// for a

}