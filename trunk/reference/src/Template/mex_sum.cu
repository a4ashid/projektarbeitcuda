
#include "mex.h"
#include "matrix.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#include "mex_sum_kernel.cu"


void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{

  /* Variable declaration */
  double *ex;
  double *hx;
  double *res;


  /* Local variables */
  int n; 

  
  double *ex_gpu;
  double *hx_gpu;
 
  /* Check for proper number of input and output arguments */    
  if (nrhs != 2) {
	mexErrMsgTxt("2 input arguments required.");
  }
  if (nlhs > 1) {
	mexErrMsgTxt("Too many output arguments.");
  }

  /* Check if Matrix is not Sparse */
  if ((mxGetClassID(prhs[0]) != mxDOUBLE_CLASS) ||
	  (mxGetClassID(prhs[1]) != mxDOUBLE_CLASS)   ) {
	mexErrMsgTxt("Cannot handle sparse arrays in this mex-function.");
  }
  
/*=================================================================
 *  INPUT Arguments
 *=================================================================*/
  

  n = mxGetM(prhs[0]);

  ex = mxGetPr(prhs[0]);

  if (mxGetM(prhs[1]) != n) {
	mexErrMsgTxt("Wrong length of h.");
  }
  hx = mxGetPr(prhs[1]);


/*=================================================================
 *  OUTPUT Arguments
 *=================================================================*/

  plhs[0] = mxCreateDoubleMatrix(n,1,mxREAL);
  res = mxGetPr(plhs[0]);
  
/*=================================================================
 *  CUDA Malloc
 *=================================================================*/
    
    hipMalloc( (void **) &ex_gpu, sizeof(double)*n);
    hipMalloc( (void **) &hx_gpu, sizeof(double)*n);


/*=================================================================
 *  MEMORYCOPY FROM HOST TO DEVICE
 *=================================================================*/
  
  hipMemcpy( ex_gpu, ex, sizeof(double)*n, hipMemcpyHostToDevice);
  hipMemcpy( hx_gpu, hx, sizeof(double)*n, hipMemcpyHostToDevice);


/*=================================================================
 *  START
 *=================================================================*/
  
  dim3 dimBlock(24);
  dim3 dimGrid(n/dimBlock.x);

  
    
    summup<<< dimGrid, dimBlock>>>(hx_gpu, ex_gpu, n);


    hipDeviceSynchronize();



/*=================================================================
 *  FREE MEMORY FROM DEVICE
 *=================================================================*/

    hipMemcpy(res, hx_gpu, sizeof(double)*n, hipMemcpyDeviceToHost);
  
    hipFree(ex_gpu);
    hipFree(hx_gpu);

}
/*=================================================================*/
