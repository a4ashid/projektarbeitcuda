
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define CUDA_UTIL_ERRORCHECK(MSG)        if( e != hipSuccess ) \
        {\
            fprintf(stderr, "*** Error on CUDA operation '%s': '%s'*** \n\n", MSG, hipGetErrorString(e));\
            exit(-3);\
        }\

#ifndef PRJACUDADOUBLE
typedef float        t_ve; /* base type of Matrizes: 'float' or 'double' */
#endif

#ifdef PRJACUDADOUBLE
typedef double       t_ve; /* base type of Matrizes: 'float' or 'double' */
#endif

int N = 10;

__global__ void minikernel(  int N_in, t_ve* out  ) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N_in ) {
       out[i] = 1000 + (t_ve)i;
    }
}


int  main () {
   printf("\n the precision- and compile-option checker \n");
   printf("\n sizeof(t_ve) = %u", sizeof(t_ve));


    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\n\n\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
        printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
    }

    t_ve* hostmem = (t_ve*) malloc(  sizeof(t_ve) * N   );

    if ( hostmem == NULL ) { printf("sorry, can not allocate memory for you"); exit(-1); }

    void* devmem;
    hipError_t e;

    e = hipMalloc ( &devmem , sizeof(int) + sizeof(t_ve) * N );
    CUDA_UTIL_ERRORCHECK("hipMalloc");

//    e = cudaMemset (devmem, 0, sizeof(t_ve) * N );
//    CUDA_UTIL_ERRORCHECK("cudaMalloc");

    dim3 dimGrid( 1 );
    dim3 dimBlock(512);

    int* basevector =  (int*) devmem;

    t_ve* outvec = (t_ve*) &basevector[1];


    minikernel<<<dimGrid,dimBlock>>>( N,  outvec );

    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("minikernel");

    e = hipMemcpy( hostmem, outvec, sizeof(t_ve) * N , hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

    for  ( int i = 0; i < N; i++ ) {
        printf("\nout[%u] = %f", i, hostmem[i]  );
    }

    free(hostmem);
    e = hipFree( devmem);
    CUDA_UTIL_ERRORCHECK("hipMalloc");
}

