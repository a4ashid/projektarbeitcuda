#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"


__host__ size_t smat_size( int cnt_elements, int cnt_cols ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * cnt_elements
           + sizeof(t_mindex)  * (cnt_cols + 1);
}

extern "C" void idrs(

                     t_SparseMatrix A_h,

                     t_ve* b_h,
                     t_mindex s,
                     t_ve  tol,
                     t_mindex maxit,
                     t_ve* x0_h,
                     t_mindex N,


                     t_ve* x_h,  /* output vector */
                     t_ve* resvec_h,
                     t_mindex* piter
                  ) {
    hipError_t e;
    size_t h_memblocksize;
    size_t d_memblocksize;

    t_SparseMatrix A_d;

    void *hostmem;
    void *devmem;

    h_memblocksize =   smat_size( A_h.nzmax, A_h.m )  /* A sparse     */
                     + N * sizeof( t_ve )             /* b full       */
                     ;

    d_memblocksize =  h_memblocksize
                    + N * sizeof( t_ve )            /* x             */
                    + N * sizeof( t_ve )            /* resvec        */
                      ;

    printf("\n using N = %u (full vector size )", N );
    printf("\n using %u bytes in Host   memory", h_memblocksize);
    printf("\n using %u bytes in Device memory", d_memblocksize);

    for ( int i = 0; i < N; i++ ) {
       printf( "\n bh %u %f", i, b_h[i] );
    }



    hostmem =   malloc( h_memblocksize );
    if ( hostmem == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you hostmem"); exit( -1 ); }

/*
      pcol       |  t_mindex  |  .nzmax
      pNZElement |  t_ve      |  .nzmax
      pRow       |  t_mindex  |  N
      b          |  t_ve      |  N
*/

    /* copy all parameter vectors to ony monoliythic block starting at hostmem */

    t_mindex *pcol = (t_mindex *) hostmem;
    memcpy( pcol, A_h.pCol, A_h.nzmax * sizeof(t_mindex) );

    t_ve* pNZElement =  (t_ve *) &pcol[A_h.nzmax] ;
    memcpy( pNZElement, A_h.pNZElement, A_h.nzmax *  sizeof(t_ve) );

    t_mindex* pRow = (t_mindex *) (&pNZElement[A_h.nzmax]);
    memcpy( pRow, A_h.pRow, ( A_h.m + 1 ) *  sizeof(t_mindex) );

    t_ve* b = (t_ve *) &pRow[A_h.m + 1];
    memcpy( b, b_h,  N *  sizeof(t_ve) );

    e = hipMalloc ( &devmem , d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMalloc")

    e = hipMemcpy( devmem, hostmem, h_memblocksize , hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpyHostToDevice");

    A_d.m = A_h.m;
    A_d.n = A_h.n;
    A_d.nzmax = A_h.nzmax;

    A_d.pCol       = (t_mindex *) devmem;
    A_d.pNZElement = (t_ve *) (&A_d.pCol[A_d.nzmax] ) ;
    A_d.pRow       = (t_mindex *) (&A_d.pNZElement[A_d.nzmax]);

    for ( int i = 0; i < N; i++ ) {
       printf( "\n b %u %f", i, b[i] );
    }

    printf("\n*** IDRS.cu - unimplemented - doing nothing  *** \n");


    e = hipFree(devmem);
    CUDA_UTIL_ERRORCHECK("hipFree")
    free( hostmem );

}

