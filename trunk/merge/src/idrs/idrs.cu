#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"


__host__ size_t smat_size( int cnt_elements, int cnt_cols ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * cnt_elements
           + sizeof(t_mindex)  * (cnt_cols + 1);
}

extern "C" void idrs(
                     t_mindex N,
                     t_SparseMatrix A_h,

                     t_ve* b_h,
                     t_mindex s,
                     t_ve  tol,
                     t_mindex maxit,
                     t_ve* x0_h,



                     t_ve* x_h,  /* output vector */
                     t_ve* resvec_h,
                     t_mindex* piter
                  ) {
    hipError_t e;
    size_t h_memblocksize;
    size_t d_memblocksize;

    t_SparseMatrix A_d;

    void *hostmem;
    void *devmem;

    h_memblocksize =   smat_size( A_h.nzmax, A_h.m )  /* A sparse     */
                     + N * sizeof( t_ve )             /* b full       */
                     ;

    d_memblocksize =  h_memblocksize
                    + N * sizeof( t_ve )            /* x             */
                    + N * sizeof( t_ve )            /* resvec        */
                      ;

    printf("\n using N = %u (full vector size )", N );
    printf("\n using %u bytes in Host   memory", h_memblocksize);
    printf("\n using %u bytes in Device memory", d_memblocksize);




    hostmem =   malloc( h_memblocksize );
    if ( hostmem == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you hostmem"); exit( -1 ); }

    e = hipMalloc ( &devmem , d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMalloc")

    e = hipMemcpy( devmem, hostmem, h_memblocksize , hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpyHostToDevice");

    A_d.m = A_h.m;
    A_d.n = A_h.n;
    A_d.nzmax = A_h.nzmax;

    A_d.pCol       = (t_mindex *) devmem;
    A_d.pNZElement = (t_ve *) (&A_d.pCol[A_d.nzmax] ) ;
    A_d.pRow       = (t_mindex *) (&A_d.pNZElement[A_d.nzmax]);


    printf("\n*** IDRS.cu - unimplemented - doing nothing  *** \n");


    e = hipFree(devmem);
    CUDA_UTIL_ERRORCHECK("hipFree")
    free( hostmem );

}

