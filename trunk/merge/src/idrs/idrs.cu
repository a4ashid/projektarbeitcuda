#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "kernels/sparseMatrixMul_kernel.h"


extern "C" size_t idrs_sizetve() {
  return sizeof(t_ve);
}

__global__ void sub_arrays_gpu( t_ve *in1, t_ve *in2, t_ve *out, t_mindex N)
{
    t_mindex i = threadIdx.y * blockDim.x + threadIdx.x;
    if ( i < N )
        out[i] = in1[i] - in2[i];
}

__host__ size_t smat_size( int cnt_elements, int cnt_cols ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * cnt_elements
           + sizeof(t_mindex)  * (cnt_cols + 1);
}


extern "C" void idrs2nd(
    t_FullMatrix P,
    t_ve tol,
    unsigned int s,
    unsigned int maxit,
    t_idrshandle ih_in, /* Context Handle we got from idrs_1st */
    t_ve* x,
    t_ve* resvec,
   unsigned int* piter
) {
    printf("do nothing");
}


/*
__global__ void testsparseMatrixMul( t_FullMatrix pResultVector,t_SparseMatrix pSparseMatrix, t_FullMatrix b ) {

    t_mindex tix = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tix  < pSparseMatrix.m ) {
        //printf ( "\n block %u thread %u tix %u N %u", blockIdx.x, threadIdx.x, tix, pSparseMatrix.m );
        //printf("\n %u %f", tix, b.pElement[tix] );
        pResultVector.pElement[tix] = b.pElement[tix] - 1;
    }
    if ( tix == 0 ) {
        for ( t_mindex i = 0; i < pSparseMatrix.m + 1 ; i++ ) {
             printf("\n pRow[%u] =  %u", i, pSparseMatrix.pRow[i] );
        }
        for ( t_mindex i = 0; i < pSparseMatrix.nzmax ; i++ ) {
            printf("\n pNZElement[%u] =  %f", i, pSparseMatrix.pNZElement[i] );
        }
        for ( t_mindex i = 0; i < pSparseMatrix.nzmax ; i++ ) {
            printf("\n pCol[%u] =  %u", i, pSparseMatrix.pCol[i] );
        }
    }

}
*/

__host__ void set_sparse_data( t_SparseMatrix A_in, t_SparseMatrix* A_out, void* mv ) {

    A_out->m     = A_in.m;
    A_out->n     = A_in.n;
    A_out->nzmax = A_in.nzmax;

    A_out->pCol       = (t_mindex *)  mv;
    A_out->pNZElement = (t_ve *)     (&A_out->pCol[A_out->nzmax] ) ;
    A_out->pRow       = (t_mindex *) (&A_out->pNZElement[A_out->nzmax]);

}

extern "C" void idrs_1st(

                     t_SparseMatrix A_in,    /* A Matrix in buyu-sparse-format */
                     t_ve*          b_in,    /* b as in A * b = x */
                     t_ve*          xe_in,
                     t_mindex N,

                     t_ve*          r_out,    /* the r from idrs.m line 6 : r = b - A*x; */

                     t_idrshandle*  ih_out  /* handle for haloding all the device pointers between matlab calls */

           ) {

    int cnt_multiprozessors;

    hipError_t e;
    size_t h_memblocksize;
    size_t d_memblocksize;

    t_SparseMatrix A_d;

    t_ve* d_tmpAb;
    t_ve* d_b;
    t_ve* d_xe;
    t_ve* d_r;
    t_ve* xe;

    void *hostmem;
    void *devmem;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
        cnt_multiprozessors = deviceProp.multiProcessorCount;
    }


    h_memblocksize =   smat_size( A_in.nzmax, A_in.m )  /* A sparse     */
                     + N * sizeof( t_ve )               /* b full       */
                     + N * sizeof( t_ve )               /* xe        */
                     ;

    d_memblocksize =  h_memblocksize
                    + N * sizeof( t_ve )            /* d_tmpAb         */
                    + N * sizeof( t_ve )            /* d_r             */
                    + N * sizeof( t_ve )            /* x               */
                    + N * sizeof( t_ve )            /* resvec          */

                      ;

    printf("\n using N = %u (full vector size )", N );
    printf("\n using %u bytes in Host   memory", h_memblocksize);
    printf("\n using %u bytes in Device memory", d_memblocksize);



    hostmem =   malloc( h_memblocksize );
    if ( hostmem == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you hostmem"); exit( -1 ); }

/*
      pcol       |  t_mindex  |  .nzmax
      pNZElement |  t_ve      |  .nzmax
      pRow       |  t_mindex  |  N
      b          |  t_ve      |  N
      d_xe       |  t_ve      |  N
      d_tmpAb    |  t_ve      |  N
      d_r        |  t_ve      |  N

*/

    /* copy all parameter vectors to ony monoliythic block starting at hostmem */

    t_mindex *pcol = (t_mindex *) hostmem;
    memcpy( pcol, A_in.pCol, A_in.nzmax * sizeof(t_mindex) );

    t_ve* pNZElement =  (t_ve *) &pcol[A_in.nzmax] ;
    memcpy( pNZElement, A_in.pNZElement, A_in.nzmax *  sizeof(t_ve) );

    t_mindex* pRow = (t_mindex *) (&pNZElement[A_in.nzmax]);
    memcpy( pRow, A_in.pRow, ( A_in.m + 1 ) *  sizeof(t_mindex) );

    t_ve* b = (t_ve *) &pRow[A_in.m + 1];
    memcpy( b, b_in,  N *  sizeof(t_ve) );

    xe = (t_ve *) &b[N];
    memcpy( xe, xe_in,  N *  sizeof(t_ve) );

    e = hipMalloc ( &devmem , d_memblocksize );
    CUDA_UTIL_ERRORCHECK("hipMalloc")

    e = hipMemcpy( devmem, hostmem, h_memblocksize , hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpyHostToDevice");

    set_sparse_data(  A_in, &A_d, devmem );
    d_b     = (t_ve *) &A_d.pRow[A_in.m + 1];
    d_xe    = (t_ve *) &d_b[N];

    d_tmpAb = (t_ve *) &d_xe[N];
    d_r     = (t_ve *) &d_tmpAb[N];

    dim3 dimGrid ( cnt_multiprozessors );
    dim3 dimGridsub( N / 512 + 1 );
    dim3 dimBlock(512);

    /* --------------------------------------------------------------------- */

    t_FullMatrix mxe;
    t_FullMatrix result;

    mxe.m        = N;
    mxe.n        = 1;
    mxe.pElement = d_xe;

    result.pElement = d_tmpAb;
    result.m    = N ;
    result.n    = 1;
    //testsparseMatrixMul<<<dimGrid,dimBlock>>>( result, A_d, mb );
    sparseMatrixMul<<<dimGrid,dimBlock>>>( result, A_d, mxe );
    e = hipGetLastError();
    CUDA_UTIL_ERRORCHECK("testsparseMatrixMul");


//   add_arrays_gpu( t_ve *in1, t_ve *in2, t_ve *out, t_mindex N)
    sub_arrays_gpu<<<dimGridsub,dimBlock>>>( d_b, d_tmpAb, d_r, N);
    /* --------------------------------------------------------------------- */
    e = hipMemcpy( r_out, d_r, sizeof(t_ve) * N, hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK("hipMemcpyDeviceToHost");

    printf("\n*** IDRS.cu - unimplemented - doing nothing  *** \n");


    printf("\n first call of idrs_1st - unimplemented \n\n " );

    *ih_out = 0;

}


