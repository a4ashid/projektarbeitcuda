/* matlab interface for 

				  */


#include <math.h> 
#include "mex.h"

typedef unsigned int t_mindex;
typedef float t_ve;

typedef struct SparseMatrix{
    t_mindex m;
    t_mindex n;
    t_mindex nzmax;
	//size m+1
    t_mindex *pRow;
    //size nzmax
	t_mindex *pCol;
	//size : nzmax
    t_ve* pNZElement;
} t_SparseMatrix;

int smat_size( int count_nzmax, int cunt_rows ) {

    return   ( sizeof(t_ve) + sizeof(t_mindex) ) * count_nzmax
           + sizeof(t_mindex)  * (cunt_rows + 1);
}

// ---------------------------------------------------------------------
 void set_sparse_data( t_SparseMatrix* m, void* mv ) {

   m->pCol = (t_mindex *) mv;
   m->pNZElement = (t_ve *) (&m->pCol[m->nzmax] ) ;
   m->pRow = (t_mindex *) (&m->pNZElement[m->nzmax]);

}
/* Gateway function */
//[x,resvec,iter]=mexInterface_idrs(A,b,s,tol,maxit,x0);
void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[])
{
	int inputIdx;
	t_SparseMatrix A;
	t_ve *b,tol, *x0, *x, *resvec;
	unsigned int s, maxit, N, *piter;
    unsigned int m,n;
    unsigned int nzmax;
    int *ir,*jc,i;
    //unsigned int *irs,*jcs;
    //int cmplx,isfull;
    double *pr,*pi;//,*si,*sr;
    int size_resvec;
	int msize ;
	void *devicemem;
    //=======read input===============================================================
	//read spaser Matrix A
	printf("read spaser Matrix A!\n");
	inputIdx = 0;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
    ir = mxGetIr(prhs[inputIdx]);
    jc = mxGetJc(prhs[inputIdx]);
    nzmax = mxGetNzmax(prhs[inputIdx]);
	
	// inupt matrix rotate
	A.m = n;
	A.n = m;
	A.nzmax = nzmax;
	msize = smat_size( A.nzmax, A.n );
	
	devicemem = mxMalloc ( msize );
	set_sparse_data(&A, devicemem);
	printf("ir0 = %d, pr0 = %lf\n",ir[0],pr[0]);
	for(i = 0; i < nzmax; i++){
        A.pNZElement[i] =(t_ve)pr[i];
        A.pCol[i] = ir[i];
		printf("ir = %d, pr = %lf\n",ir[i],pr[i]);
    }
    for(i = 0; i < n+1; i++){  
        A.pRow[i] = jc[i];
		
    }

	//read b size of N = m*n
	printf("read Vector b!\n");
	inputIdx = 1;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	b = (t_ve*)mxMalloc(sizeof(t_ve)*m*n);
	for(i = 0; i < m*n; i++){
		b[i] = (t_ve)pr[i];
	}
	
	//read s
	printf("read s!\n");
	inputIdx = 2;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	s = (unsigned int)pr[0];
	
	//read tol
	inputIdx = 3;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	tol = (unsigned int)pr[0];
	 
	//read maxit
	inputIdx = 4;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	maxit = (unsigned int)pr[0];
	
	//read x0 size of N = m*n
	inputIdx = 5;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
    x0 = (t_ve*)mxMalloc(sizeof(t_ve)*m*n);
	for(i = 0; i < m*n; i++){
		x0[i] = (t_ve)pr[i];
	}
	//read N
	inputIdx = 6;
	m  = mxGetM(prhs[inputIdx]);
    n  = mxGetN(prhs[inputIdx]);
    pr = mxGetPr(prhs[inputIdx]);
    pi = mxGetPi(prhs[inputIdx]);
	N = (unsigned int)pr[0];
	
	 printf("create output!\n");

	
	//create output vector x of size N
	x = (t_ve*)mxMalloc(sizeof(t_ve)*N);
	//create output vector resvec, size ??????
	resvec = (t_ve*)mxMalloc(sizeof(t_ve)*m*n); 
	//output piter
	piter = (unsigned int *)mxMalloc(sizeof(unsigned int)*1);
	//=======================================================================
	//call idrs interface 
	//idrs(A,b,s,tol,maxit,x0,N,x,resvec,piter);
	
	/*
	extern "C" void idrswhole(
    t_SparseMatrix A_in,    // A Matrix in buyu-sparse-format 
    t_ve*          b_in,    // b as in A * b = x 
    t_mindex s,
    t_ve tol,
    t_mindex maxit,
    t_ve*          x0_in,

    t_mindex N,

    t_ve* x_out,
    t_ve* resvec_out,
    unsigned int* piter
	);
	*/
	//=======================================================================
	//output x, resvec,piter in matlab
	int outPutIdx;
	//x
	outPutIdx = 0;
    plhs[outPutIdx] = mxCreateDoubleMatrix(N,1,mxREAL);
	pr = mxGetPr(plhs[outPutIdx]);
	for(i = 0; i < N; i++){
		pr[i] = (double)x[i];
	}
	//output resvec
	outPutIdx = 1;
    size_resvec =10;
    plhs[outPutIdx] = mxCreateDoubleMatrix(size_resvec,1,mxREAL);
	pr = mxGetPr(plhs[outPutIdx]);
	for(i = 0; i < size_resvec; i++){
		pr[i] = (double)resvec[i];
	}
	//output iter of scalar value
	outPutIdx = 2;
	//plhs[outPutIdx] = mxCreateNumericMatrix(1,1,mxUINT32_CLASS,mxREAL);
	plhs[outPutIdx] = mxCreateDoubleMatrix(1,1,mxREAL);
	pr = mxGetPr(plhs[outPutIdx]);
	for(i = 0; i < 1; i++){
		pr[i] = piter[i];
	}
	//=======================================================================
	mxFree(b);
	mxFree(x);
	mxFree(resvec);
	mxFree(piter);
	mxFree(devicemem);
	
	
}