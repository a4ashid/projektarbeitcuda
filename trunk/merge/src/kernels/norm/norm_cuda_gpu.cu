#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"



__global__ void kernel_norm(t_ve* in,t_ve* out )
{
    __shared__ t_ve Vs [DEF_BLOCKSIZE];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    Vs[threadIdx.x] = in[idx] * in[idx];


    __syncthreads();
    if ( threadIdx.x < 256 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  + 256 ]; }
    __syncthreads();

    if ( threadIdx.x < 128 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  + 128 ];}
    __syncthreads();

    if ( threadIdx.x <  64 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  +  64 ];}
    __syncthreads();


#ifndef PRJCUDAEMU

    if ( threadIdx.x <  32 ) {
        Vs[threadIdx.x] += Vs[ threadIdx.x + 32 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x + 16 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  8 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  4 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  2 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  1 ];

        if ( threadIdx.x == 0 ) {
            //out[blockIdx.x] =  Vs[0]  ;
            out[blockIdx.x] =  Vs[0]  ;
        }
    }

#endif

#ifdef PRJCUDAEMU

    if ( threadIdx.x <  32 ) {
        Vs[threadIdx.x] += Vs[ threadIdx.x + 32 ];
    }
    __syncthreads();

    if ( threadIdx.x <  16 ) {
        Vs[threadIdx.x] += Vs[ threadIdx.x + 16 ];
    }
    __syncthreads();

    if ( threadIdx.x <  8 ) {

        Vs[threadIdx.x] += Vs[ threadIdx.x +  8 ];
    }
    __syncthreads();

    if ( threadIdx.x <  4 ) {
        Vs[threadIdx.x] += Vs[ threadIdx.x +  4 ];
    }
    __syncthreads();

    if ( threadIdx.x <  2 ) {
        Vs[threadIdx.x] += Vs[ threadIdx.x +  2 ];
    }
    __syncthreads();

    if ( threadIdx.x <  1 ) {
        Vs[threadIdx.x] += Vs[ threadIdx.x +  1 ];
    }
    __syncthreads();

    if ( threadIdx.x == 0 ) {
            //out[blockIdx.x] =  Vs[0]  ;
            out[blockIdx.x] =  Vs[0]  ;
    }


#endif

}


__host__ void dbg_norm_checkresult ( t_ve *in1,

                                     t_ve tobeckecked,
                                     t_mindex N ,
                                     char* debugname
                                      )

                          {

    hipError_t e;


    t_ve* v1 = (t_ve*) malloc( sizeof( t_ve ) * N );
    if (  v1 == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you C"); exit( -1 ); }



    e = hipMemcpy( v1, in1, sizeof(t_ve) * N , hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");


    t_ve calresult = 0;

    for( t_mindex i = 0; i < N; i++ ) {
        calresult += v1[i] * v1[i];
    }

    t_ve calnorm = sqrt(calresult);
    t_ve tolerance = calnorm / 10000;
//    t_ve tolerance = 0.0001;
    if ( abs( calnorm - tobeckecked ) > tolerance ) {
//        printf("\n Norm %s OK", debugname );
//    }
//    else {
        printf("\n Norm %s *not* OK :  expected  %f (CPU), got %f (GPU) (tolerance %f)", debugname, calnorm, tobeckecked, tolerance );
        for( t_mindex i = 0; i < N; i++ ) {
            printf("\n in[%u]=%f", i, v1[i] );
        }
        printf("\n Norm %s *not* OK :  expected  %f (CPU), got %f (GPU) (tolerance %f)", debugname, calnorm, tobeckecked, tolerance );
        exit( - 1 );
    }

    free(v1);


}


