#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"



__global__ void kernel_norm(t_ve* in,t_ve* out )
{
    __shared__ t_ve Vs [DEF_BLOCKSIZE];


    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    Vs[threadIdx.x] = in[idx] * in[idx];


    __syncthreads();
    if ( threadIdx.x < 256 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  + 256 ]; }
    __syncthreads();

    if ( threadIdx.x < 128 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  + 128 ];}
    __syncthreads();

    if ( threadIdx.x <  64 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  +  64 ];}
    __syncthreads();


#ifndef PRJCUDAEMU

    if ( threadIdx.x <  32 ) {
        Vs[threadIdx.x] += Vs[ threadIdx.x + 32 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x + 16 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  8 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  4 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  2 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  1 ];

        if ( threadIdx.x == 0 ) {
            //out[blockIdx.x] =  Vs[0]  ;
            out[blockIdx.x] =  Vs[0]  ;
        }
    }

#endif

#ifdef PRJCUDAEMU

    if ( threadIdx.x <  32 )
        Vs[threadIdx.x] += Vs[ threadIdx.x + 32 ];
    __syncthreads();
    if ( threadIdx.x <  16 )
        Vs[threadIdx.x] += Vs[ threadIdx.x + 16 ];
    __syncthreads();
    if ( threadIdx.x <  8 )
        Vs[threadIdx.x] += Vs[ threadIdx.x +  8 ];
    __syncthreads();
    if ( threadIdx.x <  4 )
        Vs[threadIdx.x] += Vs[ threadIdx.x +  4 ];
    __syncthreads();
    if ( threadIdx.x <  2 )
        Vs[threadIdx.x] += Vs[ threadIdx.x +  2 ];
    __syncthreads();
    if ( threadIdx.x <  1 )
        Vs[threadIdx.x] += Vs[ threadIdx.x +  1 ];
    __syncthreads();
        if ( threadIdx.x == 0 ) {
            //out[blockIdx.x] =  Vs[0]  ;
            out[blockIdx.x] =  Vs[0]  ;
        }


#endif

}

