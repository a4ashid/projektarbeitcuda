#include "hip/hip_runtime.h"

#include "projektcuda.h"
#include <stdlib.h>
#include <stdio.h>

__global__ void device_gauss_solver( t_ve* p_Ab, unsigned int N, t_ve* p_x )
{


    __shared__ unsigned int i;
    __shared__ unsigned int max;

    __shared__ t_ve Ab[ GAUSS_NMAX * ( GAUSS_NMAX + 1) ];
    __shared__ t_ve x[ GAUSS_NMAX ];

    unsigned int tidx =  threadIdx.x;


    if ( threadIdx.x < N + 1 ) {
        for ( short l = 0; l < N ; l++ ) {
           short ao = ( threadIdx.x * N + l );
           Ab[ao] = p_Ab[ao];
        }
    }

    if ( tidx == 0 ) { i = 1; }

    __syncthreads();

    while ( i <= N ) {                  /* for ( i = 1; i <= N ; i++ ) */

       if ( tidx == 0 ) {
            unsigned int j;
            max = i;

            for( j = i + 1; j <= N; j++ ) {
                if ( abs( Ab[ ab(j,i) ] ) > abs( Ab[ ab(max,i) ] )  ) {
                    max = j;
                }
            }
       }
       __syncthreads();

       unsigned int k = tidx + 1;

       if ( ( k >= i ) && ( k <= N + 1 ) ) {
           t_ve t          = Ab[ ab(i  ,k) ];
           Ab[ ab(i,k)   ] = Ab[ ab(max,k) ];
           Ab[ ab(max,k) ] = t;
      }

      __syncthreads();

      {
          unsigned int j = threadIdx.x + 1;
          if (  ( j >= i +1 ) && ( j <= N ) && threadIdx.y == 0 ) {       /*   for ( j = i +1; j <= N ; j++ ) */
              unsigned int  k ;
              for ( k = N + 1; k >= i ; k-- ) {
                 Ab[ ab(j,k) ] -= Ab[ ab(i,k) ] * Ab[ ab(j,i) ] /  Ab[ ab(i, i) ];
              }
           }
       }
       __syncthreads();
       if ( tidx == 0 ) { i++; }
       __syncthreads();
    }
    __syncthreads();

    if ( tidx == 0 ) {

        /* the substitute part */
        unsigned int j,k;
        for (j = N; j >= 1; j-- ) {
            t_ve t = 0.0;
            for ( k = j + 1; k <= N; k++ ) {
                t +=  Ab[ ab(j,k) ] * x[ k - 1 ];
            }
            x[ j - 1 ] = ( Ab[ ab(j,N+1) ] - t ) / Ab[ ab(j,j) ] ;
        }

    }
    __syncthreads();
    if ( threadIdx.x < N ) {
        p_x[threadIdx.x] = x[threadIdx.x];
    }

   __syncthreads();
}


__host__ void dbg_solver_check_result( t_ve* Ab_in, t_mindex N, t_ve* x_in ) {

    hipError_t e;

    //return ;

    t_ve* Ab = (t_ve*) malloc( sizeof( t_ve ) * (N+1) * N );
    if ( Ab == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you Ab"); exit( -1 ); }
    t_ve* x  = (t_ve*) malloc( sizeof( t_ve ) * N );

    e = hipMemcpy( Ab, Ab_in, sizeof(t_ve) * (N+1) * N , hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

    e = hipMemcpy( x, x_in, sizeof(t_ve) * (N), hipMemcpyDeviceToHost);
    CUDA_UTIL_ERRORCHECK(" hipMemcpy debugbuffer");

// -------------------------------------------------------------

   t_mindex i ;
   t_mindex j;

    for ( j = 1; j <= N; j++ ) {
        t_ve sum = 0;
        for ( i = 1; i <= N; i++ ) {
            sum += Ab[ ab(j,i) ] * x[ (i-1) ] ;
        }
        //printf("\n %u %f   b %f", j, sum, p_Ab[ ab(j,N+1) ] );
        //if ( sum != Ab[ ab(j,N+1) ] ) {
        t_ve tolerance;
        t_ve diff;
        diff = abs( sum - Ab[ ab(j,N+1)] );
        if ( abs( Ab[ ab(j,N+1) ] ) > 1 ) {
           tolerance = abs( Ab[ ab(j,N+1)] / 50 );
        }
        else {
            tolerance = 0.01;
        }
        if ( diff  > tolerance ) {
            printf("\n Gauss Solver check not ok row=%u, sum %f   b=%f (tol=%f, diff=%f )", j, sum , Ab[ ab(j,N+1)], tolerance, diff  );

            for ( int k = 1; k <=N; k++ ) {
                printf("\n b[%u]=%f ", k, Ab[ ab(k,N+1) ] );
            }
            for ( int s = 1; s <=N; s++ ) {
                for ( int k = 1; k <=N; k++ ) {
                    printf("\n A(%u,%u)=%f ", k, s, Ab[ ab(k,s) ] );
                }
            }
            for ( int k = 1; k <=N; k++ ) {
                printf("\n x[%u]=%f ", k, x[ (k-1) ] );
            }

            exit(-1); /*  needs to be changed to retunr instead of die!!! */
        }
    }
// -------------------------------------------------------------

    free(x);
    free(Ab);

}

