#include "hip/hip_runtime.h"
#include <stdio.h>
#include "projektcuda.h"


/* Kernel to square elements of the array on the GPU */
/*
	input Matrix pSparseMatrix  ,input Vector pVector
	Vector pResultVector output vector
	C=A*B
description:
	each row of A occuppy one block. if gridDim is smaller than the row number of A
Release:
	enhancedSparseMatrixMul_kernel.cu
*/
__global__ void enhancedSparseMatrixMul(t_FullMatrix pResultVector,t_SparseMatrix pSparseMatrix, t_FullMatrix pVector,t_ve n)
{
	__shared__ t_ve Cs[VECTOR_BLOCK_SIZE];//VECTOR_BLOCK_SIZE shuld equal blockDim 512
	//define gridIndex, if gridDim < mA, gridIndex > 0;
	int gridIndex = 0;
	//int idx = gridIndex*gridDim.x + blockIdx.x*blockDim.x+threadIdx.x;
    t_ve *pMatrixElements, *pVectorElements, *pResultElements;
    unsigned int m, n;//, i, j;
    unsigned int *pRow, *pCol;
    //unsigned int colbegin, colend;
    pMatrixElements = pSparseMatrix.pNZElement;
    pVectorElements = pVector.pElement;
    pResultElements = pResultVector.pElement;
    m = pSparseMatrix.m;
    n = pSparseMatrix.n;
	int aBegin = 0;
	int aEnd = pSparseMatrix.m;
	int bBegin = 0;
	//int aStep = gridDim.x;
	int bStep = VECTOR_BLOCK_SIZE; // blockDim.x
	//int aEnd = mA;
	int bEnd;
    //==check size of Arguments========================================================
    if(m != pResultVector.m*(pResultVector.n)){
        //printf("Result Vector does not match the Matrix\n");
        return;
    }
    if(n != pVector.m*(pVector.n)){
        //printf("input Vector does not match the Matrix\n");
        return;
    }
	//pRow and pCol may should in share memory or texture
    pRow = pSparseMatrix.pRow;
    pCol = pSparseMatrix.pCol;
    //cal

		if(threadIdx.x==0){
			pResultElements[blockIdx.x]=0;
		//C[gridIndex*gridDim.x+blockIdx.x]=0;
		}
		//following is operations within one block
		// initialize the dot product for each row in A and vector B
		t_ve blocksum = 0;
		//if nB> blockDim, split repeat the
		bBegin = pRow[blockIdx.x];
		bEnd = pRow[blockIdx.x + 1];
		for(int b = bBegin; (b < bEnd)&&((threadIdx.x+b) < bEnd); b += bStep ) {

			Cs[threadIdx.x] = 0;
			__syncthreads();

			if (( (gridIndex*gridDim.x+blockIdx.x)<aEnd)&&((b+threadIdx.x) < bEnd)) {

				Cs[threadIdx.x] = n*pMatrixElements[b + threadIdx.x] * pVectorElements[pCol[b + threadIdx.x ]];
			}
			__syncthreads();

			if(threadIdx.x == 0){
				int kEnd = bEnd-b;
				if(kEnd > VECTOR_BLOCK_SIZE)kEnd = VECTOR_BLOCK_SIZE;
				//Because I add Cs[0...k], if blockSize and Matrix does not fit, Parts of Cs[k] are not initialized as 0.
				for (int k = 0; k < kEnd; k++) blocksum += Cs[k];
				//blocksum = 2;

			}
			__syncthreads();

			//Cs[threadIdx.x] = 0;
			//__syncthreads();
		}//for b
		__syncthreads();

		if(threadIdx.x == 0) pResultElements[blockIdx.x] = blocksum;//?????????????
		__syncthreads();


}