#include "hip/hip_runtime.h"

#include "projektcuda.h"



__global__ void kernel_dotmul( t_ve *in1,
                               t_ve *in2,
                               t_ve *out
                             ) {
    __shared__ t_ve Vs [DEF_BLOCKSIZE];


    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    Vs[threadIdx.x] = in1[idx] * in2[idx];


    __syncthreads();
    if ( threadIdx.x < 256 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  + 256 ]; }
    __syncthreads();

    if ( threadIdx.x < 128 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  + 128 ];}
    __syncthreads();

    if ( threadIdx.x <  64 ) { Vs[threadIdx.x] += Vs[ threadIdx.x  +  64 ];}
    __syncthreads();


    if ( threadIdx.x <  32 ) {
        Vs[threadIdx.x] += Vs[ threadIdx.x + 32 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x + 16 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  8 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  4 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  2 ];
        Vs[threadIdx.x] += Vs[ threadIdx.x +  1 ];

        if ( threadIdx.x == 0 ) {
            out[blockIdx.x] =  Vs[0]  ;
        }
    }
}

