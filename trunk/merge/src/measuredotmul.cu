#include <stdlib.h>
#include <stdio.h>

#include "projektcuda.h"

#include "measurehelp.h"

#include "kernels/dotMul_cuda_gpu.h"

#include <time.h>


__host__ t_ve cpu_imp( t_mindex N, t_ve *in1, t_ve *in2 ) {

    t_ve calresult = 0;

    for( t_mindex i = 0; i < N; i++ ) {
        calresult += in1[i] * in2[i];
    }
    return calresult;
}

__host__ void dodotmul (  t_mindex N_in ) {

    hipError_t e;
    pt_ve v1, v2, vout, vd1, vd2, vdout;

    size_t devsize = SIZE_VE *  ( N_in + 512 ) * 3 ;

    v1 = ( pt_ve ) malloc( devsize );
    if (  v1 == NULL ) { fprintf(stderr, "sorry, can not allocate memory for you P.pElement"); exit( -1 ); }


    memset( v1, 0, devsize );

    v2   = &v1[ N_in + 512];
    vout = &v2[ N_in + 512 ];

    for ( t_mindex i = 0; i < N_in; i++ ) {
        v1[i]   = 1;
        v2[i]   = 2;
        vout[i] = 0;
    }

    dim3 dimBlock(512);
    dim3 dimGridsub( N_in / 512 + 1 );

    e = hipMalloc ( &vd1 , devsize );
    CUDA_UTIL_ERRORCHECK("hipMalloc");

    e = hipMemcpy( vd1, v1, devsize, hipMemcpyHostToDevice);
    CUDA_UTIL_ERRORCHECK("hipMemcpy");

    vd2   = &vd1[ N_in + 512 ];
    vdout = &vd2[ N_in + 512 ];



   float gpudot_ms;

   {
            START_CUDA_TIMER

            kernel_dotmul<<<dimGridsub,dimBlock>>>( vd1, vd2, vdout ) ;
            e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("device_dotMul");

             e = hipMemcpy( vout, vdout, SIZE_VE *  ( N_in / 512 + 1 ) , hipMemcpyDeviceToHost);
             CUDA_UTIL_ERRORCHECK("hipMemcpy( h_om1, om1, sizeof(t_ve) * N * 2, hipMemcpyDeviceToHost)");

             t_ve  sum = 0;

              for ( t_mindex blockidx = 0; blockidx < N_in/ 512 + 1; blockidx++ ) {
                  sum += vout[blockidx];
              }
              //printf("GPU result: %f", sum );

              STOP_CUDA_TIMER( &gpudot_ms )

    }

   float gpudotwom_ms;

   {
            START_CUDA_TIMER

            kernel_dotmul<<<dimGridsub,dimBlock>>>( vd1, vd2, vdout ) ;
            e = hipGetLastError();
            CUDA_UTIL_ERRORCHECK("device_dotMul");

            // e = hipMemcpy( vout, vdout, SIZE_VE *  ( N_in / 512 + 1 ) , hipMemcpyDeviceToHost);
            // CUDA_UTIL_ERRORCHECK("hipMemcpy( h_om1, om1, sizeof(t_ve) * N * 2, hipMemcpyDeviceToHost)");

             t_ve  sum = 0;

              //for ( t_mindex blockidx = 0; blockidx < N_in/ 512 + 1; blockidx++ ) {
              //    sum += vout[blockidx];
              //}
              //printf("GPU result: %f", sum );

              STOP_CUDA_TIMER( &gpudotwom_ms )

    }

   float cpudot_ms;

    {
         START_CUDA_TIMER
         t_ve cpures = cpu_imp( N_in, v1, v2 );
         STOP_CUDA_TIMER( &cpudot_ms )
    }


    e = hipFree( vd1 );
    CUDA_UTIL_ERRORCHECK("e = hipFree( devmem );");

    free( v1 );

    printf("\n%u\t%f\t%f\t%f", N_in, gpudot_ms, cpudot_ms, gpudotwom_ms  );
}

int main( int argc, char *argv[] )
{
   printf("\n measure dotmul");
   printf( "\n Build configuration: sizeof(t_ve) = %u \n", sizeof(t_ve));



    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);



        printf("\nDevice %d: \"%s\"\n \n", dev, deviceProp.name);
        printf("  Number of multiprocessors:                     %d\n", deviceProp.multiProcessorCount);
        printf("  CUDA Capability Major revision number:         %d\n", deviceProp.major);
        printf("  CUDA Capability Minor revision number:         %d\n", deviceProp.minor);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    }




   t_mindex order  = 3;

   if ( argc > 1 ) {
      order = atoi( argv[1] );
   }
   t_mindex  nbase = 10;
   for ( t_mindex o = 0; o < order; o++ ) {
      t_mindex n = nbase;
       for ( int i = 0; i < 9; i++ ) {
           dodotmul( n );
           n += nbase;
       }
       nbase *= 10;
   }
}

